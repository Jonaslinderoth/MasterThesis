#include "hip/hip_runtime.h"
#include "MineClusKernels.h"
#include <assert.h>
#include <iostream>

/*
  Naive kernel for creating the itemSet. 
  
*/
__global__ void createItemSet(float* data, unsigned int dim, unsigned int numberOfPoints, unsigned int centroidId, float width, unsigned int* output){
	unsigned int point = blockIdx.x*blockDim.x+threadIdx.x;
	if(point < numberOfPoints){
		unsigned int numberOfOutputs = ceilf((float)dim/32);
		for(unsigned int i = 0; i < numberOfOutputs; i++){
			unsigned int output_block = 0;
			for(unsigned int j = 0; j < 32; j++){
				if(i == numberOfOutputs-1 && j >= dim%32){
					break;
				}else{
					assert(dim*centroidId+i*32+j < numberOfpoints*dim);
					assert(point*dim+i*32+j < numberOfpoints*dim);
					output_block |= ((abs(data[dim*centroidId+i*32+j] - data[point*dim+i*32+j]) < width) << j);
				}
			}
			assert(numberOfPoints*i+point < numberOfPoints*ceilf(dim/32));
			output[numberOfPoints*i+point] = output_block;
		}	
	}
}


std::vector<unsigned int> createItemSetTester(std::vector<std::vector<float>*>* data, unsigned int centroid, float width){
	uint size = data->size();
	uint dim = data->at(0)->size();
	uint size_of_data = size*dim*sizeof(float);
	size_t size_of_output = size*ceilf((float)dim/32)*sizeof(unsigned int);
	float* data_h;
	unsigned int* output_h;
	hipHostMalloc((void**) &data_h, size_of_data);
	hipHostMalloc((void**) &output_h, size_of_output);
	for(int i = 0; i < size; i++){
		for(int j = 0; j < dim; j++){
			data_h[i*dim+j] = data->at(i)->at(j);
		}
	}
	float* data_d;
	hipMalloc((void**) &data_d, size_of_data);
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
		
	unsigned int* output_d;
	hipMalloc((void**) &output_d, size_of_output);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)size/dimBlock);
	createItemSet<<<dimGrid, dimBlock>>>(data_d, dim, size, centroid, width, output_d);

	
	
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*size;i++){
		res.push_back(output_h[i]);
	}
	return res;
}


