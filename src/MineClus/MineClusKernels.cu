#include "hip/hip_runtime.h"
#include "MineClusKernels.h"
#include <assert.h>
#include <iostream>

/*
  Naive kernel for creating the itemSet. 
  
*/
__global__ void createItemSet(float* data, unsigned int dim, unsigned int numberOfPoints, unsigned int centroidId, float width, unsigned int* output){
	unsigned int point = blockIdx.x*blockDim.x+threadIdx.x;
	if(point < numberOfPoints){
		unsigned int numberOfOutputs = ceilf((float)dim/32);
		for(unsigned int i = 0; i < numberOfOutputs; i++){
			unsigned int output_block = 0;
			for(unsigned int j = 0; j < 32; j++){
				if(i == numberOfOutputs-1 && j >= dim%32){
					break;
				}else{
					assert(dim*centroidId+i*32+j < numberOfpoints*dim);
					assert(point*dim+i*32+j < numberOfpoints*dim);
					output_block |= ((abs(data[dim*centroidId+i*32+j] - data[point*dim+i*32+j]) < width) << j);
				}
			}
			assert(numberOfPoints*i+point < numberOfPoints*ceilf(dim/32));
			output[numberOfPoints*i+point] = output_block;
		}	
	}
}


std::vector<unsigned int> createItemSetTester(std::vector<std::vector<float>*>* data, unsigned int centroid, float width){
	uint size = data->size();
	uint dim = data->at(0)->size();
	uint size_of_data = size*dim*sizeof(float);
	size_t size_of_output = size*ceilf((float)dim/32)*sizeof(unsigned int);
	float* data_h;
	unsigned int* output_h;
	hipHostMalloc((void**) &data_h, size_of_data);
	hipHostMalloc((void**) &output_h, size_of_output);
	for(int i = 0; i < size; i++){
		for(int j = 0; j < dim; j++){
			data_h[i*dim+j] = data->at(i)->at(j);
		}
	}
	float* data_d;
	hipMalloc((void**) &data_d, size_of_data);
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
		
	unsigned int* output_d;
	hipMalloc((void**) &output_d, size_of_output);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)size/dimBlock);
	createItemSet<<<dimGrid, dimBlock>>>(data_d, dim, size, centroid, width, output_d);

	
	
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*size;i++){
		res.push_back(output_h[i]);
	}
	return res;
}


__global__ void createInitialCandidates(unsigned int dim, unsigned int* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocksPrPoint = ceilf((float)dim/32);
	unsigned int myBlock = candidate/32;
	// make sure all are 0;
	for(int i = 0; i < numberOfBlocksPrPoint; i++){
		output[candidate+dim * i] = 0;
	}
	// set the correct candidate
	unsigned int output_block = (1 << (candidate%32));
	output[candidate+dim*myBlock] = output_block;
}


std::vector<unsigned int> createInitialCandidatesTester(unsigned int dim){
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)dim/dimBlock);

	size_t sizeof_output = dim*ceilf((float)dim/32)*sizeof(unsigned int);

	unsigned int* output_h = (unsigned int*) malloc(sizeof_output);
	unsigned int* output_d;

	hipMalloc((void**) &output_d, sizeof_output);

	createInitialCandidates<<<dimGrid, dimBlock>>>(dim, output_d);

	hipMemcpy(output_h, output_d, sizeof_output, hipMemcpyDeviceToHost);
	
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*dim;i++){
		res.push_back(output_h[i]);
	}
	return res;
	
}