#include "hip/hip_runtime.h"
#include "MineClusKernels.h"
#include <assert.h>
#include <iostream>
#include <tuple>
#include "../randomCudaScripts/Utils.h"
#include "../MineClusGPU/HashTable.h"

/*
  Naive kernel for creating the itemSet. 
  Takes the data, and a index for a centroid, and creates the itemSet 
  The items are stored with the points as columns, and the dimensions as rows, 
  and then a row major fasion
*/
__global__ void createItemSet(float* data, unsigned int dim, unsigned int numberOfPoints, unsigned int centroidId, float width, unsigned int* output){
	unsigned int point = blockIdx.x*blockDim.x+threadIdx.x;
	if(point < numberOfPoints){
		unsigned int numberOfOutputs = ceilf((float)dim/32);
		// For each of the blocks in the output in this dimension
		for(unsigned int i = 0; i < numberOfOutputs; i++){
			unsigned int output_block = 0;
			// for each bit in a block
			for(unsigned int j = 0; j < 32; j++){
				// break if the last block dont line up with 32 bits
				if(i == numberOfOutputs-1 && j == dim%32 && j != 0){
					break;
				}else{
					assert(dim*centroidId+i*32+j < numberOfPoints*dim);
					assert(point*dim+i*32+j < numberOfPoints*dim);
					// Check if the dimension are within the width, and write to block in register
					output_block |= ((abs(data[dim*centroidId+i*32+j] - data[point*dim+i*32+j]) < width) << j);
				}
			}
			// write block to global memory.
			assert(numberOfPoints*i+point < numberOfPoints*ceilf((float)dim/32));
			output[numberOfPoints*i+point] = output_block;
		}	
	}
}

/**
Thin wrapper for createItemSet
*/
void createItemSetWrapper(unsigned int dimGrid,
						  unsigned int dimBlock,
						  hipStream_t stream,
						  float* data,
						  unsigned int dim,
						  unsigned int numberOfPoints,
						  unsigned int centroidId,
						  float width,
						  unsigned int* output){
	createItemSet<<<dimGrid, dimBlock, 0, stream>>>(data, dim, numberOfPoints, centroidId, width, output);
}


/**
   This function is only for testing that the kernel works correctly
*/
std::vector<unsigned int> createItemSetTester(std::vector<std::vector<float>*>* data, unsigned int centroid, float width){
	uint size = data->size();
	uint dim = data->at(0)->size();
	uint size_of_data = size*dim*sizeof(float);
	size_t size_of_output = size*ceilf((float)dim/32)*sizeof(unsigned int);
	float* data_h;
	unsigned int* output_h;
	hipHostMalloc((void**) &data_h, size_of_data);
	hipHostMalloc((void**) &output_h, size_of_output);
	for(int i = 0; i < size; i++){
		for(int j = 0; j < dim; j++){
			data_h[i*dim+j] = data->at(i)->at(j);
		}
	}
	float* data_d;
	hipMalloc((void**) &data_d, size_of_data);
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
		
	unsigned int* output_d;
	hipMalloc((void**) &output_d, size_of_output);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)size/dimBlock);
	createItemSet<<<dimGrid, dimBlock>>>(data_d, dim, size, centroid, width, output_d);

	
	
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*size;i++){
		res.push_back(output_h[i]);
	}
	return res;
}


/**
   Creates the initial candidtes given the dimensions. 
*/
__global__ void createInitialCandidates(unsigned int dim, unsigned int* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocksPrPoint = ceilf((float)dim/32);
	unsigned int myBlock = candidate/32;
	
	if(candidate < dim){
	
		// make sure all are 0;
		for(int i = 0; i < numberOfBlocksPrPoint; i++){
			assert(candidate+dim * i < dim*(ceilf((float)dim/32)));
			output[candidate+dim * i] = 0;
		}
		// set the correct candidate
		unsigned int output_block = (1 << (candidate%32));
		//printf("candidat %u have value %u, writing output at position %u\n", candidate, output_block, candidate+dim*myBlock);
		output[candidate+dim*myBlock] = output_block;
	}
}


/**
   Thin wrapper for CreateInitialCandidates
*/
void createInitialCandidatesWrapper(unsigned int dimGrid,
									unsigned int dimBlock,
									hipStream_t stream,
									unsigned int dim,
									unsigned int* output
									){
	createInitialCandidates<<<dimGrid, dimBlock, 0, stream>>>(dim, output);
}

/**
   This function is only for testing
*/
std::vector<unsigned int> createInitialCandidatesTester(unsigned int dim){
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)dim/dimBlock);

	size_t sizeof_output = dim*ceilf((float)dim/32)*sizeof(unsigned int);

	unsigned int* output_h = (unsigned int*) malloc(sizeof_output);
	unsigned int* output_d;

	hipMalloc((void**) &output_d, sizeof_output);

	createInitialCandidates<<<dimGrid, dimBlock>>>(dim, output_d);

	hipMemcpy(output_h, output_d, sizeof_output, hipMemcpyDeviceToHost);
	
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*dim;i++){
		res.push_back(output_h[i]);
	}
	return res;
}

__global__ void countSupport(unsigned int* candidates, unsigned int* itemSet,
							 unsigned int dim, unsigned int numberOfItems,
							 unsigned int numberOfCandidates,
							 unsigned int minSupp, float beta,
							 unsigned int* outSupp, float* outScore,
							 bool* outToBeDeleted){
	
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;

	unsigned int numberOfBlocksPrItem = ceilf((float)dim/32);
	
	if(candidate < numberOfCandidates){
		unsigned int count = 0;
		for(unsigned int i = 0; i < numberOfItems; i++){
			bool isSubset = true;
			for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
				unsigned int itemBlock = itemSet[j*numberOfItems+i];
				unsigned int candidateBlock = candidates[j*numberOfCandidates + candidate];
				unsigned int candidateBlockCount = __popc(candidateBlock);
				unsigned int unionCount = __popc(itemBlock&candidateBlock);
				// if(candidate == 0){
				// 	printf("candiate %u item value %u \n", candidate, itemBlock);
				// }
				isSubset &= candidateBlockCount == unionCount;
			}
			
			count += isSubset;
		}
		outSupp[candidate] = count;
		// the subspace count below could be done in the loop above, to have one less load of the candidate.
		
		unsigned int subSpaceCount = 0;
		for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
			unsigned int candidateBlock = candidates[j*numberOfCandidates + candidate];
			subSpaceCount += __popc(candidateBlock);
		}
		outScore[candidate] = count*pow(((float) 1/beta),subSpaceCount) ; // calculate score and store
		//printf("candidate %u, have score %f \n", candidate, outScore[candidate]);
		outToBeDeleted[candidate] = count < minSupp;
	}
}

/**
Thin wrapper for CountSupport kernel
*/
void countSupportWrapper(unsigned int dimGrid,
						 unsigned int dimBlock,
						 hipStream_t stream,
						 unsigned int* candidates,
						 unsigned int* itemSet,
						 unsigned int dim,
						 unsigned int numberOfItems,
						 unsigned int numberOfCandidates,
						 unsigned int minSupp,
						 float beta,
						 unsigned int* outSupp,
						 float* outScore,
						 bool* outToBeDeleted
						 ){
	countSupport<<<dimGrid, dimBlock, 0, stream>>>(candidates,
												   itemSet,
												   dim,
												   numberOfItems,
												   numberOfCandidates,
												   minSupp,
												   beta,
												   outSupp,
												   outScore,
												   outToBeDeleted);
};


/**
   ONLY For testing the kernel countSupport
*/
std::tuple<
	std::vector<unsigned int>,
	std::vector<float>,
	std::vector<bool>> countSupportTester(std::vector<std::vector<bool>> candidates, std::vector<std::vector<bool>> itemSet,
							 unsigned int minSupp, float beta){
	unsigned int numberOfCandidates = candidates.size();
	unsigned int numberOfItems = itemSet.size();
	unsigned int dim = itemSet.at(0).size();
	unsigned int numberOfBlocksPrElement = ceilf((float)dim/32);
	unsigned int bitsInLastBlock = dim%32;

	size_t sizeOfCandidates = numberOfCandidates*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfItemSet = numberOfItems*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfScores = numberOfCandidates*sizeof(float);
	size_t sizeOfSupport = numberOfCandidates*sizeof(unsigned int);
	size_t sizeOfToBeDeleted = numberOfCandidates*sizeof(bool);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)dim/1024);

	unsigned int* candidates_h;
	unsigned int* itemSet_h;
	unsigned int* outSupport_h;
	float* outScores_h;
	bool* outToBeDeleted_h;

	unsigned int* candidates_d;
	unsigned int* itemSet_d;
	unsigned int* outSupport_d;
	float* outScores_d;
	bool* outToBeDeleted_d;

	hipHostMalloc((void**) &candidates_h, sizeOfCandidates);
	hipHostMalloc((void**) &itemSet_h, sizeOfItemSet);
	hipHostMalloc((void**) &outSupport_h, sizeOfSupport);
	hipHostMalloc((void**) &outScores_h, sizeOfScores);
	hipHostMalloc((void**) &outToBeDeleted_h, sizeOfToBeDeleted);

	hipMalloc((void**) &candidates_d, sizeOfCandidates);
	hipMalloc((void**) &itemSet_d, sizeOfItemSet);
	hipMalloc((void**) &outSupport_d, sizeOfSupport);
	hipMalloc((void**) &outScores_d, sizeOfScores);
	hipMalloc((void**) &outToBeDeleted_d, sizeOfToBeDeleted);

	// fill candidates
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				candidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (candidates.at(i).at(j) << j);
		}
		candidates_h[i+blockNr*numberOfCandidates] = block;
	}

	// fill itemSet
	for(unsigned int i = 0; i < numberOfItems; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				itemSet_h[i+blockNr*numberOfItems] = block;
				block = 0;
				blockNr++;
			}
			block |= (itemSet.at(i).at(j) << j);
			
		}
		itemSet_h[i+blockNr*numberOfItems] = block;
	}

	checkCudaErrors(hipMemcpy(candidates_d, candidates_h, sizeOfCandidates, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(itemSet_d, itemSet_h, sizeOfItemSet, hipMemcpyHostToDevice));
	
	countSupport<<<dimGrid, dimBlock>>>(candidates_d, itemSet_d, dim, numberOfItems, numberOfCandidates, minSupp, beta, outSupport_d, outScores_d, outToBeDeleted_d); 
	checkCudaErrors(hipMemcpy(outSupport_h, outSupport_d, sizeOfSupport, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outScores_h, outScores_d, sizeOfScores, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outToBeDeleted_h, outToBeDeleted_d, sizeOfToBeDeleted, hipMemcpyDeviceToHost));
	auto support = std::vector<unsigned int>();
	auto score = std::vector<float>();
	auto toBeDeleted = std::vector<bool>();
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		support.push_back(outSupport_h[i]);
		score.push_back(outScores_h[i]);
		toBeDeleted.push_back(outToBeDeleted_h[i]);
	}
	
	std::tuple<
	std::vector<unsigned int>,
	std::vector<float>,
	std::vector<bool>
		> result;

	std::get<0>(result) = support;
	std::get<1>(result) = score;
	std::get<2>(result) = toBeDeleted;

	return result;
}



__global__ void mergeCandidates(unsigned int* candidates, unsigned int numberOfCandidates, unsigned int dim, unsigned int iterNr,
								unsigned int* output, bool* toBeDeleted){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int i = numberOfCandidates - 2- floorf(sqrtf(-8*k + 4*numberOfCandidates*(numberOfCandidates-1)-7)/ 2.0 - 0.5);
	unsigned int j = k + i + 1 - numberOfCandidates*(numberOfCandidates-1)/2 + (numberOfCandidates-i)*((numberOfCandidates-i)-1)/2;
	unsigned int numberOfNewCandidates = (numberOfCandidates*(numberOfCandidates+1))/2 - numberOfCandidates;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	
	assert(iterNr >= 2);
	if(k < numberOfNewCandidates){
		unsigned int interSectionCount=0;
		for(unsigned int a = 0; a < numberOfBlocks; a++){
			assert(a*numberOfNewCandidates+k < numberOfBlocks*numberOfNewCandidates);
			output[a*numberOfNewCandidates+k] = (candidates[a*numberOfCandidates+i] | candidates[a*numberOfCandidates+j]);
			interSectionCount += __popc(candidates[a*numberOfCandidates+i] & candidates[a*numberOfCandidates+j]);
		}
		toBeDeleted[k] = !((int)interSectionCount == (((int)iterNr)-2));
	}
}

/**
   Thin wrapper for mergeCandidates
*/
void mergeCandidatesWrapper(unsigned int dimGrid,
							unsigned int dimBlock,
							hipStream_t stream,
							unsigned int* candidates,
							unsigned int numberOfCandidates,
							unsigned int dim,
							unsigned int itrNr,
							unsigned int* output,
							bool* toBeDeleted
							){
	mergeCandidates<<<dimGrid, dimBlock, 0, stream>>>(candidates, numberOfCandidates, dim, itrNr, output, toBeDeleted);
};


std::pair<std::vector<unsigned int>,std::vector<bool>> mergeCandidatesTester(std::vector<std::vector<bool>> candidates, unsigned int itrNr){
	unsigned int numberOfCandidates = candidates.size();
	unsigned int dim = candidates.at(0).size();
	unsigned int numberOfNewCandidates = ((numberOfCandidates*(numberOfCandidates+1)) / 2) - numberOfCandidates;
	unsigned int numberOfBlocks = ceilf((float)dim/32);

	size_t sizeOfOutput = numberOfNewCandidates*numberOfBlocks*sizeof(unsigned int);
	size_t sizeOfCandidates = numberOfCandidates*numberOfBlocks*sizeof(unsigned int);
	size_t sizeOfToBeDeleted = numberOfNewCandidates*sizeof(bool);
	
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfNewCandidates/dimBlock);

	unsigned int* candidates_h;
	unsigned int* output_h;

	unsigned int* candidates_d;
	unsigned int* output_d;

	bool* toBeDeleted_h;
	bool* toBeDeleted_d;

	hipHostMalloc((void**) &candidates_h, sizeOfCandidates);
	hipHostMalloc((void**) &output_h, sizeOfOutput);
	hipHostMalloc((void**) &toBeDeleted_h, sizeOfToBeDeleted);

	hipMalloc((void**) &candidates_d, sizeOfCandidates);
	hipMalloc((void**) &output_d, sizeOfOutput);
	hipMalloc((void**) &toBeDeleted_d, sizeOfToBeDeleted);
	

	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				candidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (candidates.at(i).at(j) << j);
		}
		candidates_h[i+blockNr*numberOfCandidates] = block;
	}

	hipMemcpy(candidates_d, candidates_h, sizeOfCandidates, hipMemcpyHostToDevice);

	mergeCandidates<<<dimGrid, dimBlock>>>(candidates_d, numberOfCandidates, dim, itrNr, output_d, toBeDeleted_d);

	hipMemcpy(output_h, output_d, sizeOfOutput, hipMemcpyDeviceToHost);
	hipMemcpy(toBeDeleted_h, toBeDeleted_d, sizeOfToBeDeleted, hipMemcpyDeviceToHost);

	auto result = std::vector<unsigned int>();
	for(int i = 0; i < numberOfNewCandidates*numberOfBlocks; i++){
		
		result.push_back(output_h[i]);
	}

	auto result2 = std::vector<bool>();
	for(int i = 0; i < numberOfNewCandidates; i++){
		
		result2.push_back(toBeDeleted_h[i]);
	}
	return std::make_pair(result, result2);
			   
	
}



__global__ void findDublicatesNaive(unsigned int* candidates, unsigned int numberOfCandidates, unsigned int dim,
									bool* isAlreadyDeleted, bool* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	
	if(candidate < numberOfCandidates && !isAlreadyDeleted[candidate]){
		for(unsigned int i = candidate+1; i < numberOfCandidates; i++){
			bool equal = true;
			for(unsigned int j = 0; j < numberOfBlocks; j++){
				equal &= (candidates[candidate + numberOfCandidates*j] == candidates[i + numberOfCandidates*j]);
			}
			if(equal){
				output[i] = true;
			}
		}		
	}
}



__global__ void findDublicatesBreaking(unsigned int* candidates, unsigned int numberOfCandidates, unsigned int dim,
									bool* isAlreadyDeleted, bool* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(candidate < numberOfCandidates && !isAlreadyDeleted[candidate]){
		for(unsigned int i = candidate+1; i < numberOfCandidates; i++){
			bool equal = true;
			for(unsigned int j = 0; j < numberOfBlocks; j++){
				equal &= (candidates[candidate + numberOfCandidates*j] == candidates[i + numberOfCandidates*j]);
			}
			if(equal){
				output[i] = true;
				break;
			}
		}
	}
}

__global__ void findDublicatesMoreBreaking(unsigned int* candidates, unsigned int numberOfCandidates, unsigned int dim,
									bool* isAlreadyDeleted, bool* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(candidate < numberOfCandidates && !isAlreadyDeleted[candidate]){
		for(unsigned int i = candidate+1; i < numberOfCandidates; i++){
			bool equal = true;
			for(unsigned int j = 0; j < numberOfBlocks; j++){
				equal &= (candidates[candidate + numberOfCandidates*j] == candidates[i + numberOfCandidates*j]);
				if(!equal){
					break;
				}
			}
			if(equal){
				output[i] = true;
				break;
			}
		}
	}
}


/**
   Thin Wrapper for findDublicates
*/
void findDublicatesWrapper(unsigned int dimGrid,
						   unsigned int dimBlock,
						   hipStream_t stream,
						   unsigned int* candidates,
						   unsigned int numberOfCandidates,
						   unsigned int dim,
						   bool* alreadyDeleted,
						   bool* output,
						   dublicatesType version
						   ){
	if(version == Naive){
		findDublicatesNaive<<<dimGrid, dimBlock, 0, stream>>>(candidates, numberOfCandidates, dim, alreadyDeleted, output);			
	}else if(version == Breaking){
		findDublicatesBreaking<<<dimGrid, dimBlock, 0, stream>>>(candidates, numberOfCandidates, dim, alreadyDeleted, output);			
	}else if(version == MoreBreaking){
		findDublicatesMoreBreaking<<<dimGrid, dimBlock, 0, stream>>>(candidates, numberOfCandidates, dim, alreadyDeleted, output);			
	}else if(version == Hash){
		findDublicatesHashTableWrapper(dimGrid, dimBlock, stream, candidates, numberOfCandidates, dim, alreadyDeleted, output);		 
	}
};

/**
   ONLY FOR TESTING
*/
std::vector<bool> findDublicatesTester(std::vector<std::vector<bool>> candidates, dublicatesType version){
	unsigned int numberOfCandidates = candidates.size();
	unsigned int dim = candidates.at(0).size();
	unsigned int numberOfBlocks = ceilf((float)dim/32);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfCandidates/dimBlock);

	size_t sizeOfCandidates = numberOfCandidates*numberOfBlocks*sizeof(unsigned int);
	size_t sizeOfOutput = numberOfCandidates*sizeof(bool);

	unsigned int* candidates_h;
	bool* output_h;
	
	unsigned int* candidates_d;
	bool* output_d;

	hipHostMalloc((void**) &candidates_h, sizeOfCandidates);
	hipHostMalloc((void**) &output_h, sizeOfOutput);


	hipMalloc((void**) &candidates_d, sizeOfCandidates);
	hipMalloc((void**) &output_d, sizeOfOutput);



	bool* alreadyDeleted_d;
	hipMalloc((void**) &alreadyDeleted_d, numberOfCandidates*sizeof(bool));
	hipMemset(alreadyDeleted_d, 0, numberOfCandidates*sizeof(bool));

	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				candidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (candidates.at(i).at(j) << j);
		}
		candidates_h[i+blockNr*numberOfCandidates] = block;
	}



	hipMemcpy(candidates_d, candidates_h, sizeOfCandidates, hipMemcpyHostToDevice);
	hipMemset(output_d, false, sizeOfOutput);
	if(version == Naive){
		findDublicatesNaive<<<dimGrid, dimBlock>>>(candidates_d, numberOfCandidates, dim, alreadyDeleted_d, output_d);
	}else if(version == Breaking){
		findDublicatesBreaking<<<dimGrid, dimBlock>>>(candidates_d, numberOfCandidates, dim, alreadyDeleted_d, output_d);		
	}else if(version == MoreBreaking){
		findDublicatesMoreBreaking<<<dimGrid, dimBlock>>>(candidates_d, numberOfCandidates, dim, alreadyDeleted_d, output_d);		
	}else if(version == Hash){
		findDublicatesHashTableTester(dimGrid, dimBlock, candidates_d, numberOfCandidates, dim, alreadyDeleted_d, output_d);
	}


	hipMemcpy(output_h, output_d, sizeOfOutput, hipMemcpyDeviceToHost);

	auto result = std::vector<bool>();
	for(int i = 0; i < numberOfCandidates; i++){
		result.push_back(output_h[i]);
	}

	return result;
}


__global__ void extractMax(unsigned int* candidates, float* scores, unsigned int centroid, unsigned int numberOfCandidates,
					  unsigned int* bestIndex,
					  unsigned int dim, unsigned int* bestCandidate, float* bestScore, unsigned int* bestCentroid){
	unsigned int block = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(scores[0] > bestScore[0]){
		if(block < numberOfBlocks){
			bestCandidate[block] = candidates[block*numberOfCandidates+bestIndex[0]];
		}
		if(block == numberOfBlocks){
			bestCentroid[0] = centroid;
		}
		if(block == numberOfBlocks+1){
			bestScore[0] = scores[bestIndex[0]];
		}	
	}
}



void extractMaxWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
					   unsigned int* candidates, float* scores, unsigned int centroid, unsigned int numberOfCandidates,
					   unsigned int* bestIndex,
					   unsigned int dim, unsigned int* bestCandidate, float* bestScore, unsigned int* bestCentroid){
	extractMax<<<dimGrid, dimBlock, 0, stream>>>(candidates, scores, centroid, numberOfCandidates,
												 bestIndex,
												 dim, bestCandidate, bestScore, bestCentroid);
}


std::pair<std::vector<unsigned int>, float> extractMaxTester(std::vector<bool> oldCandidate,
															 unsigned int oldScore, unsigned int oldCentroid,
															 std::vector<std::vector<bool>> newCandidates,
															 std::vector<float> newScores, unsigned int newCentroid,
															 unsigned int index){
	unsigned int numberOfBlocks = ceilf((float)oldCandidate.size()/32);
	unsigned int dim = oldCandidate.size();
	unsigned int numberOfCandidates = newCandidates.size();

	unsigned int* oldCandidate_h;
	unsigned int* newCandidates_h;

	unsigned int* oldCentroid_h;
	unsigned int* newCentroid_h;

	float* oldScore_h;
	float* newScores_h;

	hipHostMalloc((void**) &oldCandidate_h, numberOfBlocks*sizeof(unsigned int));
	hipHostMalloc((void**) &newCandidates_h, numberOfCandidates*numberOfBlocks*sizeof(unsigned int));

	hipHostMalloc((void**) &oldCentroid_h, sizeof(unsigned int));
	hipHostMalloc((void**) &newCentroid_h, sizeof(unsigned int));

	hipHostMalloc((void**) &oldScore_h, sizeof(float));
	hipHostMalloc((void**) &newScores_h, numberOfCandidates*sizeof(float));


	unsigned int* oldCandidate_d;
	unsigned int* newCandidates_d;

	unsigned int* oldCentroid_d;
	unsigned int* newCentroid_d;

	float* oldScore_d;
	float* newScores_d;
	
	hipMalloc((void**) &oldCandidate_d, numberOfBlocks*sizeof(unsigned int));
	hipMalloc((void**) &newCandidates_d, numberOfCandidates*numberOfBlocks*sizeof(unsigned int));

	hipMalloc((void**) &oldCentroid_d, sizeof(unsigned int));
	hipMalloc((void**) &newCentroid_d, sizeof(unsigned int));

	hipMalloc((void**) &oldScore_d, sizeof(float));
	hipMalloc((void**) &newScores_d, numberOfCandidates*sizeof(float));

	for(unsigned int i = 0; i < newScores.size(); i++){
		newScores_h[i] = newScores.at(i);
	}
	
	unsigned int value = 0;
	unsigned int blockNr = 0;
	for(unsigned int i = 0; i < oldCandidate.size(); i++){
		if(i %32 == 0 && i !=0){
			oldCandidate_h[blockNr] = value;
			blockNr++;
			value = 0;
		}
		value |= (oldCandidate.at(i) << i);
	}
	oldCandidate_h[blockNr] = value;

	oldScore_h[0] = oldScore;
	oldCentroid_h[0] = oldCentroid;


	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				newCandidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (newCandidates.at(i).at(j) << j);
		}
		newCandidates_h[i+blockNr*numberOfCandidates] = block;
	}
	
	unsigned int* index_h;
	hipHostMalloc((void**) &index_h, sizeof(unsigned int));
	index_h[0] = index;
	
	unsigned int* index_d;
	hipMalloc((void**) &index_d, sizeof(unsigned int));
	
	hipMemcpy(index_d, index_h, sizeof(unsigned int), hipMemcpyDeviceToHost);

	newCentroid_h[0] = newCentroid;

	hipMemcpy(oldCandidate_d, oldCandidate_h, numberOfBlocks*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(oldScore_d, oldScore_h, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(oldCentroid_d, oldCentroid_h, sizeof(unsigned int), hipMemcpyHostToDevice);

	
	hipMemcpy(newCandidates_d, newCandidates_h, numberOfCandidates*numberOfBlocks*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(newScores_d, newScores_h, numberOfCandidates*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(newCentroid_d, newCentroid_h, sizeof(unsigned int), hipMemcpyHostToDevice);

	extractMax<<<ceilf((float)(dim+2)/1024), 1024>>>(newCandidates_d, newScores_d, newCentroid, newCandidates.size(), index_d, dim,
												 oldCandidate_d, oldScore_d, oldCentroid_d);

	hipMemcpy(oldCandidate_h, oldCandidate_d, numberOfBlocks*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(oldScore_h, oldScore_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(oldCentroid_h, oldCentroid_d, sizeof(float), hipMemcpyDeviceToHost);

	std::vector<unsigned int> bestCandidate;
	for(int i = 0; i < numberOfBlocks; i++){
		bestCandidate.push_back(oldCandidate_h[i]);
	}

	bestCandidate.push_back(oldCentroid_h[0]);

	std::pair<std::vector<unsigned int>, float> result;
	result = make_pair(bestCandidate, oldScore_h[0]);
	return result;

	
	}


__global__ void findPointsInCluster(unsigned int* candidate, float* data, float* centroid, unsigned int dim, unsigned int numberOfPoints, float width, bool* pointsContained){
	unsigned int point = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(point < numberOfPoints){
		bool isContained = true;
		for(unsigned int i = 0; i < numberOfBlocks; i++){
			unsigned int block = candidate[i];
			for(unsigned int j = 0; j < 32; j++){
				if(i*32+j < dim){
					bool isDimChosen = (block >> j) & 1;
					float cent = centroid[i*32+j];
					float poin = data[point*dim+i*32+j];
					bool r = (not(isDimChosen)) || ((abs(cent - poin)) < width);
					isContained &= r;
				}else{
					break;
				}
			}
		}
		pointsContained[point] = isContained;
	}
}


void findPointInClusterWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							   unsigned int* candidate, float* data, float* centroid, unsigned int dim,
							   unsigned int numberOfPoints, float width, bool* pointsContained){
	findPointsInCluster<<<dimGrid, dimBlock, 0, stream>>>(candidate, data, centroid, dim, numberOfPoints, width, pointsContained);
}



std::vector<bool> findPointsInClusterTester(std::vector<bool> candidate, std::vector<std::vector<float>*>* data, unsigned int centroid, float width){
	unsigned int numberOfPoints = data->size();
	unsigned int dim = data->at(0)->size();
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	
	size_t sizeOfCandidate = numberOfBlocks*sizeof(unsigned int);
	size_t sizeOfCentroid = dim*sizeof(float);
	size_t sizeOfData = numberOfPoints * dim * sizeof(float);
	size_t sizeOfPointsContained = numberOfPoints * sizeof(bool);
	
	unsigned int* candidate_h;
	float* centroid_h;
	float* data_h;
	bool* pointsContained_h;

	hipHostMalloc((void**) &candidate_h, sizeOfCandidate);
	hipHostMalloc((void**) &centroid_h, sizeOfCentroid);
	hipHostMalloc((void**) &data_h, sizeOfData);
	hipHostMalloc((void**) &pointsContained_h, sizeOfPointsContained);

	unsigned int* candidate_d;
	float* centroid_d;
	float* data_d;
	bool* pointsContained_d;

	hipMalloc((void**) &candidate_d, sizeOfCandidate);
	hipMalloc((void**) &centroid_d, sizeOfCentroid);
	hipMalloc((void**) &data_d, sizeOfData);	
	hipMalloc((void**) &pointsContained_d, sizeOfPointsContained);
	
	unsigned int value = 0;
	unsigned int blockNr = 0;
	for(unsigned int i = 0; i < candidate.size(); i++){
		if(i %32 == 0 && i !=0){
			candidate_h[blockNr] = value;
			blockNr++;
			value = 0;
		}
		value |= (candidate.at(i) << i);
	}
	candidate_h[blockNr] = value;

	for(unsigned int i = 0; i < numberOfPoints; i++){
		for(unsigned int j = 0; j < dim; j++){
			data_h[i*dim+j] = data->at(i)->at(j);
		}
	}

	for(int i = 0; i < dim; i++){
		centroid_h[i] = data->at(centroid)->at(i);
	}

	
	
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfPoints/dimBlock);

	hipMemcpy(candidate_d, candidate_h, sizeOfCandidate, hipMemcpyHostToDevice);
	hipMemcpy(data_d, data_h, sizeOfData, hipMemcpyHostToDevice);
	hipMemcpy(centroid_d, centroid_h, sizeOfCentroid, hipMemcpyHostToDevice);


	
	findPointsInCluster<<<dimGrid, dimBlock>>>(candidate_d, data_d, centroid_d, dim, numberOfPoints, width, pointsContained_d);

	hipMemcpy(pointsContained_h, pointsContained_d, sizeOfPointsContained, hipMemcpyDeviceToHost);
	
	auto result = std::vector<bool>();
	for(unsigned int i = 0; i < numberOfPoints; i++){
		result.push_back(pointsContained_h[i]);
	}
	return result;
}


__global__ void orKernel(unsigned int numberOfElements, bool* a, bool* b){
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i < numberOfElements){
		a[i] |= b[i];	
	}
}

void orKernelWrapper(unsigned int dimGrid, unsigned int dimBlock,  hipStream_t stream,
					  unsigned int numberOfElements, bool* a, bool* b){
	orKernel<<<dimGrid, dimBlock, 0, stream>>>(numberOfElements, a, b);
}



__global__ void disjointClusters(unsigned int* centroids, float* scores, unsigned int* subspaces, float* data, const unsigned int numberOfClusters, unsigned int dim, float width, unsigned int* output){
	extern __shared__ unsigned int out[];
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfComparisons = (numberOfClusters*(numberOfClusters+1))/2 - numberOfClusters;

	
	// setting the output
	if(k < numberOfClusters){
		output[k] = true;
	}
	
	
	// setting the shared memory
	for(unsigned int i = 0; i < ceilf((float)numberOfClusters/blockDim.x); i++){
		if(threadIdx.x+blockDim.x*i < numberOfClusters){
			out[threadIdx.x+blockDim.x*i] = 1;	
		}
	}
		
	if(k < numberOfComparisons){
		unsigned int i = numberOfClusters - 2- floorf(sqrtf(-8*k + 4*numberOfClusters*(numberOfClusters-1)-7)/ 2.0 - 0.5);
		unsigned int j = k + i + 1 - numberOfClusters*(numberOfClusters-1)/2 + (numberOfClusters-i)*((numberOfClusters-i)-1)/2;
		bool isDisjoint = true;
		unsigned int blockNr = 0;
		unsigned int currentBlock = 0;
		unsigned int centroidI = centroids[i];
		unsigned int centroidJ = centroids[j];
		unsigned int numberOfBlocks = ceilf((float)dim/32);

		for(unsigned int a = 0; a < dim; a++){
			if(a%32 == 0){
				blockNr = a/32;
				assert(i*numberOfBlocks+blockNr < numberOfBlocks*numberOfClusters);
				assert(j*numberOfBlocks+blockNr < numberOfBlocks*numberOfClusters);
				currentBlock = subspaces[i*numberOfBlocks+blockNr] & subspaces[j*numberOfBlocks+blockNr];
			}
			isDisjoint &= (!(currentBlock >> a%32) & 1) || ((abs(data[centroidI*dim+a] - data[centroidJ*dim+a]) >= 2*width));	
		}
		
		if(isDisjoint){
			//printf("k,i,j: %u, %u, %u are disjoint\n",k, i,j);
			atomicAnd(&out[i], 1);
			atomicAnd(&out[j], 1);
		}else if(scores[i] < scores[j]){
			//printf("k,i,j: %u,  %u, %u; score %f < %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], j,i);
			atomicAnd(&out[i], 0);
			atomicAnd(&out[j], 1);
		}else if(scores[i] == scores[j]){
			//printf("k,i,j: %u, %u, %u; score %f == %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], min(i,j),max(i,j));
			atomicAnd(&out[min(i,j)], 1);
			atomicAnd(&out[max(i,j)], 0);
		}else{
			//printf("k,i,j: %u, %u, %u; score %f > %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], i,j);
			atomicAnd(&out[i], 1);				
			atomicAnd(&out[j], 0);
		}
	}

	
	for(unsigned int i = 0; i < ceilf((float)numberOfClusters/blockDim.x); i++){
		if(threadIdx.x+blockDim.x*i < numberOfClusters){
			atomicAnd(&output[threadIdx.x+blockDim.x*i], out[threadIdx.x+blockDim.x*i]);	
		}
	}
}

void disjointClustersWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							 unsigned int* centroids, float* scores, unsigned int* subspaces,
							 float* data, unsigned int numberOfClusters, unsigned int dim,
							 float width, unsigned int* output){


	unsigned int smem = numberOfClusters*sizeof(unsigned int);
	disjointClusters<<<dimGrid, dimBlock, smem, stream>>>(centroids, scores, subspaces,
														  data, numberOfClusters, dim,
														  width, output);
}

std::vector<bool> disjointClustersTester(std::vector<std::vector<float>*>* data_v, std::vector<unsigned int> centroids_v, std::vector<unsigned int> subspaces_v, std::vector<float> scores_v){
	unsigned int width = 10;
	unsigned int dim = data_v->at(0)->size();
	unsigned int numberOfPoints = data_v->size();
	unsigned int smem = scores_v.size()*sizeof(unsigned int);
	
	float* data;
	float* scores;
	unsigned int* centroids;
	unsigned int* subspaces;
	unsigned int* output;

	unsigned int numberOfComparisons = (scores_v.size()*(scores_v.size()+1))/2-scores_v.size();

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfComparisons/dimBlock);
	assert(dimGrid <= 1);

	size_t sizeofData = numberOfPoints*dim*sizeof(float);
	size_t sizeofScores = scores_v.size()*sizeof(float);
	size_t sizeofCentroids = centroids_v.size()*sizeof(unsigned int);
	size_t sizeofSubspaces = subspaces_v.size()*sizeof(unsigned int);
	size_t sizeofOutput = centroids_v.size()*sizeof(unsigned int);


	hipMallocManaged((void**) &data, sizeofData);
	hipMallocManaged((void**) &scores, sizeofScores);
	hipMallocManaged((void**) &centroids, sizeofCentroids);
	hipMallocManaged((void**) &subspaces, sizeofSubspaces);
	hipMallocManaged((void**) &output, sizeofOutput);

	for(int i = 0; i < data_v->size(); i++){
		for(int j = 0; j < dim; j++){
			data[i*dim+j] = data_v->at(i)->at(j);	
		}
	}

	for(int i = 0; i < scores_v.size(); i++){
		scores[i] = scores_v.at(i);
	}

	for(int i = 0; i < subspaces_v.size(); i++){
		subspaces[i] = subspaces_v.at(i);
	}

	for(int i = 0; i < centroids_v.size(); i++){
		centroids[i] = centroids_v.at(i);
	}

	disjointClusters<<<dimGrid, dimBlock, smem>>>(centroids, scores, subspaces,
														  data, scores_v.size(), dim,
														  width, output);

	hipDeviceSynchronize();
	std::vector<bool> output_v;

	for(int i = 0; i < centroids_v.size(); i++){
		output_v.push_back(output[i]);
	}

	hipFree(data);
	hipFree(centroids);
	hipFree(scores);
	hipFree(subspaces);
	hipFree(output);
	return output_v;
	
}


__global__ void unsignedIntToBoolArray(unsigned int* input, unsigned int numberOfElements, bool* output){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	if(k < numberOfElements){
		output[k] = input[k];
	}
}


void unsignedIntToBoolArrayWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							unsigned int* input, unsigned int numberOfElements, bool* output){
	unsignedIntToBoolArray<<<dimGrid, dimBlock, 0, stream>>>(input, numberOfElements, output);
}


__global__ void copyCentroid(unsigned int* centroids, float* data, unsigned int dim, unsigned int numberOfCentroids, float* centroidsOut){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	if(k < dim*numberOfCentroids){
		unsigned int centroidToUse = k/dim;
		unsigned int dimInCentroid = k%dim;
		unsigned int centroidIndex = centroids[centroidToUse];
		centroidsOut[centroidToUse*dim+dimInCentroid] = data[centroidIndex*dim+dimInCentroid];
	}
}

void copyCentroidWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
						 unsigned int* centroids, float* data, unsigned int dim,
						 unsigned int numberOfCentroids, float* centroidsOut){
	copyCentroid<<<dimGrid, dimBlock, 0, stream>>>(centroids, data, dim, numberOfCentroids, centroidsOut);
}