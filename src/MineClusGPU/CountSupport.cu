#include "hip/hip_runtime.h"
#include "CountSupport.h"

__device__ __forceinline__ bool  countSupportBlock(unsigned int* items, unsigned int* candidates, size_t itemsIndex, size_t candidateIndex){
	unsigned int itemBlock = items[itemsIndex];
	unsigned int candidateBlock = candidates[candidateIndex];
	unsigned int candidateBlockCount = __popc(candidateBlock);
	unsigned int unionCount = __popc(itemBlock&candidateBlock);
	
	return candidateBlockCount == unionCount;
		}

__global__ void countSupport(unsigned int* candidates, unsigned int* itemSet,
							 unsigned int dim, unsigned int numberOfItems,
							 unsigned int numberOfCandidates,
							 unsigned int minSupp, float beta,
							 unsigned int* outSupp, float* outScore,
							 bool* outToBeDeleted){
	
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;

	unsigned int numberOfBlocksPrItem = ceilf((float)dim/32);
	
	if(candidate < numberOfCandidates){
		unsigned int count = 0;
		for(unsigned int i = 0; i < numberOfItems; i++){
			bool isSubset = true;
			for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
				isSubset &=  countSupportBlock(itemSet, candidates, j*numberOfItems+i, j*numberOfCandidates + candidate);
			}
			count += isSubset;
		}
		outSupp[candidate] = count;
		
		// the subspace count below could be done in the loop above, to have one less load of the candidate.
		unsigned int subSpaceCount = 0;
		for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
			unsigned int candidateBlock = candidates[j*numberOfCandidates + candidate];
			subSpaceCount += __popc(candidateBlock);
		}
		outScore[candidate] = count*pow(((float) 1/beta),subSpaceCount) ; // calculate score and store
		//printf("candidate %u, have score %f \n", candidate, outScore[candidate]);
		outToBeDeleted[candidate] = count < minSupp;
	}
}


__device__ __forceinline__ unsigned int writeBit(unsigned int value, unsigned int bitnr){
	if(bitnr < 32){
		return value | (1 << (bitnr));
	}else{
		return 0;
	}
}


__global__ void countSupportSharedMemory(unsigned int* candidates, unsigned int* transactions,
										 unsigned int dim, unsigned int numberOfTransactions,
										 unsigned int numberOfCandidates,
										 unsigned int minSupp, float beta,
										 unsigned int* outSupp, float* outScore,
										 bool* outToBeDeleted){

	extern __shared__ unsigned int transactions_s[];

	unsigned int candidates_chunk[6]; // in register allocation

	unsigned int isSubset_r[2];

	unsigned int subspaceCount = 0;
	unsigned int supportCount = 0;
	//unsigned int nPoints = 62;
	//unsigned int chunkSize = 6; // time 32 dimensions



	unsigned int transactionsProcessed = 0;
	unsigned int dimsProcessed = 0; 
	
	unsigned int currentTransactionChunk = 0;
	unsigned int currentTransactionChunkSize = 0;

	unsigned int currentDimChunk = 0;
	unsigned int currentDimChunkSize = 0;

	unsigned int smemUtilised = 0;
	while(transactionsProcessed < numberOfTransactions){
		isSubset_r[0] = 0xffffffff;
		isSubset_r[1] = 0xffffffff;

		// compute the number of transactions each chunk is responsible for
		if(transactionsProcessed+62*32 < numberOfTransactions){
			currentTransactionChunkSize = 62;
			transactionsProcessed += currentTransactionChunkSize*32;
			smemUtilised = 62*32;
		}else{
			unsigned int diff = transactionsProcessed - numberOfTransactions;
			currentTransactionChunkSize = diff/32;
			if(diff%32 < threadIdx.x/32){
				currentTransactionChunkSize++;
				transactionsProcessed += (diff+1)*(threadIdx.x/32);
				
			}else{
				transactionsProcessed += (diff%32) + diff*(threadIdx.x/32);
			}
			smemUtilised = diff;
		}
		
		dimsProcessed = 0;
		while(dimsProcessed < dim){
			if(dimsProcessed+6 < dim){
				currentDimChunkSize = 6;
				dimsProcessed += currentDimChunkSize;
			}else{
				currentDimChunkSize = dimsProcessed+6 - dim;
				dimsProcessed += currentDimChunkSize;
			}
			smemUtilised *= currentDimChunkSize;

			// Load candidate into registers			
			{
				unsigned int id = threadIdx.x%32 + blockIdx.x*32; /*the index of the candidate*/
				if(id < numberOfTransactions){
					for(unsigned int i = 0; i < currentDimChunkSize; i++){ // loads a candidate into registers
						candidates_chunk[i] = candidates[id + (i+dimsProcessed)*numberOfTransactions];
						subspaceCount += __popc(candidates_chunk[i]); // done in all warps, but only used in last warp, only one extra instruction, and one register read.
						candidateIndex += numberOfCandidates;
					}				
				}
			}

			__syncthreads();

			
			// Load transactions into shared memory
			for(unsigned int i = 0; i < ceilf((float)currentTransactionChunkSize/32); i++){
				unsigned int currentWarpPos = threadIdx.x%32;
				unsigned int id = transactionsProcessed + i*32+currentWarpPos;
				for(unsigned int j = 0; j < currentDimChunkSize; j++){
					if(id+j*numberofTransactions < transactionsProcessed+currentTransactionChunkSize){
						transactions_s[i*32+currentWarpPos+j*currentTransactionChunkSize] = transactions[id+j*numberofTransactions];
					}
				}
			}

			
			// for(unsigned int i = 0; i < ceilf((float)nPoints*32*chunkSize/blockDim.x); i++){
			// 	unsigned int j = i*blockDim.x+threadIdx.x; // the if if it were a normal for loop
			// 	unsigned int transactionNumber = (j % (nPoints*32)) +  currentChunk*nPoints*32;
			// 	unsigned int currentDim = (j / (nPoints*32)) + currentDimChunk*chunkSize;
			// 	unsigned int id = currentDim*numberOfTransactions + transactionNumber;
			// 	if((id < numberOfTransactions*ceilf((float)dim/32))  && (j < nPoints*chunkSize*32)){
			// 		transactions_s[j] = transactions[id];
			// 		printf("%u thread: %u, transaction %u value %u id %u transaction number %u, currentDim %u\n",
			// 			   i, threadIdx.x, j, transactions_s[j], id, transactionNumber, currentDim);
			// 	}
			// }

			__syncthreads();
			// Compute the count
			unsigned int temp = 0;
			unsigned int currentBank = threadIdx.x/32;
			for(unsigned int j = 0; j < currentDimChunkSize; j++){
				bool isSubset = true;
				for(unsigned int i = 0; i < currentTransactionChunkSize; i++){
					unsigned int transactionIndex = i+currentBank + j*32*currentTransactionChunkSize;
					if(transactionIndex < smemUtilised){
						isSubset = isSubset && countSupportBlock(transactions_s, candidates_chunk, transactionIndex ,i);
					}
					
				}
				if(isSubset){
					temp = writeBit(temp, i%32);
				}
				if(i != 0 && i%32 == 0 || i == currentTransactionChunkSize-1){
					isSubset_r[i/32] &= temp;
					temp = 0;
				}
			}
			
			// unsigned int temp = 0;
			// for(unsigned int j = 0; j < nPoints; j++){
			// 	bool isSubset = true;
			// 	for(unsigned int i = 0; i < chunkSize; i++){
			// 		unsigned int currentBank = threadIdx.x/32;
			// 		unsigned int transactionIndex = 32*j + currentBank + nPoints*i;
			// 		if(transactionIndex < numberOfTransactions*ceilf((float)dim/32)){
			// 			isSubset = isSubset && countSupportBlock(transactions_s, candidates_chunk, transactionIndex ,i);

			// 		}
			// 	}
			// 	if(isSubset){
			// 		temp = writeBit(temp, j%32);
			// 	}
			// 	if(j != 0 && j%32 == 0 || j == ceilf((float)nPoints/32)-1){
			// 		isSubset_r[j/32] &= temp;
			// 		temp = 0;
			// 	}
			// }
		}
		smemUtilised /= currentDimChunkSize;
		supportCount += __popc(isSubset_r[0]);
		if(currentTransactionChunkSize) >= 32){
			supportCount += __popc(isSubset_r[1]);	
		}
	}
	__syncthreads();
	// Utilise shared memory for the reduction sum, sine we are done using it for the candidates we can simply reuse it
	// Loop unroled, all in a warp should always go in the same condition
	transactions_s[threadIdx.x] = supportCount;
	__syncthreads();
	if((threadIdx.x/32)%2 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32];
	}
	__syncthreads();
	if((threadIdx.x/32)%4 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*2];
	}
	__syncthreads();
	if((threadIdx.x/32)%8 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*4];
	}
	__syncthreads();
	if((threadIdx.x/32)%16 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*8];
	}
	__syncthreads();
	if((threadIdx.x/32)%32 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*16];
	}
	__syncthreads();
	if(threadIdx.x/32 == 0){
		unsigned int i = blockIdx.x*32+threadIdx.x;
		outSupp[i] = transactions_s[threadIdx.x];
		outScore[i] = transactions_s[threadIdx.x]*pow(((float) 1/beta),subspaceCount) ; // calculate score and store
		outToBeDeleted[i] = transactions_s[threadIdx.x] < minSupp;
	}
}

/**
   Thin wrapper for CountSupport kernel
*/
void countSupportWrapper(unsigned int dimGrid,
						 unsigned int dimBlock,
						 hipStream_t stream,
						 unsigned int* candidates,
						 unsigned int* itemSet,
						 unsigned int dim,
						 unsigned int numberOfItems,
						 unsigned int numberOfCandidates,
						 unsigned int minSupp,
						 float beta,
						 unsigned int* outSupp,
						 float* outScore,
						 bool* outToBeDeleted
						 ){
	countSupport<<<dimGrid, dimBlock, 0, stream>>>(candidates,
												   itemSet,
												   dim,
												   numberOfItems,
												   numberOfCandidates,
												   minSupp,
												   beta,
												   outSupp,
												   outScore,
												   outToBeDeleted);
};


/**
   ONLY For testing the kernel countSupport
*/
std::tuple<
	std::vector<unsigned int>,
	std::vector<float>,
	std::vector<bool>> countSupportTester(std::vector<std::vector<bool>> candidates, std::vector<std::vector<bool>> itemSet,
										  unsigned int minSupp, float beta,
										 countSupportType version){
	unsigned int numberOfCandidates = candidates.size();
	unsigned int numberOfItems = itemSet.size();
	unsigned int dim = itemSet.at(0).size();
	unsigned int numberOfBlocksPrElement = ceilf((float)dim/32);
	unsigned int bitsInLastBlock = dim%32;

	size_t sizeOfCandidates = numberOfCandidates*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfItemSet = numberOfItems*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfScores = numberOfCandidates*sizeof(float);
	size_t sizeOfSupport = numberOfCandidates*sizeof(unsigned int);
	size_t sizeOfToBeDeleted = numberOfCandidates*sizeof(bool);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)dim/1024);

	unsigned int* candidates_h;
	unsigned int* itemSet_h;
	unsigned int* outSupport_h;
	float* outScores_h;
	bool* outToBeDeleted_h;

	unsigned int* candidates_d;
	unsigned int* itemSet_d;
	unsigned int* outSupport_d;
	float* outScores_d;
	bool* outToBeDeleted_d;

	hipHostMalloc((void**) &candidates_h, sizeOfCandidates);
	hipHostMalloc((void**) &itemSet_h, sizeOfItemSet);
	hipHostMalloc((void**) &outSupport_h, sizeOfSupport);
	hipHostMalloc((void**) &outScores_h, sizeOfScores);
	hipHostMalloc((void**) &outToBeDeleted_h, sizeOfToBeDeleted);

	hipMalloc((void**) &candidates_d, sizeOfCandidates);
	hipMalloc((void**) &itemSet_d, sizeOfItemSet);
	hipMalloc((void**) &outSupport_d, sizeOfSupport);
	hipMalloc((void**) &outScores_d, sizeOfScores);
	hipMalloc((void**) &outToBeDeleted_d, sizeOfToBeDeleted);

	// fill candidates
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				candidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (candidates.at(i).at(j) << j);
		}
		candidates_h[i+blockNr*numberOfCandidates] = block;
	}

	// fill itemSet
	for(unsigned int i = 0; i < numberOfItems; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				itemSet_h[i+blockNr*numberOfItems] = block;
				block = 0;
				blockNr++;
			}
			block |= (itemSet.at(i).at(j) << j);
			
		}
		itemSet_h[i+blockNr*numberOfItems] = block;
	}

	checkCudaErrors(hipMemcpy(candidates_d, candidates_h, sizeOfCandidates, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(itemSet_d, itemSet_h, sizeOfItemSet, hipMemcpyHostToDevice));

	if(version == NaiveCount){
		countSupport<<<dimGrid, dimBlock>>>(candidates_d, itemSet_d, dim, numberOfItems, numberOfCandidates, minSupp, beta, outSupport_d, outScores_d, outToBeDeleted_d);		
	}else if(version == SmemCount){
		dimGrid = ceilf((float)numberOfCandidates/(32));
		unsigned int smemSize = ceilf((float)dim/32) > 6 ? 6 : ceilf((float)dim/32);
		smemSize = smemSize*sizeof(unsigned int)*32*62;
		smemSize = smemSize < dimBlock*sizeof(unsigned int) ? dimBlock*sizeof(unsigned int) : smemSize;
		std::cout << "smsm: " << smemSize << " dimGrid "<< dimGrid << " dim: " << dim << std::endl;
		
		
		countSupportSharedMemory<<<dimGrid, dimBlock,smemSize>>>(candidates_d, itemSet_d, dim, numberOfItems, numberOfCandidates, minSupp, beta, outSupport_d, outScores_d, outToBeDeleted_d);				
	}



	
	checkCudaErrors(hipMemcpy(outSupport_h, outSupport_d, sizeOfSupport, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outScores_h, outScores_d, sizeOfScores, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outToBeDeleted_h, outToBeDeleted_d, sizeOfToBeDeleted, hipMemcpyDeviceToHost));
	auto support = std::vector<unsigned int>();
	auto score = std::vector<float>();
	auto toBeDeleted = std::vector<bool>();
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		support.push_back(outSupport_h[i]);
		score.push_back(outScores_h[i]);
		toBeDeleted.push_back(outToBeDeleted_h[i]);
	}
	
	std::tuple<
		std::vector<unsigned int>,
		std::vector<float>,
		std::vector<bool>
		> result;

	std::get<0>(result) = support;
	std::get<1>(result) = score;
	std::get<2>(result) = toBeDeleted;

	return result;
}
