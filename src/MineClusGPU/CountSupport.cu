#include "hip/hip_runtime.h"
#include "CountSupport.h"

__device__ __forceinline__ bool  countSupportBlock(unsigned int* items, unsigned int* candidates, size_t itemsIndex, size_t candidateIndex){
	unsigned int itemBlock = items[itemsIndex];
	unsigned int candidateBlock = candidates[candidateIndex];
	unsigned int candidateBlockCount = __popc(candidateBlock);
	unsigned int unionCount = __popc(itemBlock&candidateBlock);
	
	return candidateBlockCount == unionCount;
		}

__global__ void countSupport(unsigned int* candidates, unsigned int* itemSet,
							 unsigned int dim, unsigned int numberOfItems,
							 unsigned int numberOfCandidates,
							 unsigned int minSupp, float beta,
							 unsigned int* outSupp, float* outScore,
							 bool* outToBeDeleted){
	
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;

	unsigned int numberOfBlocksPrItem = ceilf((float)dim/32);
	
	if(candidate < numberOfCandidates){

		unsigned int count = 0;
		for(unsigned int i = 0; i < numberOfItems; i++){
			bool isSubset = true;
			for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
				isSubset &=  countSupportBlock(itemSet, candidates, j*numberOfItems+i, j*numberOfCandidates + candidate);
			}
			count += isSubset;
		}
		outSupp[candidate] = count;
		
		// the subspace count below could be done in the loop above, to have one less load of the candidate.
		unsigned int subSpaceCount = 0;
		for(unsigned int j = 0; j < numberOfBlocksPrItem; j++){
			unsigned int candidateBlock = candidates[j*numberOfCandidates + candidate];
			// if(threadIdx.x == 0 && blockIdx.x == 0) printf("Naive thread %u loads from %u value %u\n",threadIdx.x,j*numberOfCandidates + candidate,candidateBlock);
			subSpaceCount += __popc(candidateBlock);
		}
		// if(threadIdx.x == 0 && blockIdx.x == 0)printf("naive subspace count %u \n", subSpaceCount);
		outScore[candidate] = logf(count)+logf((float) 1/beta)*subSpaceCount; // calculate score and store
		//printf("candidate %u, have score %f \n", candidate, outScore[candidate]);
		outToBeDeleted[candidate] = count < minSupp;
	}
}


__device__ __forceinline__ unsigned int writeBit(unsigned int value, unsigned int bitnr){
	if(bitnr < 32){
		return value | (1 << (bitnr));
	}else{
		return 0;
	}
}


__global__ void countSupportSharedMemory(unsigned int* candidates, unsigned int* transactions,
										 unsigned int dim, unsigned int numberOfTransactions,
										 unsigned int numberOfCandidates,
										 unsigned int minSupp, float beta,
										 unsigned int* outSupp, float* outScore,
										 bool* outToBeDeleted){

	extern __shared__ unsigned int transactions_s[];

	unsigned int candidates_chunk[6]; // in register allocation

	unsigned int isSubset_r[2];

	unsigned int subspaceCount = 0;
	unsigned int supportCount = 0;
	//unsigned int nPoints = 62;
	//unsigned int chunkSize = 6; // time 32 dimensions

	bool firstIter = true;

	unsigned int transactionsProcessed = 0;
	unsigned int dimsProcessed = 0; 
	
	unsigned int currentTransactionChunkSize = 0;
	unsigned int restTransactionChunk = 0;

	unsigned int currentDimChunkSize = 0;
	
	while(transactionsProcessed < numberOfTransactions){
		isSubset_r[0] = 0xffffffff;
		isSubset_r[1] = 0xffffffff;
		// if(threadIdx.x == 0 && blockIdx.x == 0) printf("start iteration\n");
		// compute the number of transactions each chunk is responsible for
		if(transactionsProcessed+62*32 < numberOfTransactions){
			// if(threadIdx.x == 0 && blockIdx.x == 0) printf("got here3\n");
			currentTransactionChunkSize = 62;
			transactionsProcessed += currentTransactionChunkSize*(threadIdx.x/32);
			restTransactionChunk = currentTransactionChunkSize*32 - currentTransactionChunkSize*(threadIdx.x/32);
		}else{
			unsigned int diff = numberOfTransactions - transactionsProcessed;
			currentTransactionChunkSize = diff/32;
			if(diff%32 > threadIdx.x/32){
				currentTransactionChunkSize++;
				// if(threadIdx.x == 0 && blockIdx.x == 0) printf("got here1\n");
				transactionsProcessed += currentTransactionChunkSize*(threadIdx.x/32);
				restTransactionChunk = diff - currentTransactionChunkSize*(threadIdx.x/32);
			}else{
				// if(threadIdx.x == 0 && blockIdx.x == 0) printf("got here2\n");
				transactionsProcessed += (diff%32) + currentTransactionChunkSize*(threadIdx.x/32);
				restTransactionChunk = diff - ((diff%32) + currentTransactionChunkSize*(threadIdx.x/32));
			}
		}

		
		dimsProcessed = 0;
		if(dim > 6*32 || firstIter) subspaceCount = 0;
		unsigned int dimChunks = (unsigned int)ceilf((float)dim/32);
		while(dimsProcessed < dimChunks){
			if(dimsProcessed+6 < dimChunks){
				currentDimChunkSize = 6;
			}else{
				currentDimChunkSize = dimChunks - dimsProcessed;
			}
			// if(threadIdx.x == 0 && blockIdx.x == 0) printf("chunk size %u transactionsProcessed %u restTransactionChunk %u\n",currentDimChunkSize,transactionsProcessed,restTransactionChunk);

			// Load candidate into registers			
			if(dim > 6*32 || firstIter){ // To avoid reloading the transactions if they do not change
				unsigned int id = threadIdx.x%32 + blockIdx.x*32; /*the index of the candidate*/
				if(id < numberOfCandidates){
					for(unsigned int i = 0; i < currentDimChunkSize; i++){ // loads a candidate into registers
						candidates_chunk[i] = candidates[id + (i+dimsProcessed)*numberOfCandidates];
						
						subspaceCount += __popc(candidates_chunk[i]); // done in all warps, but only used in last warp, only one extra instruction, and one register read.
						// if(threadIdx.x == 0 && blockIdx.x == 0) printf("thread %u loads from %u value %u subspaceCount %u\n",threadIdx.x,id + (i+dimsProcessed)*numberOfCandidates,candidates_chunk[i],subspaceCount);
						
					}				
				}
			}
			// if(threadIdx.x == 0 && blockIdx.x == 0) printf("SMEM thread %u subspaceCount %u\n",threadIdx.x,subspaceCount);
			firstIter = false;

			__syncthreads();
			
			
			// Load transactions into shared memory
			for(unsigned int i = 0; i < ceilf((float)currentTransactionChunkSize/32); i++){
				unsigned int currentWarpPos = threadIdx.x%32;
				unsigned int id = (currentWarpPos + i*32)*32 +(threadIdx.x/32); 
				for(unsigned int j = 0; j < currentDimChunkSize; j++){
					if(transactionsProcessed+currentWarpPos+i*32 < transactionsProcessed+currentTransactionChunkSize){
						// if(blockIdx.x == 0) printf("Thread %u reads from %u numberOfTransactions %u j %u transactionsProcessed %u i %u restTransactionChunk %u currentTransactionChunkSize %u\n", threadIdx.x,
												   // transactionsProcessed+currentWarpPos+i*32 + (dimsProcessed+j)*numberOfTransactions,
												   // numberOfTransactions, j, transactionsProcessed, i, restTransactionChunk,
												   // currentTransactionChunkSize);
						
						transactions_s[id+ j*currentTransactionChunkSize*32] = transactions[transactionsProcessed+currentWarpPos+i*32 + (dimsProcessed+j)*numberOfTransactions];
					}
				}
			}



			__syncthreads();
			// Compute the count
			unsigned int temp = 0;
			unsigned int currentBank = threadIdx.x/32;
			for(unsigned int i = 0; i < currentTransactionChunkSize; i++){
				bool isSubset = true;
				for(unsigned int j = 0; j < currentDimChunkSize; j++){
					unsigned int transactionIndex = i*32+currentBank + j*32*currentTransactionChunkSize;

						isSubset = isSubset && countSupportBlock(transactions_s, candidates_chunk, transactionIndex ,j);
					
					

				}
				if(isSubset){
					//if(threadIdx.x == 0 && blockIdx.x == 0) printf("thread %u transaction %u isSubset\n", threadIdx.x, i);
					temp = writeBit(temp, i%32);
				}
				
				if(i != 0 && i%32 == 31 || i == currentTransactionChunkSize-1){
					isSubset_r[i/32] &= temp;
					// if(threadIdx.x == 0 && blockIdx.x == 150 && transactionsProcessed == 0) printf("tmp %u block %u value %u\n", temp,i/32, isSubset_r[i/32]);
					temp = 0;
				}

			}			

			dimsProcessed += currentDimChunkSize;
		}
		if(currentTransactionChunkSize > 0){
			supportCount += __popc(isSubset_r[0]);
		}
		if(currentTransactionChunkSize > 32){
			supportCount += __popc(isSubset_r[1]);	
		}
		transactionsProcessed += restTransactionChunk;
	}
	__syncthreads();
	// Utilise shared memory for the reduction sum, sine we are done using it for the candidates we can simply reuse it
	// Loop unroled, all in a warp should always go in the same condition
	transactions_s[threadIdx.x] = supportCount;
	__syncthreads();
	if((threadIdx.x/32)%2 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32];
	}
	__syncthreads();
	if((threadIdx.x/32)%4 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*2];
	}
	__syncthreads();
	if((threadIdx.x/32)%8 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*4];
	}
	__syncthreads();
	if((threadIdx.x/32)%16 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*8];
	}
	__syncthreads();
	if((threadIdx.x/32)%32 == 0){
		transactions_s[threadIdx.x] += transactions_s[threadIdx.x+32*16];
	}
	__syncthreads();
	if(threadIdx.x/32 == 0){
		unsigned int i = blockIdx.x*32+threadIdx.x;
		if(i < numberOfCandidates){
			// if(i == 0) printf("thid: %u i %u value %u result %u subspace %u\n",threadIdx.x,i, transactions_s[threadIdx.x],  transactions_s[threadIdx.x] < minSupp, subspaceCount);
			outSupp[i] = transactions_s[threadIdx.x];
			// if(threadIdx.x == 0 && blockIdx.x == 0)printf("Smem subspace count %u support %u\n", subspaceCount,transactions_s[threadIdx.x]);
			outScore[i] = logf(transactions_s[threadIdx.x])+logf((float) 1/beta)*subspaceCount; // calculate score and store
			// printf("i %u \n",i);
			outToBeDeleted[i] = transactions_s[threadIdx.x] < minSupp;	
		}
	}
}


/**
   Thin wrapper for CountSupport kernel
*/
void countSupportWrapper(unsigned int dimGrid,
						 unsigned int dimBlock,
						 hipStream_t stream,
						 unsigned int* candidates,
						 unsigned int* itemSet,
						 unsigned int dim,
						 unsigned int numberOfItems,
						 unsigned int numberOfCandidates,
						 unsigned int minSupp,
						 float beta,
						 unsigned int* outSupp,
						 float* outScore,
						 bool* outToBeDeleted,
						 countSupportType version
						 ){
	if(version == NaiveCount){
		countSupport<<<dimGrid, dimBlock, 0, stream>>>(candidates,
													   itemSet,
													   dim,
													   numberOfItems,
													   numberOfCandidates,
													   minSupp,
													   beta,
													   outSupp,
													   outScore,
													   outToBeDeleted);
	}else if(version == SmemCount){
		dimGrid = ceilf((float)numberOfCandidates/(32));
		unsigned int smemSize = ceilf((float)dim/32) > 6 ? 6 : ceilf((float)dim/32);
		smemSize = smemSize*sizeof(unsigned int)*32*62;
		smemSize = smemSize < dimBlock*sizeof(unsigned int) ? dimBlock*sizeof(unsigned int) : smemSize;
		//std::cout << "smsm: " << smemSize << " dimGrid "<< dimGrid << " dim: " << dim << std::endl;
		
		
		countSupportSharedMemory<<<dimGrid, dimBlock,smemSize,stream>>>(candidates,
																		itemSet,
																		dim,
																		numberOfItems,
																		numberOfCandidates,
																		minSupp,
																		beta,
																		outSupp,
																		outScore,
																		outToBeDeleted);
		
	}
};


/**
   ONLY For testing the kernel countSupport
*/
std::tuple<
	std::vector<unsigned int>,
	std::vector<float>,
	std::vector<bool>> countSupportTester(std::vector<std::vector<bool>> candidates, std::vector<std::vector<bool>> itemSet,
										  unsigned int minSupp, float beta,
										 countSupportType version){
	unsigned int numberOfCandidates = candidates.size();
	unsigned int numberOfItems = itemSet.size();
	unsigned int dim = itemSet.at(0).size();
	unsigned int numberOfBlocksPrElement = ceilf((float)dim/32);
	unsigned int bitsInLastBlock = dim%32;

	size_t sizeOfCandidates = numberOfCandidates*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfItemSet = numberOfItems*numberOfBlocksPrElement*sizeof(unsigned int);
	size_t sizeOfScores = numberOfCandidates*sizeof(float);
	size_t sizeOfSupport = numberOfCandidates*sizeof(unsigned int);
	size_t sizeOfToBeDeleted = numberOfCandidates*sizeof(bool);

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfCandidates/1024);

	unsigned int* candidates_h;
	unsigned int* itemSet_h;
	unsigned int* outSupport_h;
	float* outScores_h;
	bool* outToBeDeleted_h;

	unsigned int* candidates_d;
	unsigned int* itemSet_d;
	unsigned int* outSupport_d;
	float* outScores_d;
	bool* outToBeDeleted_d;

	hipHostMalloc((void**) &candidates_h, sizeOfCandidates);
	hipHostMalloc((void**) &itemSet_h, sizeOfItemSet);
	hipHostMalloc((void**) &outSupport_h, sizeOfSupport);
	hipHostMalloc((void**) &outScores_h, sizeOfScores);
	hipHostMalloc((void**) &outToBeDeleted_h, sizeOfToBeDeleted);

	hipMalloc((void**) &candidates_d, sizeOfCandidates);
	hipMalloc((void**) &itemSet_d, sizeOfItemSet);
	hipMalloc((void**) &outSupport_d, sizeOfSupport);
	hipMalloc((void**) &outScores_d, sizeOfScores);
	hipMalloc((void**) &outToBeDeleted_d, sizeOfToBeDeleted);

	// fill candidates
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				candidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (candidates.at(i).at(j) << j);
		}
		candidates_h[i+blockNr*numberOfCandidates] = block;
	}
	

	// fill itemSet
	for(unsigned int i = 0; i < numberOfItems; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				itemSet_h[i+blockNr*numberOfItems] = block;
				block = 0;
				blockNr++;
			}
			block |= (itemSet.at(i).at(j) << j);
			
		}
		itemSet_h[i+blockNr*numberOfItems] = block;
	}

	checkCudaErrors(hipMemcpy(candidates_d, candidates_h, sizeOfCandidates, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(itemSet_d, itemSet_h, sizeOfItemSet, hipMemcpyHostToDevice));

	if(version == NaiveCount){
		countSupport<<<dimGrid, dimBlock>>>(candidates_d, itemSet_d, dim, numberOfItems, numberOfCandidates, minSupp, beta, outSupport_d, outScores_d, outToBeDeleted_d);		
	}else if(version == SmemCount){
		dimGrid = ceilf((float)numberOfCandidates/(32));
		unsigned int smemSize = ceilf((float)dim/32) > 6 ? 6 : ceilf((float)dim/32);
		smemSize = smemSize*sizeof(unsigned int)*32*62;
		smemSize = smemSize < dimBlock*sizeof(unsigned int) ? dimBlock*sizeof(unsigned int) : smemSize;
		//std::cout << "smsm: " << smemSize << " dimGrid "<< dimGrid << " dim: " << dim << std::endl;
		
		
		countSupportSharedMemory<<<dimGrid, dimBlock,smemSize>>>(candidates_d, itemSet_d, dim, numberOfItems, numberOfCandidates, minSupp, beta, outSupport_d, outScores_d, outToBeDeleted_d);				
	}



	
	checkCudaErrors(hipMemcpy(outSupport_h, outSupport_d, sizeOfSupport, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outScores_h, outScores_d, sizeOfScores, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(outToBeDeleted_h, outToBeDeleted_d, sizeOfToBeDeleted, hipMemcpyDeviceToHost));
	auto support = std::vector<unsigned int>();
	auto score = std::vector<float>();
	auto toBeDeleted = std::vector<bool>();
	for(unsigned int i = 0; i < numberOfCandidates; i++){
		support.push_back(outSupport_h[i]);
		score.push_back(outScores_h[i]);
		toBeDeleted.push_back(outToBeDeleted_h[i]);
	}
	
	std::tuple<
		std::vector<unsigned int>,
		std::vector<float>,
		std::vector<bool>
		> result;

	std::get<0>(result) = support;
	std::get<1>(result) = score;
	std::get<2>(result) = toBeDeleted;

	return result;
}
