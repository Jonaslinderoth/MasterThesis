#include "hip/hip_runtime.h"
#include "MineClusKernels.h"
#include <assert.h>
#include <iostream>
#include <tuple>
#include "../randomCudaScripts/Utils.h"


/**
   Creates the initial candidtes given the dimensions. 
*/
__global__ void createInitialCandidates(unsigned int dim, unsigned int* output){
	unsigned int candidate = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocksPrPoint = ceilf((float)dim/32);
	unsigned int myBlock = candidate/32;
	
	if(candidate < dim){
	
		// make sure all are 0;
		for(int i = 0; i < numberOfBlocksPrPoint; i++){
			assert(candidate+dim * i < dim*(ceilf((float)dim/32)));
			output[candidate+dim * i] = 0;
		}
		// set the correct candidate
		unsigned int output_block = (1 << (candidate%32));
		//printf("candidat %u have value %u, writing output at position %u\n", candidate, output_block, candidate+dim*myBlock);
		output[candidate+dim*myBlock] = output_block;
	}
}


/**
   Thin wrapper for CreateInitialCandidates
*/
void createInitialCandidatesWrapper(unsigned int dimGrid,
									unsigned int dimBlock,
									hipStream_t stream,
									unsigned int dim,
									unsigned int* output
									){
	createInitialCandidates<<<dimGrid, dimBlock, 0, stream>>>(dim, output);
}

/**
   This function is only for testing
*/
std::vector<unsigned int> createInitialCandidatesTester(unsigned int dim){
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)dim/dimBlock);

	size_t sizeof_output = dim*ceilf((float)dim/32)*sizeof(unsigned int);

	unsigned int* output_h = (unsigned int*) malloc(sizeof_output);
	unsigned int* output_d;

	hipMalloc((void**) &output_d, sizeof_output);

	createInitialCandidates<<<dimGrid, dimBlock>>>(dim, output_d);

	hipMemcpy(output_h, output_d, sizeof_output, hipMemcpyDeviceToHost);
	
	std::vector<unsigned int> res;
	for(int i = 0; i < ceilf((float)dim/32)*dim;i++){
		res.push_back(output_h[i]);
	}
	return res;
}





__global__ void extractMax(unsigned int* candidates, float* scores, unsigned int centroid, unsigned int numberOfCandidates,
					  unsigned int* bestIndex,
					  unsigned int dim, unsigned int* bestCandidate, float* bestScore, unsigned int* bestCentroid){
	unsigned int block = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(scores[0] > bestScore[0]){
		if(block < numberOfBlocks){
			bestCandidate[block] = candidates[block*numberOfCandidates+bestIndex[0]];
		}
		if(block == numberOfBlocks){
			bestCentroid[0] = centroid;
		}
		if(block == numberOfBlocks+1){
			bestScore[0] = scores[bestIndex[0]];
		}	
	}
}



void extractMaxWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
					   unsigned int* candidates, float* scores, unsigned int centroid, unsigned int numberOfCandidates,
					   unsigned int* bestIndex,
					   unsigned int dim, unsigned int* bestCandidate, float* bestScore, unsigned int* bestCentroid){
	extractMax<<<dimGrid, dimBlock, 0, stream>>>(candidates, scores, centroid, numberOfCandidates,
												 bestIndex,
												 dim, bestCandidate, bestScore, bestCentroid);
}


std::pair<std::vector<unsigned int>, float> extractMaxTester(std::vector<bool> oldCandidate,
															 unsigned int oldScore, unsigned int oldCentroid,
															 std::vector<std::vector<bool>> newCandidates,
															 std::vector<float> newScores, unsigned int newCentroid,
															 unsigned int index){
	unsigned int numberOfBlocks = ceilf((float)oldCandidate.size()/32);
	unsigned int dim = oldCandidate.size();
	unsigned int numberOfCandidates = newCandidates.size();

	unsigned int* oldCandidate_h;
	unsigned int* newCandidates_h;

	unsigned int* oldCentroid_h;
	unsigned int* newCentroid_h;

	float* oldScore_h;
	float* newScores_h;

	hipHostMalloc((void**) &oldCandidate_h, numberOfBlocks*sizeof(unsigned int));
	hipHostMalloc((void**) &newCandidates_h, numberOfCandidates*numberOfBlocks*sizeof(unsigned int));

	hipHostMalloc((void**) &oldCentroid_h, sizeof(unsigned int));
	hipHostMalloc((void**) &newCentroid_h, sizeof(unsigned int));

	hipHostMalloc((void**) &oldScore_h, sizeof(float));
	hipHostMalloc((void**) &newScores_h, numberOfCandidates*sizeof(float));


	unsigned int* oldCandidate_d;
	unsigned int* newCandidates_d;

	unsigned int* oldCentroid_d;
	unsigned int* newCentroid_d;

	float* oldScore_d;
	float* newScores_d;
	
	hipMalloc((void**) &oldCandidate_d, numberOfBlocks*sizeof(unsigned int));
	hipMalloc((void**) &newCandidates_d, numberOfCandidates*numberOfBlocks*sizeof(unsigned int));

	hipMalloc((void**) &oldCentroid_d, sizeof(unsigned int));
	hipMalloc((void**) &newCentroid_d, sizeof(unsigned int));

	hipMalloc((void**) &oldScore_d, sizeof(float));
	hipMalloc((void**) &newScores_d, numberOfCandidates*sizeof(float));

	for(unsigned int i = 0; i < newScores.size(); i++){
		newScores_h[i] = newScores.at(i);
	}
	
	unsigned int value = 0;
	unsigned int blockNr = 0;
	for(unsigned int i = 0; i < oldCandidate.size(); i++){
		if(i %32 == 0 && i !=0){
			oldCandidate_h[blockNr] = value;
			blockNr++;
			value = 0;
		}
		value |= (oldCandidate.at(i) << i);
	}
	oldCandidate_h[blockNr] = value;

	oldScore_h[0] = oldScore;
	oldCentroid_h[0] = oldCentroid;


	for(unsigned int i = 0; i < numberOfCandidates; i++){
		unsigned int block = 0;
		unsigned int blockNr = 0;
		for(int j = 0; j < dim; j++){
			if (j % 32 == 0 && j != 0){
				newCandidates_h[i+blockNr*numberOfCandidates] = block;
				block = 0;
				blockNr++;
			}
			block |= (newCandidates.at(i).at(j) << j);
		}
		newCandidates_h[i+blockNr*numberOfCandidates] = block;
	}
	
	unsigned int* index_h;
	hipHostMalloc((void**) &index_h, sizeof(unsigned int));
	index_h[0] = index;
	
	unsigned int* index_d;
	hipMalloc((void**) &index_d, sizeof(unsigned int));
	
	hipMemcpy(index_d, index_h, sizeof(unsigned int), hipMemcpyDeviceToHost);

	newCentroid_h[0] = newCentroid;

	hipMemcpy(oldCandidate_d, oldCandidate_h, numberOfBlocks*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(oldScore_d, oldScore_h, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(oldCentroid_d, oldCentroid_h, sizeof(unsigned int), hipMemcpyHostToDevice);

	
	hipMemcpy(newCandidates_d, newCandidates_h, numberOfCandidates*numberOfBlocks*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(newScores_d, newScores_h, numberOfCandidates*sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(newCentroid_d, newCentroid_h, sizeof(unsigned int), hipMemcpyHostToDevice);

	extractMax<<<ceilf((float)(dim+2)/1024), 1024>>>(newCandidates_d, newScores_d, newCentroid, newCandidates.size(), index_d, dim,
												 oldCandidate_d, oldScore_d, oldCentroid_d);

	hipMemcpy(oldCandidate_h, oldCandidate_d, numberOfBlocks*sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(oldScore_h, oldScore_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(oldCentroid_h, oldCentroid_d, sizeof(float), hipMemcpyDeviceToHost);

	std::vector<unsigned int> bestCandidate;
	for(int i = 0; i < numberOfBlocks; i++){
		bestCandidate.push_back(oldCandidate_h[i]);
	}

	bestCandidate.push_back(oldCentroid_h[0]);

	std::pair<std::vector<unsigned int>, float> result;
	result = make_pair(bestCandidate, oldScore_h[0]);
	return result;

	
	}


__global__ void findPointsInCluster(unsigned int* candidate, float* data, float* centroid, unsigned int dim, unsigned int numberOfPoints, float width, bool* pointsContained){
	unsigned int point = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	if(point < numberOfPoints){
		bool isContained = true;
		for(unsigned int i = 0; i < numberOfBlocks; i++){
			unsigned int block = candidate[i];
			for(unsigned int j = 0; j < 32; j++){
				if(i*32+j < dim){
					bool isDimChosen = (block >> j) & 1;
					float cent = centroid[i*32+j];
					float poin = data[point*dim+i*32+j];
					bool r = (not(isDimChosen)) || ((abs(cent - poin)) < width);
					isContained &= r;
				}else{
					break;
				}
			}
		}
		pointsContained[point] = isContained;
	}
}


void findPointInClusterWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							   unsigned int* candidate, float* data, float* centroid, unsigned int dim,
							   unsigned int numberOfPoints, float width, bool* pointsContained){
	findPointsInCluster<<<dimGrid, dimBlock, 0, stream>>>(candidate, data, centroid, dim, numberOfPoints, width, pointsContained);
}



std::vector<bool> findPointsInClusterTester(std::vector<bool> candidate, std::vector<std::vector<float>*>* data, unsigned int centroid, float width){
	unsigned int numberOfPoints = data->size();
	unsigned int dim = data->at(0)->size();
	unsigned int numberOfBlocks = ceilf((float)dim/32);
	
	size_t sizeOfCandidate = numberOfBlocks*sizeof(unsigned int);
	size_t sizeOfCentroid = dim*sizeof(float);
	size_t sizeOfData = numberOfPoints * dim * sizeof(float);
	size_t sizeOfPointsContained = numberOfPoints * sizeof(bool);
	
	unsigned int* candidate_h;
	float* centroid_h;
	float* data_h;
	bool* pointsContained_h;

	hipHostMalloc((void**) &candidate_h, sizeOfCandidate);
	hipHostMalloc((void**) &centroid_h, sizeOfCentroid);
	hipHostMalloc((void**) &data_h, sizeOfData);
	hipHostMalloc((void**) &pointsContained_h, sizeOfPointsContained);

	unsigned int* candidate_d;
	float* centroid_d;
	float* data_d;
	bool* pointsContained_d;

	hipMalloc((void**) &candidate_d, sizeOfCandidate);
	hipMalloc((void**) &centroid_d, sizeOfCentroid);
	hipMalloc((void**) &data_d, sizeOfData);	
	hipMalloc((void**) &pointsContained_d, sizeOfPointsContained);
	
	unsigned int value = 0;
	unsigned int blockNr = 0;
	for(unsigned int i = 0; i < candidate.size(); i++){
		if(i %32 == 0 && i !=0){
			candidate_h[blockNr] = value;
			blockNr++;
			value = 0;
		}
		value |= (candidate.at(i) << i);
	}
	candidate_h[blockNr] = value;

	for(unsigned int i = 0; i < numberOfPoints; i++){
		for(unsigned int j = 0; j < dim; j++){
			data_h[i*dim+j] = data->at(i)->at(j);
		}
	}

	for(int i = 0; i < dim; i++){
		centroid_h[i] = data->at(centroid)->at(i);
	}

	
	
	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfPoints/dimBlock);

	hipMemcpy(candidate_d, candidate_h, sizeOfCandidate, hipMemcpyHostToDevice);
	hipMemcpy(data_d, data_h, sizeOfData, hipMemcpyHostToDevice);
	hipMemcpy(centroid_d, centroid_h, sizeOfCentroid, hipMemcpyHostToDevice);


	
	findPointsInCluster<<<dimGrid, dimBlock>>>(candidate_d, data_d, centroid_d, dim, numberOfPoints, width, pointsContained_d);

	hipMemcpy(pointsContained_h, pointsContained_d, sizeOfPointsContained, hipMemcpyDeviceToHost);
	
	auto result = std::vector<bool>();
	for(unsigned int i = 0; i < numberOfPoints; i++){
		result.push_back(pointsContained_h[i]);
	}
	return result;
}


__global__ void orKernel(unsigned int numberOfElements, bool* a, bool* b){
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i < numberOfElements){
		a[i] |= b[i];	
	}
}

void orKernelWrapper(unsigned int dimGrid, unsigned int dimBlock,  hipStream_t stream,
					  unsigned int numberOfElements, bool* a, bool* b){
	orKernel<<<dimGrid, dimBlock, 0, stream>>>(numberOfElements, a, b);
}



__global__ void disjointClusters(unsigned int* centroids, float* scores, unsigned int* subspaces, float* data, const unsigned int numberOfClusters, unsigned int dim, float width, unsigned int* output){
	extern __shared__ unsigned int out[];
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberOfComparisons = (numberOfClusters*(numberOfClusters+1))/2 - numberOfClusters;

	
	// setting the output
	if(k < numberOfClusters){
		output[k] = true;
	}
	
	// setting the shared memory
	for(unsigned int i = 0; i < ceilf((float)numberOfClusters/blockDim.x); i++){
		if(threadIdx.x+blockDim.x*i < numberOfClusters){
			out[threadIdx.x+blockDim.x*i] = 1;	
		}
	}
	
	__syncthreads();
	
	if(k < numberOfComparisons){
		unsigned int i = numberOfClusters - 2- floorf(sqrtf(-8*k + 4*numberOfClusters*(numberOfClusters-1)-7)/ 2.0 - 0.5);
		unsigned int j = k + i + 1 - numberOfClusters*(numberOfClusters-1)/2 + (numberOfClusters-i)*((numberOfClusters-i)-1)/2;
		bool isDisjoint = false;
		unsigned int blockNr = 0;
		unsigned int currentBlock = 0;
		unsigned int centroidI = centroids[i];
		unsigned int centroidJ = centroids[j];
		unsigned int numberOfBlocks = ceilf((float)dim/32);

				// if the score i 0, There can be no points in the cluster, thereby it can be deleted
		if(scores[i] == 0 || scores[j] == 0){
			atomicAnd(&out[i], scores[i] != 0);
			atomicAnd(&out[j], scores[j] != 0);
		}else{
			for(unsigned int a = 0; a < dim; a++){
				if(a%32 == 0){
					blockNr = a/32;
					assert(i*numberOfBlocks+blockNr < numberOfBlocks*numberOfClusters);
					assert(j*numberOfBlocks+blockNr < numberOfBlocks*numberOfClusters);
					currentBlock = subspaces[i*numberOfBlocks+blockNr] & subspaces[j*numberOfBlocks+blockNr];
				}
				float tempI = data[centroidI*dim+a];
				float tempJ = data[centroidJ*dim+a];
				isDisjoint |= ((currentBlock >> a%32) & 1) && ((abs(tempI - tempJ) >= 2*width));	
			}


			if(isDisjoint){
				//printf("k,i,j: %u, %u, %u are disjoint\n",k, i,j);
				atomicAnd(&out[i], 1);
				atomicAnd(&out[j], 1);
			}else if(scores[i] < scores[j]){
				//printf("k,i,j: %u,  %u, %u; score %f < %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], j,i);
				atomicAnd(&out[i], 0);
				atomicAnd(&out[j], 1);
			}else if(scores[i] == scores[j]){
				//printf("k,i,j: %u, %u, %u; score %f == %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], min(i,j),max(i,j));
				atomicAnd(&out[min(i,j)], 1);
				atomicAnd(&out[max(i,j)], 0);
			}else{
				//printf("k,i,j: %u, %u, %u; score %f > %f, keeping %u, deleting %u\n",k, i,j,scores[i], scores[j], i,j);
				atomicAnd(&out[i], 1);				
				atomicAnd(&out[j], 0);
			}
		}
	}

	__syncthreads();	
	for(unsigned int i = 0; i < ceilf((float)numberOfClusters/blockDim.x); i++){
		if(threadIdx.x+blockDim.x*i < numberOfClusters){
			atomicAnd(&output[threadIdx.x+blockDim.x*i], out[threadIdx.x+blockDim.x*i]);	
		}
	}
}

void disjointClustersWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							 unsigned int* centroids, float* scores, unsigned int* subspaces,
							 float* data, unsigned int numberOfClusters, unsigned int dim,
							 float width, unsigned int* output){


	unsigned int smem = numberOfClusters*sizeof(unsigned int);
	disjointClusters<<<dimGrid, dimBlock, smem, stream>>>(centroids, scores, subspaces,
														  data, numberOfClusters, dim,
														  width, output);
}

std::vector<bool> disjointClustersTester(std::vector<std::vector<float>*>* data_v, std::vector<unsigned int> centroids_v, std::vector<unsigned int> subspaces_v, std::vector<float> scores_v){
	unsigned int width = 10;
	unsigned int dim = data_v->at(0)->size();
	unsigned int numberOfPoints = data_v->size();
	unsigned int smem = scores_v.size()*sizeof(unsigned int);
	
	float* data;
	float* scores;
	unsigned int* centroids;
	unsigned int* subspaces;
	unsigned int* output;

	unsigned int numberOfComparisons = (scores_v.size()*(scores_v.size()+1))/2-scores_v.size();

	unsigned int dimBlock = 1024;
	unsigned int dimGrid = ceilf((float)numberOfComparisons/dimBlock);
	assert(dimGrid <= 1);

	size_t sizeofData = numberOfPoints*dim*sizeof(float);
	size_t sizeofScores = scores_v.size()*sizeof(float);
	size_t sizeofCentroids = centroids_v.size()*sizeof(unsigned int);
	size_t sizeofSubspaces = subspaces_v.size()*sizeof(unsigned int);
	size_t sizeofOutput = centroids_v.size()*sizeof(unsigned int);


	hipMallocManaged((void**) &data, sizeofData);
	hipMallocManaged((void**) &scores, sizeofScores);
	hipMallocManaged((void**) &centroids, sizeofCentroids);
	hipMallocManaged((void**) &subspaces, sizeofSubspaces);
	hipMallocManaged((void**) &output, sizeofOutput);

	for(int i = 0; i < data_v->size(); i++){
		for(int j = 0; j < dim; j++){
			data[i*dim+j] = data_v->at(i)->at(j);	
		}
	}

	for(int i = 0; i < scores_v.size(); i++){
		scores[i] = scores_v.at(i);
	}

	for(int i = 0; i < subspaces_v.size(); i++){
		subspaces[i] = subspaces_v.at(i);
	}

	for(int i = 0; i < centroids_v.size(); i++){
		centroids[i] = centroids_v.at(i);
	}

	disjointClusters<<<dimGrid, dimBlock, smem>>>(centroids, scores, subspaces,
														  data, scores_v.size(), dim,
														  width, output);

	hipDeviceSynchronize();
	std::vector<bool> output_v;

	for(int i = 0; i < centroids_v.size(); i++){
		output_v.push_back(output[i]);
	}

	hipFree(data);
	hipFree(centroids);
	hipFree(scores);
	hipFree(subspaces);
	hipFree(output);
	return output_v;
	
}


__global__ void unsignedIntToBoolArray(unsigned int* input, unsigned int numberOfElements, bool* output){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	if(k < numberOfElements){
		output[k] = input[k];
	}
}


void unsignedIntToBoolArrayWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							unsigned int* input, unsigned int numberOfElements, bool* output){
	unsignedIntToBoolArray<<<dimGrid, dimBlock, 0, stream>>>(input, numberOfElements, output);
}


__global__ void copyCentroid(unsigned int* centroids, float* data, unsigned int dim, unsigned int numberOfCentroids, float* centroidsOut){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	if(k < dim*numberOfCentroids){
		unsigned int centroidToUse = k/dim;
		unsigned int dimInCentroid = k%dim;
		unsigned int centroidIndex = centroids[centroidToUse];
		centroidsOut[centroidToUse*dim+dimInCentroid] = data[centroidIndex*dim+dimInCentroid];
	}
}

void copyCentroidWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
						 unsigned int* centroids, float* data, unsigned int dim,
						 unsigned int numberOfCentroids, float* centroidsOut){
	copyCentroid<<<dimGrid, dimBlock, 0, stream>>>(centroids, data, dim, numberOfCentroids, centroidsOut);
}


__global__ void indexToBoolVector(unsigned int* index, unsigned int numberOfElements, bool* output){
	unsigned int k = blockIdx.x*blockDim.x+threadIdx.x;
	if(k < numberOfElements){
		output[k] = (index[0] == k);
	}
}

void indexToBoolVectorWrapper(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
							  unsigned int* index, unsigned int numberOfElements, bool* output
							  ){
	indexToBoolVector<<<dimGrid, dimBlock, 0, stream>>>(index, numberOfElements, output);
	
};