#include "hip/hip_runtime.h"
#include "HyperCube.h"
#include <iostream>
#include <vector>
#include <math.h>

__global__ void findDimmensionsDevice(float* Xs_d, float* ps_d, bool* res_d,
									  int point_dim, int no_of_samples, int no_in_sample, int no_of_ps, float width){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	int pNo = entry/no_of_samples;
	int sampleNo = entry%no_of_samples;
	
	if(entry < no_of_samples*no_of_ps){
		for(int i = 0; i < point_dim; i++){
			bool d = true;
			float p_tmp = ps_d[pNo*point_dim+i];
			for(int j = 0; j < no_in_sample; j++){
				d &= abs(p_tmp-Xs_d[sampleNo*no_in_sample*point_dim+j*point_dim+i]) < width;
			}
			res_d[entry*point_dim+i] = d;
		}
	}
}

__global__ void pointsContainedDevice(float* data, float* centroids, bool* dims, bool* output,
									  float width, int point_dim, int no_data, int no_dims){
	// one kernel for each hypercube
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	if(entry < no_dims){
		// for each data point 
		for(int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				d &= (not (dims[entry*point_dim+i])) || (abs(centroids[entry*point_dim+i] - data[j*point_dim+i]) < width);
			}
			output[entry*no_data+j] = d;
		}
	}
}

__global__ void score(float* Cluster_size, float* Dim_count, float* score_output, int len, float beta){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	if(entry < len){
		score_output[entry] = Cluster_size[entry]*powf(1.0/beta, Dim_count[entry]);	
	}

}

__global__ void argMaxDevice(float* scores, int* scores_index, float* output ,int* output_index, int input_size){
	extern __shared__ int array[];
	int* argData = (int*)array;
	float* scoreData = (float*) &argData[blockDim.x];

	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	argData[tid] = scores_index[i];
	scoreData[tid] = scores[i];
	
	__syncthreads();

	if(i < input_size){
		
		for(unsigned int s=blockDim.x; s > 0; s/=2) {

			if(tid < s){
				if(scoreData[tid] < scoreData[tid+s]){
					scoreData[tid] = scoreData[tid+s];
					argData[tid] = argData[tid+s];
				}
			}
			__syncthreads();
		}
	
		if(tid == 0){
			output_index[blockIdx.x] = argData[0];
			output[blockIdx.x] = scoreData[0];
		}; 
		
	}
	
}







std::pair<std::vector<std::vector<float>*>*, std::vector<bool>*> findCluster(std::vector<std::vector<float>*>* data, float alpha, float beta, float width){
	float d = data->at(0)->size();
	float r = log2(2*d)/log2(1/(2*beta));
	float m = pow((2/alpha),2) * log(4);

	unsigned int number_of_ps = 2.0/alpha;
	unsigned int number_of_samples = number_of_ps*m;
	unsigned int sample_size = r;
	unsigned int number_of_points = data->size();
	unsigned int point_dim = d;
	unsigned int floats_in_data_array = point_dim*number_of_points;
	unsigned int floats_in_ps_array = point_dim*number_of_ps; 
	
};






std::vector<std::vector<bool>*>* pointsContained(std::vector<std::vector<bool>*>* dims,
												 std::vector<std::vector<float>*>* data,
												 std::vector<std::vector<float>*>* centroids, float width){

	// Calculaating sizes
	int point_dim = centroids->at(0)->size();
	int no_of_points = data->size();
	int no_of_dims = dims->size();
	int no_of_centroids = centroids->size();

	int floats_in_data = point_dim * no_of_points;
	int bools_in_dims = no_of_dims * point_dim;
	int floats_in_centorids = no_of_centroids * point_dim;
	int bools_in_output = no_of_points * no_of_dims;
	
	int size_of_data = floats_in_data*sizeof(float);
	int size_of_dims = bools_in_dims*sizeof(bool);
	int size_of_centroids = floats_in_centorids*sizeof(float);
	int size_of_output = bools_in_output*sizeof(bool);

	// allocating on the host
	float* data_h = (float*) malloc(size_of_data);
	bool* dims_h = (bool*) malloc(size_of_dims);
	float* centroids_h = (float*) malloc(size_of_centroids);
	bool* output_h = (bool*) malloc(size_of_output);

	// filling data array
	for(int i= 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	// filling dims array
	for(int i= 0; i < no_of_dims; i++){
		for(int j = 0; j < point_dim; j++){
			dims_h[i*point_dim+j] = dims->at(i)->at(j);
		}
	}

	// filling centroid array
	for(int i= 0; i < no_of_centroids; i++){
		for(int j = 0; j < point_dim; j++){
			centroids_h[i*point_dim+j] = centroids->at(i)->at(j);
		}
	}

	// allocating on device
	float* data_d;
	bool* dims_d;
	float* centroids_d;
	bool* output_d;
	
	hipMalloc((void **) &data_d, size_of_data);
	hipMalloc((void **) &dims_d, size_of_dims);
	hipMalloc((void **) &centroids_d, size_of_centroids);
	hipMalloc((void **) &output_d, size_of_output);

	//Copy from host to device
				
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
	hipMemcpy(dims_d, dims_h, size_of_dims, hipMemcpyHostToDevice);
	hipMemcpy(centroids_d, centroids_h, size_of_centroids, hipMemcpyHostToDevice);


	// Call kernel
	pointsContainedDevice<<<ceil((no_of_dims)/256.0), 256>>>(data_d, centroids_d, dims_d, output_d,
						  width, point_dim, no_of_points, no_of_dims);

	// copy from device
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);

	// construnct output
	auto output =  new std::vector<std::vector<bool>*>;
   	
	
	for(int i = 0; i < no_of_dims; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < no_of_points; j++){
			a->push_back(output_h[i*no_of_points+j]);
		}
		output->push_back(a);
	}


	hipFree(data_d);
	hipFree(dims_d);
	hipFree(centroids_d);
	hipFree(output_d);

	
	return output;
};


std::vector<std::vector<bool>*>* findDimmensions(std::vector<std::vector<float>*>* ps,
												 std::vector<std::vector<std::vector<float>*>*> Xs, float width){
	
	int point_dim = Xs.at(0)->at(0)->size();
	int no_in_sample = Xs.at(0)->size();
	int no_of_samples = Xs.size();

	
	int sizeOfXs = no_of_samples*no_in_sample*point_dim*sizeof(float);
	float* xs_h = (float*) malloc(sizeOfXs);
	for(int i = 0; i < no_of_samples; i++){
		for(int j = 0; j < no_in_sample; j++){
			for(int k = 0; k < point_dim; k++){
				xs_h[i*no_in_sample*point_dim+j*point_dim+k] = Xs.at(i)->at(j)->at(k);
			}
		}
	}

	int no_of_ps = ps->size();
	int sizeOfps = point_dim*no_of_ps*sizeof(float);
	float* ps_h = (float*) malloc(sizeOfps);
	for(int i = 0; i < no_of_ps; i++){
		for(int j = 0; j < point_dim; j++){
			ps_h[i*point_dim+j] = ps->at(i)->at(j);
		}
	}
	/*
	std::cout << "xs: " << std::endl;
	for(int i = 0; i < no_of_samples*no_in_sample*point_dim; i++){
		std::cout << xs_h[i] << ", ";
		if((i+1)% point_dim == 0){
			std::cout << std::endl;
		}
	}
	std::cout << std::endl;
	

	std::cout << "ps: " << std::endl;

	for(int i = 0; i < no_of_ps*point_dim; i++){
		std::cout << ps_h[i] << ", ";
	}
	std::cout << std::endl;
	*/
	int outputDim = no_of_ps*no_of_samples*point_dim;		
	int outputSize = outputDim*sizeof(bool);
	bool* result_h = (bool*) malloc(outputSize);


	float* Xs_d;
	float* ps_d;
	bool* result_d;
	
	hipMalloc((void **) &Xs_d, sizeOfXs);
	hipMalloc((void **) &ps_d, sizeOfps);
	hipMalloc((void **) &result_d, outputSize);

	hipMemcpy( Xs_d, xs_h, sizeOfXs, hipMemcpyHostToDevice);
    hipMemcpy( ps_d, ps_h, sizeOfps, hipMemcpyHostToDevice);

	findDimmensionsDevice<<<ceil((no_of_ps*no_of_samples)/256.0), 256>>>(Xs_d, ps_d, result_d, point_dim, no_of_samples, no_in_sample, no_of_ps, width);

   
	hipMemcpy(result_h, result_d, outputSize, hipMemcpyDeviceToHost);

	auto output =  new std::vector<std::vector<bool>*>;
	
	for(int i = 0; i < no_of_ps*no_of_samples; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < point_dim; j++){
			a->push_back(result_h[i*point_dim+j]);
		}
		output->push_back(a);
	}

	hipFree(Xs_d);
	hipFree(ps_d);
	hipFree(result_d);
	
	return output;
}



int argMax(std::vector<float>* scores){
	//Calculate size of shared Memory, block and thread dim
	//fetch device info
	// TODO: hardcoded device 0
	int smemSize, maxBlock;
	hipDeviceGetAttribute(&smemSize, 
    hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
	hipDeviceGetAttribute(&maxBlock, 
						   hipDeviceAttributeMaxThreadsPerBlock, 0); // TODO::: not working::....

	maxBlock = 64;//512; // TODO::: findout why values larger than 64 wont work for all samples larger than 64...
	//std::cout << smemSize << std::endl;

	// caluclate the maximum thread size based on shared mem requirements and maximum threads
	int dimBlock = smemSize/(sizeof(int)+sizeof(float));
	if(dimBlock > maxBlock) dimBlock = maxBlock;
	int dimGrid = ceil((float)scores->size()/(float)dimBlock);
	int sharedMemSize = ((1+dimBlock)*sizeof(int) + (1+dimBlock)*sizeof(float));


	int size_of_score = scores->size()*sizeof(float);
	int size_of_score_index = scores->size()*sizeof(int);
	int size_of_output = sizeof(float)*dimGrid;
	int size_of_output_index = sizeof(int)*dimGrid;
	
	float* scores_h = (float*) malloc(size_of_score);
	int* scores_index_h = (int*) malloc(size_of_score_index);
	float* output_h = (float*) malloc(size_of_output);
	int* output_index_h = (int*) malloc(size_of_output_index);

	//std::cout << "creating data..." << scores->size() << std::endl;
	for(int i = 0; i < scores->size(); i++){
		scores_h[i] = scores->at(i);
		scores_index_h[i] = i;
	}

	//std::cout << "data created" << std::endl;
	float* scores_d;
	int* scores_index_d;
	float* output_d;
	int* output_index_d;

	hipMalloc((void **) &scores_d, size_of_score);
	hipMalloc((void **) &scores_index_d, size_of_score_index);
	hipMalloc((void **) &output_d, size_of_output);
	hipMalloc((void **) &output_index_d, size_of_output_index);
	//std::cout << "cuda malloc" << std::endl;
	
	hipMemcpy(scores_d, scores_h, size_of_score, hipMemcpyHostToDevice);
	hipMemcpy(scores_index_d, scores_index_h, size_of_score, hipMemcpyHostToDevice);
	//std::cout << "data copied" << std::endl;

	//std::cout << dimBlock << ", " << dimGrid << ", " << sharedMemSize << std::endl;

	
	//Call kernel
	int out_size = scores->size();
	argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, output_d, output_index_d, out_size);

	/*
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_index_h, output_index_d, size_of_output_index, hipMemcpyDeviceToHost);	
	for(int i=0; i < dimGrid; i++){
		std::cout << "max value: " << output_h[i] << ", maxIndex: " << output_index_h[i] << std::endl;
	}
	*/
	//std::cout << "first pass" << std::endl;
	//int i = 1;
	while(out_size > 1){
		//std::cout << i << "th pass" << std::endl;
		//i++;
		auto temp = output_d;
		auto temp_index = output_index_d;
		output_index_d = scores_index_d;
		output_d = scores_d;
		scores_index_d = temp_index;
		scores_d = temp;
		
		out_size = dimGrid;
		dimGrid = ceil((float)out_size/(float)dimBlock);
		//std::cout << dimBlock << ", " << dimGrid << ", " << sharedMemSize << std::endl;
		argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, output_d, output_index_d, out_size);		
		}

	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_index_h, output_index_d, size_of_output_index, hipMemcpyDeviceToHost);	
	
	

	hipFree(scores_d);
	hipFree(scores_index_d);
	hipFree(output_index_d);	
	hipFree(output_d);

	return output_index_h[0] ;
	
	
}