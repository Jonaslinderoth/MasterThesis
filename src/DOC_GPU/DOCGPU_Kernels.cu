#include "hip/hip_runtime.h"
#include "DOCGPU_Kernels.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <random>
#include <assert.h>

__global__ void findDimmensionsDevice(float* Xs_d, float* ps_d, bool* res_d, unsigned int* Dsum_out,
									  unsigned int point_dim, unsigned int no_of_samples, unsigned int no_in_sample, unsigned int no_of_ps, unsigned int m, float width){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	int pNo = entry/m;
	
	if(entry < no_of_samples){
		//if(pNo >= no_of_ps){
		//	printf("entry %i, looks for centorid %i, but there is only %i centroids. m : %i\n", entry, pNo, no_of_ps, m);
		//}
		assert(pNo < no_of_ps);
		unsigned int Dsum = 0;
		// for each dimension
		for(int i = 0; i < point_dim; i++){
			bool d = true;
			float p_tmp = ps_d[pNo*point_dim+i];
			// for each point in sample
			for(int j = 0; j < no_in_sample; j++){
				d &= abs(p_tmp-Xs_d[entry*no_in_sample*point_dim+j*point_dim+i]) < width;
			}
			res_d[entry*point_dim+i] = d;
			Dsum += d;

		}
		Dsum_out[entry] = Dsum;
	}
}

__global__ void pointsContainedDevice(float* data, float* centroids, bool* dims, bool* output, unsigned int* Csum_out,
									  float width, unsigned int point_dim, unsigned int no_data, unsigned int no_dims, unsigned int m){
	// one kernel for each hypercube
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				d &= (not (dims[entry*point_dim+i])) || (abs(centroids[currentCentroid*point_dim+i] - data[j*point_dim+i]) < width);
			}
			output[entry*no_data+j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;
	}
}

__global__ void score(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, unsigned int len, float alpha, float beta, unsigned int num_points){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	if(entry < len){
		score_output[entry] = ((Cluster_size[entry])* powf(1.0/beta, (Dim_count[entry])))*(Cluster_size[entry] >= (alpha*num_points));	
	}

}



float* scoreHost(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, int len, float alpha, float beta, unsigned int number_of_points){
	unsigned int* Cluster_size_d;
	unsigned int* Dim_count_d;
	float* score_output_d;
	hipMalloc((void **) &Cluster_size_d, len*sizeof(unsigned int));
	hipMalloc((void **) &Dim_count_d, len*sizeof(unsigned int));
	hipMalloc((void **) &score_output_d, len*sizeof(float));
	
	hipMemcpy(Cluster_size_d, Cluster_size, len*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(Dim_count_d, Dim_count, len*sizeof(unsigned int), hipMemcpyHostToDevice);

	score<<<ceil((len)/256.0), 256>>>(Cluster_size_d, Dim_count_d, score_output_d, len, alpha, beta, number_of_points);


	hipMemcpy(score_output, score_output_d, len*sizeof(float), hipMemcpyDeviceToHost);
	return score_output;
	
}


__global__ void createIndices(unsigned int* index, unsigned int length){
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i < length){
		index[i] = i;	
	}

};

__global__ void argMaxDevice(float* scores, unsigned int* scores_index, unsigned int input_size){
	extern __shared__ int array[];
	int* argData = (int*)array;
	float* scoreData = (float*) &argData[blockDim.x];

	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	argData[tid] = scores_index[i];
	scoreData[tid] = scores[i];
	
	__syncthreads();

	if(i < input_size){
		
		for(unsigned int s=(blockDim.x/2); s > 0; s/=2) {

			if(tid < s){
				if(scoreData[tid] < scoreData[tid+s]){
					scoreData[tid] = scoreData[tid+s];
					argData[tid] = argData[tid+s];
				}
			}
			__syncthreads();
		}
	
		if(tid == 0){
			scores_index[blockIdx.x] = argData[0];
			scores[blockIdx.x] = scoreData[0];
		}; 
		
	}
	
}

void findDimmensionsKernel(unsigned int dimGrid, unsigned int dimBlock, float* Xs_d, float* ps_d, bool* res_d,
						   unsigned int* Dsum_out, unsigned int point_dim, unsigned int no_of_samples, unsigned int sample_size, unsigned int no_of_ps,
						   unsigned int m, float width){

    findDimmensionsDevice<<<dimGrid, dimBlock>>>(Xs_d, ps_d, res_d,  Dsum_out,
												 point_dim, no_of_samples, sample_size,
												 no_of_ps, m, width);
	
};

void pointsContainedKernel(unsigned int dimGrid, unsigned int dimBlock,
						   float* data, float* centroids, bool* dims, bool* output, unsigned int* Csum_out,
						   float width, unsigned int point_dim, unsigned int no_data, unsigned int number_of_samples, unsigned int m){

	pointsContainedDevice<<<dimGrid, dimBlock>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m);
	
};


void scoreKernel(unsigned int dimGrid, unsigned int dimBlock,
				 unsigned int* cluster_size, unsigned int* dim_count, float* score_output,
				 unsigned int len, float alpha, float beta, unsigned int num_points){

	score<<<dimGrid, dimBlock>>>(cluster_size, dim_count, score_output,
								 len, alpha, beta, num_points);
	unsigned int* out = (unsigned int*) malloc(sizeof(unsigned int)*len);
	hipMemcpy(out, cluster_size, sizeof(unsigned int)*len, hipMemcpyDeviceToHost);
	/*std::cout << "cluster_size: " << std::endl;
	for(int i = 0; i < len; i++){
		std::cout << out[i] << ", ";
	}
	std::cout << std::endl;
	std::cout << num_points << std::endl;*/
	

	
};

void createIndicesKernel(unsigned int dimGrid, unsigned int dimBlock, unsigned int* index, unsigned int length){
	createIndices<<<dimGrid, dimBlock>>>(index, length);

};

void argMaxKernel(unsigned int dimGrid, unsigned int dimBlock, unsigned int sharedMemorySize,
				  float* scores, unsigned int* scores_index, unsigned int input_size){

	unsigned int* out = (unsigned int*) malloc(sizeof(unsigned int)*input_size);
	float* outScores = (float*) malloc(sizeof(float)*input_size);
	
	
	
	unsigned int out_size = input_size;
	while(out_size > 1){
		/*argMaxDevice<<<dimGrid, dimBlock, sharedMemorySize>>>(scores, scores_index, out_size);
		hipMemcpy(out, scores_index, sizeof(unsigned int)*input_size, hipMemcpyDeviceToHost);
		hipMemcpy(outScores, scores, sizeof(float)*input_size, hipMemcpyDeviceToHost);
		std::cout << "indecies: " << std::endl;
		for(int i = 0; i < out_size; i++){
			std::cout << out[i] << ", ";
		}
		std::cout << std::endl;
		std::cout << "Scores: " << std::endl;
		for(int i = 0; i < out_size; i++){
			std::cout << outScores[i] << ", ";
		}
		std::cout << std::endl;*/
		
		out_size = dimGrid;
		dimGrid = ceil((float)out_size/(float)dimBlock);
	}
	
	argMaxDevice<<<dimGrid, dimBlock, sharedMemorySize>>>(scores, scores_index, out_size);

	
};












std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> pointsContained(std::vector<std::vector<bool>*>* dims,
																					   std::vector<std::vector<float>*>* data,
																					   std::vector<std::vector<float>*>* centroids,
																					   int m, float width){

	// Calculaating sizes
	int point_dim = centroids->at(0)->size();
	int no_of_points = data->size();
	int no_of_dims = dims->size();
	int no_of_centroids = centroids->size();

	int floats_in_data = point_dim * no_of_points;
	int bools_in_dims = no_of_dims * point_dim;
	int floats_in_centorids = no_of_centroids * point_dim;
	int bools_in_output = no_of_points * no_of_dims;
	int ints_in_output_count = no_of_dims;
	
	int size_of_data = floats_in_data*sizeof(float);
	int size_of_dims = bools_in_dims*sizeof(bool);
	int size_of_centroids = floats_in_centorids*sizeof(float);
	int size_of_output = bools_in_output*sizeof(bool);
	int size_of_output_count = ints_in_output_count*sizeof(unsigned int);

	// allocating on the host
	float* data_h = (float*) malloc(size_of_data);
	bool* dims_h = (bool*) malloc(size_of_dims);
	float* centroids_h = (float*) malloc(size_of_centroids);
	bool* output_h = (bool*) malloc(size_of_output);
	unsigned int* output_count_h = (unsigned int*) malloc(size_of_output_count);

	// filling data array
	for(int i= 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	// filling dims array
	for(int i= 0; i < no_of_dims; i++){
		for(int j = 0; j < point_dim; j++){
			dims_h[i*point_dim+j] = dims->at(i)->at(j);
		}
	}

	// filling centroid array
	for(int i= 0; i < no_of_centroids; i++){
		for(int j = 0; j < point_dim; j++){
			centroids_h[i*point_dim+j] = centroids->at(i)->at(j);
		}
	}

	// allocating on device
	float* data_d;
	bool* dims_d;
	float* centroids_d;
	bool* output_d;
	unsigned int* output_count_d;
	
	hipMalloc((void **) &data_d, size_of_data);
	hipMalloc((void **) &dims_d, size_of_dims);
	hipMalloc((void **) &centroids_d, size_of_centroids);
	hipMalloc((void **) &output_d, size_of_output);
	hipMalloc((void **) &output_count_d, size_of_output_count);

	//Copy from host to device
				
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
	hipMemcpy(dims_d, dims_h, size_of_dims, hipMemcpyHostToDevice);
	hipMemcpy(centroids_d, centroids_h, size_of_centroids, hipMemcpyHostToDevice);


	// Call kernel
	pointsContainedDevice<<<ceil((no_of_dims)/256.0), 256>>>(data_d, centroids_d, dims_d, output_d, output_count_d, 
															 width, point_dim, no_of_points, no_of_dims, m);

	
	// copy from device
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_count_h, output_count_d, size_of_output_count, hipMemcpyDeviceToHost);

	// construnct output
	auto output =  new std::vector<std::vector<bool>*>;
	auto output_count =  new std::vector<unsigned int>;
   	
	
	for(int i = 0; i < no_of_dims; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < no_of_points; j++){
			a->push_back(output_h[i*no_of_points+j]);
		}
		output->push_back(a);
	}

	for(int i = 0; i < no_of_dims; i++){
		output_count->push_back(output_count_h[i]);
	}


	hipFree(data_d);
	hipFree(dims_d);
	hipFree(centroids_d);
	hipFree(output_d);
	hipFree(output_count_d);
	free(data_h);
	free(dims_h);
	free(centroids_h);
	free(output_h);
	free(output_count_h);

	
	return std::make_pair(output,output_count);
};


std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> findDimmensions(std::vector<std::vector<float>*>* ps,
																					   std::vector<std::vector<std::vector<float>*>*> Xs, int m, float width){

	int no_of_samples = Xs.size();	
	int no_in_sample = Xs.at(0)->size();
	int point_dim = Xs.at(0)->at(0)->size();


	
	int sizeOfXs = no_of_samples*no_in_sample*point_dim*sizeof(float);
	
	float* xs_h = (float*) malloc(sizeOfXs);

	for(int i = 0; i < no_of_samples; i++){
		for(int j = 0; j < no_in_sample; j++){
			for(int k = 0; k < point_dim; k++){
				xs_h[i*no_in_sample*point_dim+j*point_dim+k] = Xs.at(i)->at(j)->at(k);
			}
		}
	}


	int no_of_ps = ps->size();
	int sizeOfps = point_dim*no_of_ps*sizeof(float);
	float* ps_h = (float*) malloc(sizeOfps);


	for(int i = 0; i < no_of_ps; i++){
		for(int j = 0; j < point_dim; j++){
			ps_h[i*point_dim+j] = ps->at(i)->at(j);
		}
	}

	/*
	std::cout << "xs: " << std::endl;
	for(int i = 0; i < no_of_samples*no_in_sample*point_dim; i++){
		std::cout << xs_h[i] << ", ";
		if((i+1)% point_dim == 0){
			std::cout << std::endl;
		}
	}
	std::cout << std::endl;
	

	std::cout << "ps: " << std::endl;

	for(int i = 0; i < no_of_ps*point_dim; i++){
		std::cout << ps_h[i] << ", ";
	}
	std::cout << std::endl;
	*/
	unsigned int size_of_count = (no_of_samples)*sizeof(unsigned int);
	
	int outputDim = no_of_samples*point_dim;		
	int outputSize = outputDim*sizeof(bool);
	bool* result_h = (bool*) malloc(outputSize);
	unsigned int* count_h = (unsigned int*) malloc(size_of_count);


	float* Xs_d;
	float* ps_d;
	bool* result_d;
	unsigned int* count_d;
	
	hipMalloc((void **) &Xs_d, sizeOfXs);
	hipMalloc((void **) &ps_d, sizeOfps);
	hipMalloc((void **) &result_d, outputSize);
	hipMalloc((void **) &count_d, size_of_count);

	hipMemcpy( Xs_d, xs_h, sizeOfXs, hipMemcpyHostToDevice);
    hipMemcpy( ps_d, ps_h, sizeOfps, hipMemcpyHostToDevice);


	findDimmensionsDevice<<<ceil((no_of_samples)/256.0), 256>>>(Xs_d, ps_d, result_d, count_d, point_dim, no_of_samples, no_in_sample, no_of_ps, m, width);

   
	hipMemcpy(result_h, result_d, outputSize, hipMemcpyDeviceToHost);
	hipMemcpy(count_h, count_d, size_of_count, hipMemcpyDeviceToHost);

	auto output =  new std::vector<std::vector<bool>*>;
	
	for(int i = 0; i < no_of_samples; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < point_dim; j++){
			a->push_back(result_h[i*point_dim+j]);
		}
		output->push_back(a);
	}


	auto count = new std::vector<unsigned int>;
	for(int i = 0; i < (no_of_samples); i++){
		count->push_back(count_h[i]);
	}

	hipFree(Xs_d);
	hipFree(ps_d);
	hipFree(result_d);
	hipFree(count_d);
	free(result_h);
	free(count_h);
	free(ps_h);
	free(xs_h);
	
	return std::make_pair(output, count);
}



int argMax(std::vector<float>* scores){
	//Calculate size of shared Memory, block and thread dim
	//fetch device info
	// TODO: hardcoded device 0
	int smemSize, maxBlock;
	hipDeviceGetAttribute(&smemSize, 
    hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
	hipDeviceGetAttribute(&maxBlock, 
						   hipDeviceAttributeMaxThreadsPerBlock, 0); 

	// caluclate the maximum thread size based on shared mem requirements and maximum threads
	int dimBlock = smemSize/(sizeof(int)+sizeof(float));
	if(dimBlock > maxBlock) dimBlock = maxBlock;
	int dimGrid = ceil((float)scores->size()/(float)dimBlock);
	int sharedMemSize = (dimBlock*sizeof(unsigned int) + dimBlock*sizeof(float));

	int size_of_score = scores->size()*sizeof(float);
	int size_of_score_index = scores->size()*sizeof(unsigned int);

	
	float* scores_h = (float*) malloc(size_of_score);
	float* scores_d;
	unsigned int* scores_index_d;

	for(int i = 0; i < scores->size(); i++){
		scores_h[i] = scores->at(i);
	}
	

	hipMalloc((void **) &scores_d, size_of_score);
	hipMalloc((void **) &scores_index_d, size_of_score_index);

	hipMemcpy(scores_d, scores_h, size_of_score, hipMemcpyHostToDevice);
	
	//Call kernel
	int out_size = scores->size();
   
	createIndices<<<dimGrid, dimBlock>>>(scores_index_d, out_size);	
	
	while(out_size > 1){
		argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, out_size);				
		out_size = dimGrid;
		dimGrid = ceil((float)out_size/(float)dimBlock);
	}
	
	argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, out_size);		

	unsigned int size_of_output = sizeof(unsigned int);
	
	unsigned int* scores_index_h = (unsigned int*) malloc(size_of_output);

	hipMemcpy(scores_index_h, scores_index_d, size_of_output, hipMemcpyDeviceToHost);
	
	int result = scores_index_h[0];
	hipFree(scores_d);
	hipFree(scores_index_d);
	free(scores_index_h);
	free(scores_h);

	return result;
	
	
}



