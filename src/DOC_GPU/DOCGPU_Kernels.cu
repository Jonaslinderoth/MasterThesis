#include "hip/hip_runtime.h"
#include "DOCGPU_Kernels.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <random>
#include <assert.h>
#include <unistd.h>
#include <stdio.h>

/*
 * Finds the hypercube the for each medoid - discriminating set combination
*/
__global__ void findDimmensionsDevice(unsigned int* Xs_d, unsigned int* ps_d, float* data, bool* res_d, unsigned int* Dsum_out,
									  unsigned int point_dim, unsigned int no_of_samples, unsigned int no_in_sample, unsigned int no_of_ps, unsigned int m, float width, unsigned int no_data){
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int pNo = entry/m;
	if(entry < no_of_samples){
		assert(pNo < no_of_ps);
		unsigned int Dsum = 0;
		// for each dimension
		for(int i = 0; i < point_dim; i++){
			bool d = true;
			unsigned int tmp = ps_d[pNo]; // tmp is the index of the medoid in the dataset
			assert(tmp < no_data);
			float p_tmp = data[tmp*point_dim+i];
			// for each point in sample
			for(unsigned j = 0; j < no_in_sample; j++){
				assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
				unsigned int sampleNo = Xs_d[entry*no_in_sample+j];
				assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
				assert(sampleNo < no_data);
				float point = data[sampleNo*point_dim+i];
				d &= abs(p_tmp-point) < width;
			}
			res_d[entry*point_dim+i] = d;
			Dsum += d;
		}
		Dsum_out[entry] = Dsum;
	}
}



// Hack
struct floatArray{
	float f0;
	float f1;
	float f2;
	float f3;
	float f4;
	float f5;
	// float f6;
	// float f7;
};

/*
 * Finds the hypercube the for each medoid - discriminating set combination
 * From simple testing and nvprof, this kernel gets around twice the memory throughput as the naive version. 
 * 6 in a chunk allows for oly use 32 registers, 8 will give somewhat better memory perfomance
*/
__global__ void findDimmensionsLoadChunks(unsigned int* Xs_d, unsigned int* ps_d, float* data, bool* res_d, unsigned int* Dsum_out,
									  unsigned int point_dim, unsigned int no_of_samples, unsigned int no_in_sample, unsigned int no_of_ps, unsigned int m, float width, unsigned int no_data){
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int pNo = entry/m;
	floatArray p_tmp;
	floatArray x_tmp;
	bool d [6];
	if(entry < no_of_samples){
		assert(pNo < no_of_ps);
		unsigned int Dsum = 0;
		// for each dimension
		for(int i = 0; i < point_dim; i +=6){
			unsigned int tmp = ps_d[pNo]; // tmp is the index of the medoid in the dataset
			assert(tmp < no_data);
			if(i+8 > point_dim){
				for(; i < point_dim; i++){
					// if(threadIdx.x + blockIdx.x*blockDim.x == 0) printf("i %u  point_dim %u\n", i, point_dim);
					bool d = true;
					unsigned int tmp = ps_d[pNo]; // tmp is the index of the medoid in the dataset
					assert(tmp < no_data);
					float p_tmp = data[tmp*point_dim+i];
					// for each point in sample
					for(unsigned j = 0; j < no_in_sample; j++){
						assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
						unsigned int sampleNo = Xs_d[entry*no_in_sample+j];
						assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
						assert(sampleNo < no_data);
						float point = data[sampleNo*point_dim+i];
						d &= abs(p_tmp-point) < width;
					}
					res_d[entry*point_dim+i] = d;
					Dsum += d;
				}
				break;
			}else{
				p_tmp = *((floatArray*) (data+tmp*point_dim+i));// data[tmp*point_dim+i];
				// for each point in sample
				d[0] = true;
				d[1] = true;
				d[2] = true;
				d[3] = true;
				d[4] = true;
				d[5] = true;
				// d[6] = true;
				// d[7] = true;
				for(unsigned j = 0; j < no_in_sample; j++){
					unsigned int sampleNo = Xs_d[entry*no_in_sample+j];
					x_tmp = *((floatArray*)(data+sampleNo*point_dim+i));// data[tmp*point_dim+i];
					d[0] &= abs(p_tmp.f0-x_tmp.f0) < width;
					d[1] &= abs(p_tmp.f1-x_tmp.f1) < width;
					d[2] &= abs(p_tmp.f2-x_tmp.f2) < width;
					d[3] &= abs(p_tmp.f3-x_tmp.f3) < width;
					d[4] &= abs(p_tmp.f4-x_tmp.f4) < width;
					d[5] &= abs(p_tmp.f5-x_tmp.f5) < width;
					// d[6] &= abs(p_tmp.f6-x_tmp.f6) < width;
					// d[7] &= abs(p_tmp.f7-x_tmp.f7) < width;

					// if(threadIdx.x == 0) printf("i %u j %u p %f x %f from %u\n", i+0, j, p_tmp.f0, x_tmp.f0, data+sampleNo*point_dim+i);
					// if(threadIdx.x == 0) printf("i %u j %u p %f x %f from %u\n", i+1, j, p_tmp.f1, x_tmp.f1, data+sampleNo*point_dim+i+1);
					// if(threadIdx.x == 0) printf("i %u j %u p %f x %f from %u\n", i+2, j, p_tmp.f2, x_tmp.f2, data+sampleNo*point_dim+i+2);
					// if(threadIdx.x == 0) printf("i %u j %u p %f x %f from %u\n", i+3, j, p_tmp.f3, x_tmp.f3, data+sampleNo*point_dim+i+3);

					
				}
				res_d[entry*point_dim+i  ] = d[0];
				res_d[entry*point_dim+i+1] = d[1];
				res_d[entry*point_dim+i+2] = d[2];
				res_d[entry*point_dim+i+3] = d[3];
				res_d[entry*point_dim+i+4] = d[4];
				res_d[entry*point_dim+i+5] = d[5];
				// res_d[entry*point_dim+i+5] = d[6];
				// res_d[entry*point_dim+i+5] = d[7];
				Dsum += d[0] + d[1] + d[2] +d[3] +d[4] +d[5];// +d[6] +d[7];
			}
		}
		Dsum_out[entry] = Dsum;
	}
}




__global__ void score(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, unsigned int len, float alpha, float beta, unsigned int num_points){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	if(entry < len){
		score_output[entry] = ((Cluster_size[entry])* powf(1.0/beta, (Dim_count[entry])))*(Cluster_size[entry] >= (alpha*num_points));	
	}

}



float* scoreHost(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, int len, float alpha, float beta, unsigned int number_of_points){
	unsigned int* Cluster_size_d;
	unsigned int* Dim_count_d;
	float* score_output_d;
	hipMalloc((void **) &Cluster_size_d, len*sizeof(unsigned int));
	hipMalloc((void **) &Dim_count_d, len*sizeof(unsigned int));
	hipMalloc((void **) &score_output_d, len*sizeof(float));
	
	hipMemcpy(Cluster_size_d, Cluster_size, len*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(Dim_count_d, Dim_count, len*sizeof(unsigned int), hipMemcpyHostToDevice);

	score<<<ceil((len)/256.0), 256>>>(Cluster_size_d, Dim_count_d, score_output_d, len, alpha, beta, number_of_points);


	hipMemcpy(score_output, score_output_d, len*sizeof(float), hipMemcpyDeviceToHost);
	return score_output;
	
}





__global__ void randIntArrayInit(hiprandState_t* states , unsigned int seed, unsigned int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < size)
		{	hiprand_init(seed,   /* the seed controls the sequence of random values that are produced */
			idx, /* the sequence number is only important with multiple cores */
			0,/* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&states[idx]);
		}
}



/**
   Number of threads hsould be the same as the number of random states
 */
__global__ void randIntArray(unsigned int *result , hiprandState_t* states , const unsigned int number_of_states,
							 const unsigned int size , const unsigned int max, const unsigned min){
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberPrThread = ceilf((float)size/(float)number_of_states); // rounded down

	/*if(idx == 0){
		printf("number pr thread %u \n", numberPrThread);
		printf("size of number of states %u \n", number_of_states);
		printf("size of array %u \n", size);
		}*/
	
	if(idx < number_of_states){
		for(int i = 0; i < numberPrThread; i++){
			if(i*number_of_states+idx < size){
				float myrandf = hiprand_uniform(&states[idx]);
				myrandf *= (max - min + 0.9999);
				unsigned int res = (unsigned int)truncf(myrandf);
				res %= max;
				res += min;
				assert(res >= min);
				if(!(res <= max)){
					printf("res: %u, max: %u \n", res, max);
				}
				assert(res <= max);
				result[i*number_of_states+idx] = res;
			}
		}		
	}

}


__global__ void notKernel(bool* array, unsigned int length){
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numThreads = blockDim.x*gridDim.x;
	unsigned int numberPrThread = ceilf((float)length/(float)numThreads); // rounded down
	for(int i = 0; i < numberPrThread; i++){
		if(numberPrThread*i+idx < length){
			array[numberPrThread*i+idx] = not array[numberPrThread*i+idx];
		}
	}	
}






void notDevice(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream, bool* array, unsigned int length){
	notKernel<<<dimGrid, dimBlock, 0, stream>>>(array, length);
}

void findDimmensionsKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
						   unsigned int* Xs_d, unsigned int* ps_d, float* data, bool* res_d,
						   unsigned int* Dsum_out, unsigned int point_dim,
						   unsigned int no_of_samples, unsigned int sample_size,
						   unsigned int no_of_ps,
						   unsigned int m, float width, unsigned int no_data){

    findDimmensionsDevice<<<dimGrid, dimBlock, 0, stream>>>(Xs_d, ps_d, data, res_d, Dsum_out,
												 point_dim, no_of_samples, sample_size,
												 no_of_ps, m, width, no_data);
	
};



void scoreKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
				 unsigned int* cluster_size, unsigned int* dim_count, float* score_output,
				 unsigned int len, float alpha, float beta, unsigned int num_points){

	score<<<dimGrid, dimBlock, 0, stream>>>(cluster_size, dim_count, score_output,
								 len, alpha, beta, num_points);

	

	
};







std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> findDimmensions(std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   std::vector<std::vector<unsigned int>*>* samples,
																					   int m, float width){

	int no_of_samples = samples->size();	
	int no_in_sample = samples->at(0)->size();
	int no_of_centroids = centroids->size();

	int no_of_points = data->size();
	int point_dim = data->at(0)->size();
   
	int sizeOfData = no_of_points*point_dim*sizeof(unsigned int);
	int sizeOfSamples = no_of_samples*no_in_sample*sizeof(unsigned int);
	int sizeOfCentroids = point_dim*no_of_centroids*sizeof(unsigned int);
	
	
	unsigned int* centroids_h = (unsigned int*) malloc(sizeOfCentroids);
	unsigned int* samples_h = (unsigned int*) malloc(sizeOfSamples);
	float* data_h = (float*) malloc(sizeOfData);
	

	for(int i = 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_samples; i++){
		for(int j = 0;  j < no_in_sample; j++){
			samples_h[i*no_in_sample+j] = samples->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	unsigned int size_of_count = (no_of_samples)*sizeof(unsigned int);
	
	int outputDim = no_of_samples*point_dim;		
	int outputSize = outputDim*sizeof(bool);
	bool* result_h = (bool*) malloc(outputSize);
	unsigned int* count_h = (unsigned int*) malloc(size_of_count);


	unsigned int* samples_d;
	unsigned int* centroids_d;
	float* data_d;
	bool* result_d;
	unsigned int* count_d;
	
	hipMalloc((void **) &samples_d, sizeOfSamples);
	hipMalloc((void **) &centroids_d, sizeOfCentroids);
	hipMalloc((void **) &data_d, sizeOfData);
	hipMalloc((void **) &result_d, outputSize);
	hipMalloc((void **) &count_d, size_of_count);

	hipMemcpy( samples_d, samples_h, sizeOfSamples, hipMemcpyHostToDevice);
    hipMemcpy( centroids_d, centroids_h, sizeOfCentroids, hipMemcpyHostToDevice);
	hipMemcpy( data_d, data_h, sizeOfData, hipMemcpyHostToDevice);

	findDimmensionsDevice<<<ceil((no_of_samples)/256.0), 256>>>(samples_d, centroids_d,
																data_d, result_d, count_d,
																point_dim, no_of_samples,
																no_in_sample, no_of_centroids,
																m, width,no_of_points );



   
	hipMemcpy(result_h, result_d, outputSize, hipMemcpyDeviceToHost);
	hipMemcpy(count_h, count_d, size_of_count, hipMemcpyDeviceToHost);


	

	auto output =  new std::vector<std::vector<bool>*>;
	
	for(int i = 0; i < no_of_samples; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < point_dim; j++){
			a->push_back(result_h[i*point_dim+j]);
		}
		output->push_back(a);
	}


	auto count = new std::vector<unsigned int>;
	for(int i = 0; i < (no_of_samples); i++){
		count->push_back(count_h[i]);
	}

	hipFree(samples_d);
	hipFree(centroids_d);
	hipFree(result_d);
	hipFree(count_d);
	free(result_h);
	free(count_h);
	free(centroids_h);
	free(samples_h);
	
	return std::make_pair(output, count);
}




std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> findDimmensionsChunk(std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   std::vector<std::vector<unsigned int>*>* samples,
																					   int m, float width){

	int no_of_samples = samples->size();	
	int no_in_sample = samples->at(0)->size();
	int no_of_centroids = centroids->size();

	int no_of_points = data->size();
	int point_dim = data->at(0)->size();
   
	int sizeOfData = no_of_points*point_dim*sizeof(unsigned int);
	int sizeOfSamples = no_of_samples*no_in_sample*sizeof(unsigned int);
	int sizeOfCentroids = point_dim*no_of_centroids*sizeof(unsigned int);
	
	
	unsigned int* centroids_h = (unsigned int*) malloc(sizeOfCentroids);
	unsigned int* samples_h = (unsigned int*) malloc(sizeOfSamples);
	float* data_h = (float*) malloc(sizeOfData);
	

	for(int i = 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_samples; i++){
		for(int j = 0;  j < no_in_sample; j++){
			samples_h[i*no_in_sample+j] = samples->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	unsigned int size_of_count = (no_of_samples)*sizeof(unsigned int);
	
	int outputDim = no_of_samples*point_dim;		
	int outputSize = outputDim*sizeof(bool);
	bool* result_h = (bool*) malloc(outputSize);
	unsigned int* count_h = (unsigned int*) malloc(size_of_count);


	unsigned int* samples_d;
	unsigned int* centroids_d;
	float* data_d;
	bool* result_d;
	unsigned int* count_d;
	
	hipMalloc((void **) &samples_d, sizeOfSamples);
	hipMalloc((void **) &centroids_d, sizeOfCentroids);
	hipMalloc((void **) &data_d, sizeOfData);
	hipMalloc((void **) &result_d, outputSize);
	hipMalloc((void **) &count_d, size_of_count);

	hipMemcpy( samples_d, samples_h, sizeOfSamples, hipMemcpyHostToDevice);
    hipMemcpy( centroids_d, centroids_h, sizeOfCentroids, hipMemcpyHostToDevice);
	hipMemcpy( data_d, data_h, sizeOfData, hipMemcpyHostToDevice);

	findDimmensionsLoadChunks<<<ceil((no_of_samples)/256.0), 256>>>(samples_d, centroids_d,
																data_d, result_d, count_d,
																point_dim, no_of_samples,
																no_in_sample, no_of_centroids,
																m, width,no_of_points );



   
	hipMemcpy(result_h, result_d, outputSize, hipMemcpyDeviceToHost);
	hipMemcpy(count_h, count_d, size_of_count, hipMemcpyDeviceToHost);


	

	auto output =  new std::vector<std::vector<bool>*>;
	
	for(int i = 0; i < no_of_samples; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < point_dim; j++){
			a->push_back(result_h[i*point_dim+j]);
		}
		output->push_back(a);
	}


	auto count = new std::vector<unsigned int>;
	for(int i = 0; i < (no_of_samples); i++){
		count->push_back(count_h[i]);
	}

	hipFree(samples_d);
	hipFree(centroids_d);
	hipFree(result_d);
	hipFree(count_d);
	free(result_h);
	free(count_h);
	free(centroids_h);
	free(samples_h);
	
	return std::make_pair(output, count);
}



/*
 * this function makes the states of the random number generator
 * this needs to be called before generateRandomIntArrayDevice.
 * "save" the states to save on compiutational time.
 */
bool generateRandomStatesArray(hipStream_t stream,
							   hiprandState* d_randomStates,
							   const size_t size,
							   const bool randomSeed,
							   unsigned int seed,
							   unsigned int dimBlock){
	//set the seed
	if(randomSeed){
		std::random_device rd;
		seed = rd();
		//std::cout << "seed: " << seed << std::endl;
	}

#ifdef NDEBUG
	// nondebug
#else
	// debug code
	dimBlock = 512;
#endif
	//calculate the ammount of blocks
	int ammountOfBlocks = size/dimBlock;
	if(size%dimBlock != 0){
		ammountOfBlocks++;
	}
	randIntArrayInit<<<ammountOfBlocks,dimBlock, 0, stream>>>(d_randomStates ,seed, size);

	return true;
}


/* This fuction makes an array of random numbers between min and max in the gpu , given the allocation
 * and the states.
 * to get the states generate random states array call generateRandomStatesArray.
 */
bool generateRandomIntArrayDevice(hipStream_t stream,
								  unsigned int* randomIndexes_d,
								  hiprandState* randomStates_d,
								  const size_t size_of_randomStates,
								  const size_t size,
								  const unsigned int max,
								  const unsigned int min,
								  unsigned int dimBlock){
	if(max<min){
		return false;
	}

	// if there is more random states than what we need, dont spawn too many threads
	size_t accual_size = size_of_randomStates;
	if(accual_size > size) accual_size = size;

	//calculate the ammount of blocks
	int ammountOfBlocks = accual_size/dimBlock;
	if(accual_size%dimBlock != 0){
		ammountOfBlocks++;
	}
	//std::cout << "number of blocks: " << ammountOfBlocks << " number of threads: " << dimBlock << " size: " << size << " number of states: " << size_of_randomStates <<std::endl; 

	//std::cout << "max: " << max << std::endl;
	//call the generation of random numbers
	randIntArray<<<ammountOfBlocks,dimBlock, 0, stream>>>(randomIndexes_d, randomStates_d, size_of_randomStates, size , max , min);

	return true;
}











