#include "hip/hip_runtime.h"
#include "DOCGPU_Kernels.h"
#include <iostream>
#include <vector>
#include <math.h>
#include <random>
#include <assert.h>
#include <unistd.h>
#include <stdio.h>

__global__ void findDimmensionsDevice(unsigned int* Xs_d, unsigned int* ps_d, float* data, bool* res_d, unsigned int* Dsum_out,
									  unsigned int point_dim, unsigned int no_of_samples, unsigned int no_in_sample, unsigned int no_of_ps, unsigned int m, float width, unsigned int no_data){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int pNo = entry/m;
	if(entry < no_of_samples){
		if(!(pNo < no_of_ps)){
			printf("%u ", no_of_ps);
		}
		assert(pNo < no_of_ps);
		unsigned int Dsum = 0;
		// for each dimension
		for(int i = 0; i < point_dim; i++){
			bool d = true;
			unsigned int tmp = ps_d[pNo];
			assert(tmp < no_data);
			float p_tmp = data[tmp*point_dim+i];
			// for each point in sample
			for(unsigned j = 0; j < no_in_sample; j++){
				assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
				unsigned int sampleNo = Xs_d[entry*no_in_sample+j];
				assert(entry*no_in_sample+j < no_of_samples*no_in_sample);
				if(!(sampleNo < no_data)){
					printf("entry %u, no_in_sample %u, j %u, sampleNo %u, entry*no_in_sample+j %u, no_data %u \n", entry, no_in_sample, j, sampleNo, entry*no_in_sample+j, no_data);
				}
				//assert(sampleNo < no_data);
				float point = data[sampleNo*point_dim+i];
				d &= abs(p_tmp-point) < width;
			}
			res_d[entry*point_dim+i] = d;
			Dsum += d;

		}
		Dsum_out[entry] = Dsum;
	}
}

__global__ void pointsContainedDevice(float* data, unsigned int* centroids, bool* dims, bool* output, unsigned int* Csum_out,
									  float width, unsigned int point_dim, unsigned int no_data, unsigned int no_dims, unsigned int m){
	// one kernel for each hypercube
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				d &= (not (dims[entry*point_dim+i])) || (abs(data[centroid_index*point_dim+i] - data[j*point_dim+i]) < width);
			}

			if(!((size_t)entry*(size_t)no_data+(size_t)j < 2026532205)){
				//printf("entry %u, no_data %u, j %u, no_dims %u \n", entry, no_data, j, no_dims);
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			//assert((size_t)entry*(size_t)no_data+(size_t)j < 2026532205);
			
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;
	}
}

__global__ void score(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, unsigned int len, float alpha, float beta, unsigned int num_points){
	int entry = blockIdx.x*blockDim.x+threadIdx.x;
	if(entry < len){
		score_output[entry] = ((Cluster_size[entry])* powf(1.0/beta, (Dim_count[entry])))*(Cluster_size[entry] >= (alpha*num_points));	
	}

}



float* scoreHost(unsigned int* Cluster_size, unsigned int* Dim_count, float* score_output, int len, float alpha, float beta, unsigned int number_of_points){
	unsigned int* Cluster_size_d;
	unsigned int* Dim_count_d;
	float* score_output_d;
	hipMalloc((void **) &Cluster_size_d, len*sizeof(unsigned int));
	hipMalloc((void **) &Dim_count_d, len*sizeof(unsigned int));
	hipMalloc((void **) &score_output_d, len*sizeof(float));
	
	hipMemcpy(Cluster_size_d, Cluster_size, len*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(Dim_count_d, Dim_count, len*sizeof(unsigned int), hipMemcpyHostToDevice);

	score<<<ceil((len)/256.0), 256>>>(Cluster_size_d, Dim_count_d, score_output_d, len, alpha, beta, number_of_points);


	hipMemcpy(score_output, score_output_d, len*sizeof(float), hipMemcpyDeviceToHost);
	return score_output;
	
}


__global__ void createIndices(unsigned int* index, unsigned int length){
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i < length){
		index[i] = i;	
	}

};

__global__ void argMaxDevice(float* scores, unsigned int* scores_index, unsigned int input_size){
	extern __shared__ int array[];
	int* argData = (int*)array;
	float* scoreData = (float*) &argData[blockDim.x];
	
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;

	argData[tid] = 0;
	scoreData[tid] = 0;

	if(i < input_size){
	argData[tid] = scores_index[i];
	scoreData[tid] = scores[i];
	
	__syncthreads();		
		for(unsigned int s=(blockDim.x/2); s > 0; s/=2) {
			if(tid < s){
				assert(tid+s < blockDim.x);
				if(scoreData[tid] < scoreData[tid+s]){
					scoreData[tid] = scoreData[tid+s];
					argData[tid] = argData[tid+s];
				}
			}
			__syncthreads();
		}
	
		if(tid == 0){
			scores_index[blockIdx.x] = argData[0];
			scores[blockIdx.x] = scoreData[0];
		}; 
		
	}
	
}
__global__ void randIntArrayInit(hiprandState_t* states , unsigned int seed, unsigned int size){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < size)
		{	hiprand_init(seed,   /* the seed controls the sequence of random values that are produced */
			idx, /* the sequence number is only important with multiple cores */
			0,/* the offset is how much extra we advance in the sequence for each call, can be 0 */
			&states[idx]);
		}
}



/**
   Number of threads hsould be the same as the number of random states
 */
__global__ void randIntArray(unsigned int *result , hiprandState_t* states , const unsigned int number_of_states,
							 const unsigned int size , const unsigned int max, const unsigned min){
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numberPrThread = ceilf((float)size/(float)number_of_states); // rounded down

	/*if(idx == 0){
		printf("number pr thread %u \n", numberPrThread);
		printf("size of number of states %u \n", number_of_states);
		printf("size of array %u \n", size);
		}*/
	
	if(idx < number_of_states){
		for(int i = 0; i < numberPrThread; i++){
			if(i*number_of_states+idx < size){
				float myrandf = hiprand_uniform(&states[idx]);
				myrandf *= (max - min + 0.9999);
				unsigned int res = (unsigned int)truncf(myrandf);
				res %= max;
				res += min;
				assert(res >= min);
				assert(res <= max);
				result[i*number_of_states+idx] = res;
			}
		}		
	}

}


__global__ void notKernel(bool* array, unsigned int length){
	unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int numThreads = blockDim.x*gridDim.x;
	unsigned int numberPrThread = ceilf((float)length/(float)numThreads); // rounded down
	for(int i = 0; i < numberPrThread; i++){
		if(numberPrThread*i+idx < length){
			array[numberPrThread*i+idx] = not array[numberPrThread*i+idx];
		}
	}	
}

void notDevice(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream, bool* array, unsigned int length){
	notKernel<<<dimGrid, dimBlock, 0, stream>>>(array, length);
}

void findDimmensionsKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
						   unsigned int* Xs_d, unsigned int* ps_d, float* data, bool* res_d,
						   unsigned int* Dsum_out, unsigned int point_dim,
						   unsigned int no_of_samples, unsigned int sample_size,
						   unsigned int no_of_ps,
						   unsigned int m, float width, unsigned int no_data){

    findDimmensionsDevice<<<dimGrid, dimBlock, 0, stream>>>(Xs_d, ps_d, data, res_d, Dsum_out,
												 point_dim, no_of_samples, sample_size,
												 no_of_ps, m, width, no_data);
	
};

void pointsContainedKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
						   float* data, unsigned int* centroids, bool* dims, bool* output, unsigned int* Csum_out,
						   float width, unsigned int point_dim, unsigned int no_data, unsigned int number_of_samples,
						   unsigned int m){

	pointsContainedDevice<<<dimGrid, dimBlock, 0, stream>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m);
	
};


void scoreKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream,
				 unsigned int* cluster_size, unsigned int* dim_count, float* score_output,
				 unsigned int len, float alpha, float beta, unsigned int num_points){

	score<<<dimGrid, dimBlock, 0, stream>>>(cluster_size, dim_count, score_output,
								 len, alpha, beta, num_points);

	

	
};

void createIndicesKernel(unsigned int dimGrid, unsigned int dimBlock, hipStream_t stream, unsigned int* index, unsigned int length){
	createIndices<<<dimGrid, dimBlock, 0, stream>>>(index, length);

};

void argMaxKernel(unsigned int dimGrid, unsigned int dimBlock, unsigned int sharedMemorySize,
				  hipStream_t stream,
				  float* scores, unsigned int* scores_index, unsigned int input_size){

	unsigned int* out = (unsigned int*) malloc(sizeof(unsigned int)*input_size);
	float* outScores = (float*) malloc(sizeof(float)*input_size);
	
	
	
	unsigned int out_size = input_size;
	while(out_size > 1){
	   
		argMaxDevice<<<dimGrid, dimBlock, sharedMemorySize, stream>>>(scores, scores_index, out_size);
		out_size = dimGrid;
		dimGrid = ceil((float)out_size/(float)dimBlock);
	}
	
};


std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> pointsContained(std::vector<std::vector<bool>*>* dims,
																					   std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   int m, float width){

	// Calculaating sizes
	int point_dim = data->at(0)->size();
	int no_of_points = data->size();
	int no_of_dims = dims->size();
	int no_of_centroids = centroids->size();

	int floats_in_data = point_dim * no_of_points;
	int bools_in_dims = no_of_dims * point_dim;
	int bools_in_output = no_of_points * no_of_dims;
	int ints_in_output_count = no_of_dims;
	
	int size_of_data = floats_in_data*sizeof(float);
	int size_of_dims = bools_in_dims*sizeof(bool);
	int size_of_centroids = no_of_centroids*sizeof(unsigned int);
	int size_of_output = bools_in_output*sizeof(bool);
	int size_of_output_count = ints_in_output_count*sizeof(unsigned int);

	// allocating on the host
	float* data_h = (float*) malloc(size_of_data);
	bool* dims_h = (bool*) malloc(size_of_dims);
	unsigned int* centroids_h = (unsigned int*) malloc(size_of_centroids);
	bool* output_h = (bool*) malloc(size_of_output);
	unsigned int* output_count_h = (unsigned int*) malloc(size_of_output_count);

	// filling data array
	for(int i= 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	// filling dims array
	for(int i= 0; i < no_of_dims; i++){
		for(int j = 0; j < point_dim; j++){
			dims_h[i*point_dim+j] = dims->at(i)->at(j);
		}
	}

	// filling centroid array
	for(int i= 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	// allocating on device
	float* data_d;
	bool* dims_d;
	unsigned int* centroids_d;
	bool* output_d;
	unsigned int* output_count_d;
	
	hipMalloc((void **) &data_d, size_of_data);
	hipMalloc((void **) &dims_d, size_of_dims);
	hipMalloc((void **) &centroids_d, size_of_centroids);
	hipMalloc((void **) &output_d, size_of_output);
	hipMalloc((void **) &output_count_d, size_of_output_count);

	//Copy from host to device
				
	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
	hipMemcpy(dims_d, dims_h, size_of_dims, hipMemcpyHostToDevice);
	hipMemcpy(centroids_d, centroids_h, size_of_centroids, hipMemcpyHostToDevice);


	// Call kernel
	pointsContainedDevice<<<ceil((no_of_dims)/256.0), 256>>>(data_d, centroids_d, dims_d, output_d, output_count_d, 
															 width, point_dim, no_of_points, no_of_dims, m);

	
	// copy from device
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_count_h, output_count_d, size_of_output_count, hipMemcpyDeviceToHost);

	// construnct output
	auto output =  new std::vector<std::vector<bool>*>;
	auto output_count =  new std::vector<unsigned int>;
   	
	
	for(int i = 0; i < no_of_dims; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < no_of_points; j++){
			a->push_back(output_h[i*no_of_points+j]);
		}
		output->push_back(a);
	}

	for(int i = 0; i < no_of_dims; i++){
		output_count->push_back(output_count_h[i]);
	}


	hipFree(data_d);
	hipFree(dims_d);
	hipFree(centroids_d);
	hipFree(output_d);
	hipFree(output_count_d);
	free(data_h);
	free(dims_h);
	free(centroids_h);
	free(output_h);
	free(output_count_h);

	
	return std::make_pair(output,output_count);
};


std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> findDimmensions(std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   std::vector<std::vector<unsigned int>*>* samples,
																					   int m, float width){

	int no_of_samples = samples->size();	
	int no_in_sample = samples->at(0)->size();
	int no_of_centroids = centroids->size();

	int no_of_points = data->size();
	int point_dim = data->at(0)->size();
   
	int sizeOfData = no_of_points*point_dim*sizeof(unsigned int);
	int sizeOfSamples = no_of_samples*no_in_sample*sizeof(unsigned int);
	int sizeOfCentroids = point_dim*no_of_centroids*sizeof(unsigned int);
	
	
	unsigned int* centroids_h = (unsigned int*) malloc(sizeOfCentroids);
	unsigned int* samples_h = (unsigned int*) malloc(sizeOfSamples);
	float* data_h = (float*) malloc(sizeOfData);
	

	for(int i = 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_samples; i++){
		for(int j = 0;  j < no_in_sample; j++){
			samples_h[i*no_in_sample+j] = samples->at(i)->at(j);
		}
	}

	for(int i = 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	unsigned int size_of_count = (no_of_samples)*sizeof(unsigned int);
	
	int outputDim = no_of_samples*point_dim;		
	int outputSize = outputDim*sizeof(bool);
	bool* result_h = (bool*) malloc(outputSize);
	unsigned int* count_h = (unsigned int*) malloc(size_of_count);


	unsigned int* samples_d;
	unsigned int* centroids_d;
	float* data_d;
	bool* result_d;
	unsigned int* count_d;
	
	hipMalloc((void **) &samples_d, sizeOfSamples);
	hipMalloc((void **) &centroids_d, sizeOfCentroids);
	hipMalloc((void **) &data_d, sizeOfData);
	hipMalloc((void **) &result_d, outputSize);
	hipMalloc((void **) &count_d, size_of_count);

	hipMemcpy( samples_d, samples_h, sizeOfSamples, hipMemcpyHostToDevice);
    hipMemcpy( centroids_d, centroids_h, sizeOfCentroids, hipMemcpyHostToDevice);
	hipMemcpy( data_d, data_h, sizeOfData, hipMemcpyHostToDevice);


	findDimmensionsDevice<<<ceil((no_of_samples)/256.0), 256>>>(samples_d, centroids_d,
																data_d, result_d, count_d,
																point_dim, no_of_samples,
																no_in_sample, no_of_centroids,
																m, width,no_of_points );

   
	hipMemcpy(result_h, result_d, outputSize, hipMemcpyDeviceToHost);
	hipMemcpy(count_h, count_d, size_of_count, hipMemcpyDeviceToHost);


	

	auto output =  new std::vector<std::vector<bool>*>;
	
	for(int i = 0; i < no_of_samples; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < point_dim; j++){
			a->push_back(result_h[i*point_dim+j]);
		}
		output->push_back(a);
	}


	auto count = new std::vector<unsigned int>;
	for(int i = 0; i < (no_of_samples); i++){
		count->push_back(count_h[i]);
	}

	hipFree(samples_d);
	hipFree(centroids_d);
	hipFree(result_d);
	hipFree(count_d);
	free(result_h);
	free(count_h);
	free(centroids_h);
	free(samples_h);
	
	return std::make_pair(output, count);
}



int argMax(std::vector<float>* scores){
	//Calculate size of shared Memory, block and thread dim
	//fetch device info
	// TODO: hardcoded device 0
	int smemSize, maxBlock;
	hipDeviceGetAttribute(&smemSize, 
    hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
	hipDeviceGetAttribute(&maxBlock, 
						   hipDeviceAttributeMaxThreadsPerBlock, 0); 

	// caluclate the maximum thread size based on shared mem requirements and maximum threads
	int dimBlock = smemSize/(sizeof(int)+sizeof(float));
	if(dimBlock > maxBlock) dimBlock = maxBlock;
	int dimGrid = ceil((float)scores->size()/(float)dimBlock);
	int sharedMemSize = (dimBlock*sizeof(unsigned int) + dimBlock*sizeof(float));

	int size_of_score = scores->size()*sizeof(float);
	int size_of_score_index = scores->size()*sizeof(unsigned int);

	
	float* scores_h = (float*) malloc(size_of_score);
	float* scores_d;
	unsigned int* scores_index_d;

	for(int i = 0; i < scores->size(); i++){
		scores_h[i] = scores->at(i);
	}
	

	hipMalloc((void **) &scores_d, size_of_score);
	hipMalloc((void **) &scores_index_d, size_of_score_index);

	hipMemcpy(scores_d, scores_h, size_of_score, hipMemcpyHostToDevice);
	
	//Call kernel
	int out_size = scores->size();
   
	createIndices<<<dimGrid, dimBlock>>>(scores_index_d, out_size);
   
	
	while(out_size > 1){
		//std::cout << "called" << std::endl;
		//std::cout << "dimGrid:" << dimGrid << " dimBlock: " << dimBlock << " out_size: " << out_size << std::endl;
		argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, out_size);				
		out_size = dimGrid;
		dimGrid = ceil((float)out_size/(float)dimBlock);
	}
	
	//argMaxDevice<<<dimGrid, dimBlock, sharedMemSize>>>(scores_d, scores_index_d, out_size);		

	unsigned int size_of_output = sizeof(unsigned int);
	
	unsigned int* scores_index_h = (unsigned int*) malloc(size_of_output);

	hipMemcpy(scores_index_h, scores_index_d, size_of_output, hipMemcpyDeviceToHost);
	
	int result = scores_index_h[0];
	hipFree(scores_d);
	hipFree(scores_index_d);
	free(scores_index_h);
	free(scores_h);

	return result;
	
	
}

/*
 * this function makes the states of the random number generator
 * this needs to be called before generateRandomIntArrayDevice.
 * "save" the states to save on compiutational time.
 */
bool generateRandomStatesArray(hipStream_t stream,
							   hiprandState* d_randomStates,
							   const size_t size,
							   const bool randomSeed,
							   unsigned int seed,
							   unsigned int dimBlock){
	//set the seed
	if(randomSeed){
		std::random_device rd;
		seed = rd();
		//std::cout << "seed: " << seed << std::endl;
	}

#ifdef NDEBUG
	// nondebug
#else
	// debug code
	dimBlock = 512;
#endif
	//calculate the ammount of blocks
	int ammountOfBlocks = size/dimBlock;
	if(size%dimBlock != 0){
		ammountOfBlocks++;
	}
	randIntArrayInit<<<ammountOfBlocks,dimBlock, 0, stream>>>(d_randomStates ,seed, size);

	return true;
}


/* This fuction makes an array of random numbers between min and max in the gpu , given the allocation
 * and the states.
 * to get the states generate random states array call generateRandomStatesArray.
 */
bool generateRandomIntArrayDevice(hipStream_t stream,
								  unsigned int* randomIndexes_d,
								  hiprandState* randomStates_d,
								  const size_t size_of_randomStates,
								  const size_t size,
								  const unsigned int max,
								  const unsigned int min,
								  unsigned int dimBlock){

	if(max<min){
		return false;
	}
	
	// if there is more random states than what we need, dont spawn too many threads
	size_t accual_size = size_of_randomStates;
	if(accual_size > size) accual_size = size;



	//calculate the ammount of blocks
	int ammountOfBlocks = accual_size/dimBlock;
	if(accual_size%dimBlock != 0){
		ammountOfBlocks++;
	}
	//std::cout << "number of blocks: " << ammountOfBlocks << " number of threads: " << dimBlock << " size: " << size << " number of states: " << size_of_randomStates <<std::endl; 

	//std::cout << "max: " << max << std::endl;
	//call the generation of random numbers
	randIntArray<<<ammountOfBlocks,dimBlock, 0, stream>>>(randomIndexes_d, randomStates_d, size_of_randomStates, size , max , min);

	return true;
}


