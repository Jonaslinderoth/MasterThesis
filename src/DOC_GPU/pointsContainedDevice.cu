#include "hip/hip_runtime.h"
#include "pointsContainedDevice.h"
#include "../randomCudaScripts/Utils.h"
#include "../randomCudaScripts/DeleteFromArray.h"
#include "../Fast_DOCGPU/whatDataInCentroid.h"
#include <assert.h>
#include <utility>
#include <vector>
#include <iostream>
#include "../randomCudaScripts/arrayEqual.h"

/*
 * This fuction returns if the points are in the hypercube made by the centroid by using a subset of the dimensions
 * It does not use anything fancy.
 */
__global__ void pointsContainedDeviceNaive(float* data, unsigned int* centroids, bool* dims, bool* output, unsigned int* Csum_out,
									  float width, unsigned int point_dim, unsigned int no_data, unsigned int no_dims, unsigned int m){
	// one kernel for each hypercube
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				//if(!(centroid_index < no_data)){
				//    printf("num_data: %u, centroid_index: %u, currentCentroid: %u \n", no_data, centroid_index, currentCentroid);
				//}
				assert(centroid_index < no_data);
				assert(entry*point_dim+i < no_dims*point_dim);
				assert(centroid_index*point_dim+i < no_data*point_dim);
				assert(j*point_dim+i < no_data*point_dim);
				const unsigned long entryDims = entry*point_dim+i;
				const float centro = data[centroid_index*point_dim+i];
				const float punto = data[j*point_dim+i];
				const float abss = abs(centro - punto);
				d &= (not (dims[entryDims])) || (abss < width);
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;

	}
}


/*
 * This fuction returns if the points are in the hypercube made by the centroid by using a subset of the dimensions
 * It does tries to break early every .
 */
__global__ void pointsContainedDeviceNaiveBreak(float* data,
											    unsigned int* centroids,
											    bool* dims,
											    bool* output,
											    unsigned int* Csum_out,
											    float width,
											    unsigned int point_dim,
											    unsigned int no_data,
											    unsigned int no_dims,
											    unsigned int m,
											    unsigned int breakingIntervall){
	// one kernel for each hypercub
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				//if(!(centroid_index < no_data)){
				//    printf("num_data: %u, centroid_index: %u, currentCentroid: %u \n", no_data, centroid_index, currentCentroid);
				//}
				
				assert(centroid_index < no_data);
				assert(entry*point_dim+i < no_dims*point_dim);
				assert(centroid_index*point_dim+i < no_data*point_dim);
				assert(j*point_dim+i < no_data*point_dim);
				const unsigned long entryDims = entry*point_dim+i;
				const float centro = data[centroid_index*point_dim+i];
				const float punto = data[j*point_dim+i];
				const float abss = abs(centro - punto);
				d &= (not (dims[entryDims])) || (abss < width);
				if(i%breakingIntervall == 0 and (not d)){
					break;

				}
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;

	}
}

/*
 * This does the same as the naive but it moved the data to shared memory before.
 * making
 *
 */
__global__ void pointsContainedDeviceSharedMemory(float* data,
		unsigned int* centroids,
		bool* dims,
		bool* output,
		unsigned int* Csum_out,
		float width,
		const unsigned int point_dim,
		const unsigned int no_data_p,
		const unsigned int no_dims,
		const unsigned int ammountOfSamplesThatUseSameCentroid,
		const unsigned int numberOfCentroids,
		const unsigned int centroidSharedMemorySize_f,
		const unsigned int dataSharedMemorySize_f,
		const unsigned int blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];


	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	// if(threadIdx.x < point_dim){  // OBS... This kernel will not work for dims higher than the block size
	// 	const long offsetByDimension_f = threadIdx.x;
	// 	const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
	// 	const float partOfACentroid_f = data[indexOfCentroidInData_f];
	// 	centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	// }

	for(unsigned int i = 0; i < ceilf((float)point_dim/blockDim.x); i++){
		const unsigned int j = i*blockDim.x+threadIdx.x;
		if(j < point_dim){
			centroidSharedMemory[j] = data[indexOfCentroidInDataNoDims_f+j];
		}
	}

	// it is okay to not have 
	

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long whatM = blockIdx.x / blocksWithSameCentroid;
	const unsigned long offEntry_p = howLongOnM_p + whatM * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;


	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){
			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;
			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;
			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f ){
				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];
			}
		}
		__syncthreads();

		const unsigned long ammountOfPointsLeft = no_data_p-i_p;
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < dataSharedMemorySize_p ; indexDataSM_p++){
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = indexDataSM_p;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;


			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and indexDataSM_p < ammountOfPointsLeft and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){

					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}
				output[indexOutput_p] = d;
				Csum += d;
			}

		}
		__syncthreads();

	}

	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;

	}

}



/*
 * This does the same as the naive but it moved the data to shared memory before.
 * this contained early breaking
 *
 */
__global__ void pointsContainedDeviceSharedMemoryBreak(float* data,
													   unsigned int* centroids,
													   bool* dims,
													   bool* output,
													   unsigned int* Csum_out,
													   float width,
													   const unsigned int point_dim,
													   const unsigned int no_data_p,
													   const unsigned int no_dims,
													   const unsigned int ammountOfSamplesThatUseSameCentroid,
													   const unsigned int numberOfCentroids,
													   const unsigned int centroidSharedMemorySize_f,
													   const unsigned int dataSharedMemorySize_f,
													   const unsigned int blocksWithSameCentroid,
													   const unsigned int breakingIntervall){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];


	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	// if(threadIdx.x < point_dim){  // OBS... This kernel will not work for dims higher than the block size
	// 	const long offsetByDimension_f = threadIdx.x;
	// 	const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
	// 	const float partOfACentroid_f = data[indexOfCentroidInData_f];
	// 	centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	// }

	for(unsigned int i = 0; i < ceilf((float)point_dim/blockDim.x); i++){
		const unsigned int j = i*blockDim.x+threadIdx.x;
		if(j < point_dim){
			centroidSharedMemory[j] = data[indexOfCentroidInDataNoDims_f+j];
		}
	}

	// it is okay to not have


	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long whatM = blockIdx.x / blocksWithSameCentroid;
	const unsigned long offEntry_p = howLongOnM_p + whatM * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;


	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){
			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;
			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;
			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f ){
				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];
			}
		}
		__syncthreads();

		const unsigned long ammountOfPointsLeft = no_data_p-i_p;
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < dataSharedMemorySize_p ; indexDataSM_p++){
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = indexDataSM_p;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;


			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and indexDataSM_p < ammountOfPointsLeft and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){

					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);
					if(dimensionIndex_f % breakingIntervall == 0 and (not d)){
						break;
					}

				}
				output[indexOutput_p] = d;
				Csum += d;
			}

		}
		__syncthreads();

	}

	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;

	}

}

__global__ void pointsContainedDeviceSharedMemoryFewBank(float* __restrict__ data,
		unsigned int* __restrict__ centroids,
		bool* __restrict__ dims,
		bool* __restrict__ output,
		unsigned int* __restrict__ Csum_out,
		float width,
		const unsigned long point_dim,
		const unsigned long no_data_p,
		const unsigned long no_dims,
		const unsigned long ammountOfSamplesThatUseSameCentroid,
		const unsigned long numberOfCentroids,
		const unsigned long centroidSharedMemorySize_f,
		const unsigned long dataSharedMemorySize_f,
		const unsigned long blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];

	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.

	// // This only allows for medoids of upt to block
	// if(threadIdx.x < point_dim){
	// 	const long offsetByDimension_f = threadIdx.x;
	// 	const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
	// 	const float partOfACentroid_f = data[indexOfCentroidInData_f];
	// 	centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	// }

	
	for(unsigned int i = 0; i < ceilf((float)point_dim/blockDim.x); i++){
		const unsigned int j = i*blockDim.x+threadIdx.x;
		if(j < point_dim){
			centroidSharedMemory[j] = data[indexOfCentroidInDataNoDims_f+j];
		}
	}
	//now i have the centroid in shared memory. but its going to cause bank conflicts :( , TODO fix this.

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long offEntry_p = howLongOnM_p + ( blockIdx.x / blocksWithSameCentroid ) * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;
	//assert(dataSharedMemorySize_f%point_dim == 0);



	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){

			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;

			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;

			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f){

				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];

			}
		}
		__syncthreads();


		const unsigned long warpId_p = 0;//threadIdx.x/(32*point_dim);
		const unsigned long warpIdToDim_p =threadIdx.x/32;
		const unsigned long dataPointsLeft = no_data_p-i_p;
		const unsigned long limit = min(dataPointsLeft,dataSharedMemorySize_p);
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < limit ; indexDataSM_p++)
		{

			//const unsigned long indexPointDataSM_p = indexDataSM_p;
			//const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = ( indexDataSM_p + warpId_p ) % limit;
			const unsigned long indexPointDataSMNotOffset_p = indexDataSM_p;
			const long offset_p = indexPointDataSM_p - indexPointDataSMNotOffset_p;
			//const unsigned long indexDimsNoDimension_f = ( offEntry_p + offset_p ) * point_dim;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p + offset_p;

			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){
					/*
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);
					*/

					const unsigned long offDimensionIndex = (dimensionIndex_f+warpIdToDim_p)%point_dim;
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + offDimensionIndex;
					const unsigned long indexDims_f = indexDimsNoDimension_f + offDimensionIndex;
					assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[offDimensionIndex];
					assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}

				//assert(indexOutput_p < ammountOfSamplesThatUseSameCentroid*numberOfCentroids*no_data_p);
				output[indexOutput_p] = d;

				Csum += d;
			}

		}
		__syncthreads();

	}
	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;
	}

}




__global__ void pointsContainedDeviceSharedMemoryFewerBank(float* __restrict__ data,
		unsigned int* __restrict__ centroids,
		bool* __restrict__ dims,
		bool* __restrict__ output,
		unsigned int* __restrict__ Csum_out,
		float width,
		const unsigned long point_dim,
		const unsigned long no_data_p,
		const unsigned long no_dims,
		const unsigned long ammountOfSamplesThatUseSameCentroid,
		const unsigned long numberOfCentroids,
		const unsigned long centroidSharedMemorySize_f,
		const unsigned long dataSharedMemorySize_f,
		const unsigned long blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];

	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	if(threadIdx.x < point_dim){
		const long offsetByDimension_f = threadIdx.x;
		const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
		const float partOfACentroid_f = data[indexOfCentroidInData_f];
		centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	}
	//now i have the centroid in shared memory. but its going to cause bank conflicts :( , TODO fix this.

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long offEntry_p = howLongOnM_p + ( blockIdx.x / blocksWithSameCentroid ) * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;
	//assert(dataSharedMemorySize_f%point_dim == 0);



	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){

			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;

			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;

			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f){

				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];

			}
		}
		__syncthreads();


		const unsigned long warpId_p = threadIdx.x/(32*point_dim);
		const unsigned long warpIdToDim_p =threadIdx.x/32;
		const unsigned long dataPointsLeft = no_data_p-i_p;
		const unsigned long limit = min(dataPointsLeft,dataSharedMemorySize_p);
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < limit ; indexDataSM_p++)
		{

			//const unsigned long indexPointDataSM_p = indexDataSM_p;
			//const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = ( indexDataSM_p + warpId_p ) % limit;
			const unsigned long indexPointDataSMNotOffset_p = indexDataSM_p;
			const long offset_p = indexPointDataSM_p - indexPointDataSMNotOffset_p;
			//const unsigned long indexDimsNoDimension_f = ( offEntry_p + offset_p ) * point_dim;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p + offset_p;

			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){
					/*
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);
					*/

					const unsigned long offDimensionIndex = (dimensionIndex_f+warpIdToDim_p)%point_dim;
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + offDimensionIndex;
					const unsigned long indexDims_f = indexDimsNoDimension_f + offDimensionIndex;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[offDimensionIndex];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}

				//assert(indexDataSM_p < dataSharedMemorySize_p);

				//assert(indexOutput_p < ammountOfSamplesThatUseSameCentroid*numberOfCentroids*no_data_p);
				output[indexOutput_p] = d;

				Csum += d;
			}

		}
		__syncthreads();

	}
	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;

	}

}


__global__ void gpuNotBoolArray(bool* inputAndOutPut,
							 std::size_t lenght){
	const size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx < lenght){
		inputAndOutPut[idx] = not inputAndOutPut[idx];
	}
}


/*
 * This fuction returns if the points are in the hypercube made by the centroid by using a subset of the dimensions
 * It does not use anything fancy.
 */
__global__ void pointsContainedDeviceNaiveFewPoints(float* data,
													unsigned int* centroids,
													bool* output,
													unsigned int* Csum_out,
													const float width,
													const unsigned int point_dim,
													const unsigned int no_data,
													const unsigned int m,
													const unsigned int numberOfCentroids){

	const unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	const unsigned int currentCentroid = idx/m;

	//check that we are not going out of bound on the centroids array
	if(idx < 1){
		const unsigned int centroidIndex = centroids[currentCentroid];
		const float* centroidArray = &data[centroidIndex*point_dim];
		unsigned int Csum = 0;

		for(unsigned int dataIndex = 0; dataIndex < no_data; ++dataIndex){

			const float* pointArray = &data[dataIndex*point_dim];
			bool d = true;
			for(unsigned int dimensionIndex = 0; dimensionIndex < point_dim; ++dimensionIndex){
				const float centroid_f = centroidArray[dimensionIndex];
				const float data_f = pointArray[dimensionIndex];
				const float diference = abs(centroid_f - data_f);
				d &= (diference < width);
			}
			output[(size_t)idx*(size_t)no_data+(size_t)dataIndex] = d;
			Csum += d;
			//printf(" no_data: %u \n dataIndex: %u \n",no_data,dataIndex);
		}

		Csum_out[idx] = Csum;
	}
	/*
	// one kernel for each hypercube
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				//if(!(centroid_index < no_data)){
				//    printf("num_data: %u, centroid_index: %u, currentCentroid: %u \n", no_data, centroid_index, currentCentroid);
				//}
				assert(centroid_index < no_data);
				assert(entry*point_dim+i < no_dims*point_dim);
				assert(centroid_index*point_dim+i < no_data*point_dim);
				assert(j*point_dim+i < no_data*point_dim);
				const unsigned long entryDims = entry*point_dim+i;
				const float centro = data[centroid_index*point_dim+i];
				const float punto = data[j*point_dim+i];
				const float abss = abs(centro - punto);
				d &= (not (dims[entryDims])) || (abss < width);
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;
	}
	*/
}



void pointsContainedKernelNaive(unsigned int dimGrid,
							    unsigned int dimBlock,
							    hipStream_t stream,
							    float* data,
							    unsigned int* centroids,
							    bool* dims, bool* output,
							    unsigned int* Csum_out,
							    float width,
							    unsigned int point_dim,
							    unsigned int no_data,
							    unsigned int no_dims,
							    unsigned int m){

	pointsContainedDeviceNaive<<<dimGrid, dimBlock, 0, stream>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, no_dims, m);


};

void pointsContainedKernelNaiveBreak(unsigned int dimGrid,
									 unsigned int dimBlock,
									 hipStream_t stream,
									 float* data,
									 unsigned int* centroids,
									 bool* dims,
									 bool* output,
									 unsigned int* Csum_out,
									 float width,
									 unsigned int point_dim,
									 unsigned int no_data,
									 unsigned int no_dims,
									 unsigned int m,
									 unsigned int breakingIntervall){

	pointsContainedDeviceNaiveBreak<<<dimGrid, dimBlock, 0, stream>>>(data,
																	  centroids,
																	  dims,
																	  output,
																	  Csum_out,
																	  width,
																	  point_dim,
																	  no_data,
																	  no_dims,
																	  m,
																	  breakingIntervall);
}


void pointsContainedKernelSharedMemory(unsigned int dimGrid,
									   unsigned int dimBlock,
									   hipStream_t stream,
									   float* data,
									   unsigned int* centroids,
									   bool* dims,
									   bool* output,
									   unsigned int* Csum_out,
									   float width,
									   unsigned int point_dim,
									   unsigned int no_data,
									   unsigned int number_of_samples,
									   unsigned int m,
									   unsigned int numberOfCentroids,
									   unsigned int maxSharedmemory){
	dimBlock = 64;
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;

	pointsContainedDeviceSharedMemory<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float), stream>>>(data,
																								 centroids,
																								 dims,
																								 output,
																								 Csum_out,
																								 width,
																								 point_dim,
																								 no_data,
																								 number_of_samples,
																								 m,
																								 numberOfCentroids,
																								 centroidSharedMemorySize_f,
																								 dataSharedMemorySize_f,
																								 blocksWithSameCentroid);

};


void pointsContainedKernelSharedMemoryBreak(unsigned int dimGrid,
									   	    unsigned int dimBlock,
									   	    hipStream_t stream,
									   	    float* data,
									   	    unsigned int* centroids,
									   	    bool* dims,
									   	    bool* output,
									   	    unsigned int* Csum_out,
									   	    float width,
									   	    unsigned int point_dim,
									   	    unsigned int no_data,
									   	    unsigned int number_of_samples,
									   	    unsigned int m,
									   	    unsigned int numberOfCentroids,
									   	    unsigned int breakingIntervall){
	dimBlock = 64;
	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;


	pointsContainedDeviceSharedMemoryBreak<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float), stream>>>(data,
																								              centroids,
																								              dims,
																								              output,
																								              Csum_out,
																								              width,
																								              point_dim,
																								              no_data,
																								              number_of_samples,
																								              m,
																								              numberOfCentroids,
																								              centroidSharedMemorySize_f,
																								              dataSharedMemorySize_f,
																								              blocksWithSameCentroid,
																								              breakingIntervall);

};

/*
 * dimBlock needs to be multiple of 32.
 * dimBlock needs to be >= then point_dim
 */
void pointsContainedKernelSharedMemoryFewBank(unsigned int dimGrid,
											  unsigned int dimBlock,
											  hipStream_t stream,
											  float* data, unsigned int* centroids,
											  bool* dims,
											  bool* output,
											  unsigned int* Csum_out,
											  float width,
											  unsigned int point_dim,
											  unsigned int no_data,
											  unsigned int number_of_samples,
											  unsigned int m,
											  unsigned int numberOfCentroids){
	dimBlock = 64;
	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;

	//std::cout << "no_data_p " << no_data << std::endl << "point_dim " << point_dim << std::endl;
	/*//calculate how much we are going to use
	unsigned long long sharedMemorySize = ( centroidSharedMemorySize +dataSharedMemorySize )*sizeof(float);
	*/
	//std::cout << "sharedMemorySize " << dataSharedMemorySize << std::endl;

	//std::cout << "dimGridv2*dimBlock*20: " << dimGridv2*dimBlock*20 << std::endl;
	pointsContainedDeviceSharedMemoryFewBank<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float), stream>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m, numberOfCentroids,centroidSharedMemorySize_f,dataSharedMemorySize_f,blocksWithSameCentroid);
	//std::cout << "done with kernel call" << std::endl;
};

void pointsContainedKernelSharedMemoryFewerBank(unsigned int dimGrid,
												unsigned int dimBlock,
												hipStream_t stream,
												float* data,
												unsigned int* centroids,
												bool* dims,
												bool* output,
												unsigned int* Csum_out,
												float width,
												unsigned int point_dim,
												unsigned int no_data,
												unsigned int number_of_samples,
												unsigned int m,
												unsigned int numberOfCentroids){
	dimBlock = 64;

	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;

	//std::cout << "no_data_p " << no_data << std::endl << "point_dim " << point_dim << std::endl;
	/*//calculate how much we are going to use
	unsigned long long sharedMemorySize = ( centroidSharedMemorySize +dataSharedMemorySize )*sizeof(float);
	*/
	//std::cout << "sharedMemorySize " << dataSharedMemorySize << std::endl;

	//std::cout << "dimGridv2*dimBlock*20: " << dimGridv2*dimBlock*20 << std::endl;
	pointsContainedDeviceSharedMemoryFewerBank<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float)>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m, numberOfCentroids,centroidSharedMemorySize_f,dataSharedMemorySize_f,blocksWithSameCentroid);
	//std::cout << "done with kernel call" << std::endl;
};



void notBoolArray(unsigned int dimBlock,
				  hipStream_t stream,
				  bool* imputAndOutput,
				  std::size_t lenght){

	unsigned int dimGrid = lenght/dimBlock;
	if(lenght%dimBlock != 0){
		dimGrid++;
	}

	gpuNotBoolArray<<<dimGrid,dimBlock,0,stream>>>(imputAndOutput,lenght);
}





std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> pointsContained(std::vector<std::vector<bool>*>* dims,
																					   std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   int m, float width){
	

	// Calculaating sizes
	int point_dim = data->at(0)->size();
	int no_of_points = data->size();
	int no_of_dims = dims->size();
	int no_of_centroids = centroids->size();

	int floats_in_data = point_dim * no_of_points;
	int bools_in_dims = no_of_dims * point_dim;
	int bools_in_output = no_of_points * no_of_dims;
	int ints_in_output_count = no_of_dims;

	int size_of_data = floats_in_data*sizeof(float);
	int size_of_dims = bools_in_dims*sizeof(bool);
	int size_of_centroids = no_of_centroids*sizeof(unsigned int);
	int size_of_output = bools_in_output*sizeof(bool);
	int size_of_output_count = ints_in_output_count*sizeof(unsigned int);

	// allocating on the host
	float* data_h = (float*) malloc(size_of_data);
	bool* dims_h = (bool*) malloc(size_of_dims);
	unsigned int* centroids_h = (unsigned int*) malloc(size_of_centroids);
	bool* output_h = (bool*) malloc(size_of_output);
	unsigned int* output_count_h = (unsigned int*) malloc(size_of_output_count);

	// filling data array
	for(int i= 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	// filling dims array
	for(int i= 0; i < no_of_dims; i++){
		for(int j = 0; j < point_dim; j++){
			dims_h[i*point_dim+j] = dims->at(i)->at(j);
		}
	}

	// filling centroid array
	for(int i= 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	// allocating on device
	float* data_d;
	bool* dims_d;
	unsigned int* centroids_d;
	bool* output_d;
	unsigned int* output_count_d;

	hipMalloc((void **) &data_d, size_of_data);
	hipMalloc((void **) &dims_d, size_of_dims);
	hipMalloc((void **) &centroids_d, size_of_centroids);
	hipMalloc((void **) &output_d, size_of_output);
	hipMalloc((void **) &output_count_d, size_of_output_count);

	//Copy from host to device

	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
	hipMemcpy(dims_d, dims_h, size_of_dims, hipMemcpyHostToDevice);
	hipMemcpy(centroids_d, centroids_h, size_of_centroids, hipMemcpyHostToDevice);


	//Call kernel
	pointsContainedDeviceNaive<<<ceil((no_of_dims)/256.0), 256>>>(data_d,
																  centroids_d,
																  dims_d,
																  output_d,
																  output_count_d,
																  width, point_dim, no_of_points, no_of_dims, m);

	// pointsContainedWrapper(ceil((no_of_dims)/256.0), 256, NULL,
	// 					   data_d,
	// 					   centroids_d,
	// 					   dims_d,
	// 					   output_d,
	// 					   output_count_d,
	// 					   width, point_dim, no_of_points, no_of_dims, m,pointContainedSmem
	// 					   );


	// copy from device
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_count_h, output_count_d, size_of_output_count, hipMemcpyDeviceToHost);

	// construnct output
	auto output =  new std::vector<std::vector<bool>*>;
	auto output_count =  new std::vector<unsigned int>;


	for(int i = 0; i < no_of_dims; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < no_of_points; j++){
			a->push_back(output_h[i*no_of_points+j]);
		}
		output->push_back(a);
	}

	for(int i = 0; i < no_of_dims; i++){
		output_count->push_back(output_count_h[i]);
	}


	hipFree(data_d);
	hipFree(dims_d);
	hipFree(centroids_d);
	hipFree(output_d);
	hipFree(output_count_d);
	free(data_h);
	free(dims_h);
	free(centroids_h);
	free(output_h);
	free(output_count_h);


	return std::make_pair(output,output_count);
};



void pointsContainedWrapper(unsigned int dimGrid,
							unsigned int dimBlock,
							hipStream_t stream,
							float* data,
							unsigned int* centroids,
							bool* dims,
							bool* output,
							unsigned int* Csum_out,
							float width,
							unsigned int point_dim,
							unsigned int no_data,
							unsigned int no_dims,
							unsigned int m,
							pointContainedType type
							){

	if(type == pointContainedNaive){
		pointsContainedKernelNaive(dimGrid,
								   dimBlock,
								   stream,
								   data,
								   centroids,
								   dims,
								   output,
								   Csum_out,
								   width,
								   point_dim,
								   no_data,
								   no_dims,
								   m);
	}else if(type == pointContainedSmem){
		pointsContainedKernelSharedMemory(dimGrid,
										  dimBlock,
										  stream,
										  data,
										  centroids,
										  dims,
										  output,
										  Csum_out,
										  width,
										  point_dim,
										  no_data,
										  no_dims,
										  m,
										  ceilf((float)no_dims/m));
	}
};





