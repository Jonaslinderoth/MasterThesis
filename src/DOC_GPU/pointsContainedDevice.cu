#include "hip/hip_runtime.h"
#include "pointsContainedDevice.h"
#include "../randomCudaScripts/Utils.h"
#include "../randomCudaScripts/DeleteFromArray.h"
#include "whatDataInCentroid.h"
#include <assert.h>
#include <utility>
#include <vector>
#include <iostream>
#include "../randomCudaScripts/arrayEqual.h"

/*
 * This fuction returns if the points are in the hypercube made by the centroid by using a subset of the dimensions
 * It does not use anything fancy.
 */
__global__ void pointsContainedDeviceNaive(float* data, unsigned int* centroids, bool* dims, bool* output, unsigned int* Csum_out,
									  float width, unsigned int point_dim, unsigned int no_data, unsigned int no_dims, unsigned int m){
	// one kernel for each hypercube
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				//if(!(centroid_index < no_data)){
				//    printf("num_data: %u, centroid_index: %u, currentCentroid: %u \n", no_data, centroid_index, currentCentroid);
				//}
				assert(centroid_index < no_data);
				assert(entry*point_dim+i < no_dims*point_dim);
				assert(centroid_index*point_dim+i < no_data*point_dim);
				assert(j*point_dim+i < no_data*point_dim);
				const unsigned long entryDims = entry*point_dim+i;
				const float centro = data[centroid_index*point_dim+i];
				const float punto = data[j*point_dim+i];
				const float abss = abs(centro - punto);
				d &= (not (dims[entryDims])) || (abss < width);
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;

	}
}

/*
 * This does the same as the naive but it moved the data to shared memory before.
 * making
 *
 */
__global__ void pointsContainedDeviceSharedMemory(float* data,
		unsigned int* centroids,
		bool* dims,
		bool* output,
		unsigned int* Csum_out,
		float width,
		const unsigned int point_dim,
		const unsigned int no_data_p,
		const unsigned int no_dims,
		const unsigned int ammountOfSamplesThatUseSameCentroid,
		const unsigned int numberOfCentroids,
		const unsigned int centroidSharedMemorySize_f,
		const unsigned int dataSharedMemorySize_f,
		const unsigned int blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];


	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	if(threadIdx.x < point_dim){
		const long offsetByDimension_f = threadIdx.x;
		const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
		const float partOfACentroid_f = data[indexOfCentroidInData_f];
		centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	}

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long whatM = blockIdx.x / blocksWithSameCentroid;
	const unsigned long offEntry_p = howLongOnM_p + whatM * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;
	//assert(dataSharedMemorySize_f%point_dim == 0);
	//testing variables
	//const unsigned long i = 1;
	//const unsigned long j = 414;


	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){

			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;

			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;

			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f ){
				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];
				/*
				if(indexInSharedMemory_f >= j*point_dim and indexInSharedMemory_f < j*point_dim+4 and blockIdx.x == 0 ){
					printf("data[indexInData_f] %f \n indexInData_f: %lu \n indexInSharedMemory_f %lu \n whatM %lu \n ammountOfSamplesThatUseSameCentroid: %u \n",
																								data[indexInData_f],
																								indexInData_f,
																								indexInSharedMemory_f,
																								whatM,
																								ammountOfSamplesThatUseSameCentroid);
				}*/

			}
		}
		__syncthreads();
		/*
		if(offEntry_p == i){
			printf("done with movinf data from global dataSharedMemorySize_p %lu \n" , dataSharedMemorySize_p);
		}*/

		const unsigned long ammountOfPointsLeft = no_data_p-i_p;
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < dataSharedMemorySize_p ; indexDataSM_p++)
		{
			/*
			if(offEntry_p == i and indexDataSM_p == j){
				printf("in the for loop i_p: %lu \n" , i_p);
			}

			if(offEntry_p == i and indexDataSM_p == j){
				printf("before if statement \n");
			}*/

			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = indexDataSM_p;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;


			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and indexDataSM_p < ammountOfPointsLeft and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){

					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}

				//assert(indexOutput_p < ammountOfSamplesThatUseSameCentroid*numberOfCentroids*no_data_p);

				output[indexOutput_p] = d;
				Csum += d;
				/*
				assert(Csum == csum2 or Csum == csum2+1);

				if(offEntry_p == i and i_p + indexDataSM_p == j){
					printf(" fancy: %d \n indexDimsNoDimension_f %lu \n indexPointDataSM_p: %lu \n indexOutput_p: %lu \n centroidSharedMemory %f %f %f %f \n dataSharedMemory %f %f %f %f \n idexesInSharedMemory: %lu %lu %lu %lu \n threadId: %u \n blockId: %u \n dataSharedMemorySize_f: %u \n" , d ,
																		   indexDimsNoDimension_f,
																		   indexPointDataSM_p,
																		   indexOutput_p,
																		   centroidSharedMemory[0],
																		   centroidSharedMemory[1],
																		   centroidSharedMemory[2],
																		   centroidSharedMemory[3],
																		   dataSharedMemory[indexPointDataSM_p*point_dim+0],
																		   dataSharedMemory[indexPointDataSM_p*point_dim+1],
																		   dataSharedMemory[indexPointDataSM_p*point_dim+2],
																		   dataSharedMemory[indexPointDataSM_p*point_dim+3],
																		   indexPointDataSM_p*point_dim + 0,
																		   indexPointDataSM_p*point_dim + 1,
																		   indexPointDataSM_p*point_dim + 2,
																		   indexPointDataSM_p*point_dim + 3,
																		   threadIdx.x,
																		   blockIdx.x,
																		   dataSharedMemorySize_f);
				}
				*/
			}

		}
		__syncthreads();

	}

	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;
		/*
		if(offEntry_p == 1 and false){

			printf("howLongOnM_p: %lu \n blockIdx.x: %u \n blocksWithSameCentroid: %u \n ammountOfSamplesThatUseSameCentroid: %u \n Csum %u \n",howLongOnM_p,
																																	 	 	 	blockIdx.x,
																																	 	 	 	blocksWithSameCentroid,
																																	 	 	 	ammountOfSamplesThatUseSameCentroid,
																																	 	 	 	Csum);


		}*/

	}

}

__global__ void pointsContainedDeviceSharedMemoryFewBank(float* __restrict__ data,
		unsigned int* __restrict__ centroids,
		bool* __restrict__ dims,
		bool* __restrict__ output,
		unsigned int* __restrict__ Csum_out,
		float width,
		const unsigned long point_dim,
		const unsigned long no_data_p,
		const unsigned long no_dims,
		const unsigned long ammountOfSamplesThatUseSameCentroid,
		const unsigned long numberOfCentroids,
		const unsigned long centroidSharedMemorySize_f,
		const unsigned long dataSharedMemorySize_f,
		const unsigned long blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];

	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	if(threadIdx.x < point_dim){
		const long offsetByDimension_f = threadIdx.x;
		const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
		const float partOfACentroid_f = data[indexOfCentroidInData_f];
		centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	}
	//now i have the centroid in shared memory. but its going to cause bank conflicts :( , TODO fix this.

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long offEntry_p = howLongOnM_p + ( blockIdx.x / blocksWithSameCentroid ) * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;
	//assert(dataSharedMemorySize_f%point_dim == 0);



	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){

			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;

			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;

			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f){

				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];

			}
		}
		__syncthreads();


		const unsigned long warpId_p = 0;//threadIdx.x/(32*point_dim);
		const unsigned long warpIdToDim_p =threadIdx.x/32;
		const unsigned long dataPointsLeft = no_data_p-i_p;
		const unsigned long limit = min(dataPointsLeft,dataSharedMemorySize_p);
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < limit ; indexDataSM_p++)
		{

			//const unsigned long indexPointDataSM_p = indexDataSM_p;
			//const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = ( indexDataSM_p + warpId_p ) % limit;
			const unsigned long indexPointDataSMNotOffset_p = indexDataSM_p;
			const long offset_p = indexPointDataSM_p - indexPointDataSMNotOffset_p;
			//const unsigned long indexDimsNoDimension_f = ( offEntry_p + offset_p ) * point_dim;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p + offset_p;

			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){
					/*
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);
					*/

					const unsigned long offDimensionIndex = (dimensionIndex_f+warpIdToDim_p)%point_dim;
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + offDimensionIndex;
					const unsigned long indexDims_f = indexDimsNoDimension_f + offDimensionIndex;
					assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[offDimensionIndex];
					assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}

				//assert(indexOutput_p < ammountOfSamplesThatUseSameCentroid*numberOfCentroids*no_data_p);
				output[indexOutput_p] = d;

				Csum += d;
			}

		}
		__syncthreads();

	}
	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;
	}

}




__global__ void pointsContainedDeviceSharedMemoryFewerBank(float* __restrict__ data,
		unsigned int* __restrict__ centroids,
		bool* __restrict__ dims,
		bool* __restrict__ output,
		unsigned int* __restrict__ Csum_out,
		float width,
		const unsigned long point_dim,
		const unsigned long no_data_p,
		const unsigned long no_dims,
		const unsigned long ammountOfSamplesThatUseSameCentroid,
		const unsigned long numberOfCentroids,
		const unsigned long centroidSharedMemorySize_f,
		const unsigned long dataSharedMemorySize_f,
		const unsigned long blocksWithSameCentroid){
	extern __shared__ float sharedMemory[];

	//shared memory spit
	float* centroidSharedMemory = sharedMemory;
	float* dataSharedMemory = (float*)&sharedMemory[centroidSharedMemorySize_f];

	//we have that every blocksWithSameCentroid the centroid chances to the next one.
	const unsigned int indexOfCentroidToCentroids = blockIdx.x/blocksWithSameCentroid;
	const unsigned int indexOfCentroidInDataNoDims_f = centroids[indexOfCentroidToCentroids]*point_dim;
	//we want to move the centroid to shared memory.
	if(threadIdx.x < point_dim){
		const long offsetByDimension_f = threadIdx.x;
		const unsigned long indexOfCentroidInData_f = indexOfCentroidInDataNoDims_f + offsetByDimension_f;
		const float partOfACentroid_f = data[indexOfCentroidInData_f];
		centroidSharedMemory[offsetByDimension_f] = partOfACentroid_f;
	}
	//now i have the centroid in shared memory. but its going to cause bank conflicts :( , TODO fix this.

	const unsigned long howLongOnM_p = ( blockIdx.x % blocksWithSameCentroid ) * blockDim.x + threadIdx.x;
	const unsigned long offEntry_p = howLongOnM_p + ( blockIdx.x / blocksWithSameCentroid ) * ammountOfSamplesThatUseSameCentroid;
	unsigned int Csum = 0;
	//_f stand for float , and _p stand for point.

	//now we need to "work" on the data
	// times the threads will need to copy data from global to shared memory.
	const unsigned long no_data_f = no_data_p*point_dim;
	const unsigned long dataSharedMemorySize_p = dataSharedMemorySize_f/point_dim;
	//assert(dataSharedMemorySize_f%point_dim == 0);



	for(unsigned long i_p = 0 ; i_p < no_data_p ; i_p += dataSharedMemorySize_p){
		//copy the data from global to shared memory
		for(unsigned long indexCopy_f = 0 ; indexCopy_f < dataSharedMemorySize_f ; indexCopy_f+=blockDim.x){

			const unsigned long indexInSharedMemory_f = indexCopy_f + threadIdx.x;

			const unsigned long indexInData_f = i_p*point_dim + indexCopy_f +threadIdx.x;

			if(indexInData_f < no_data_f and indexInSharedMemory_f < dataSharedMemorySize_f){

				dataSharedMemory[indexInSharedMemory_f] = data[indexInData_f];

			}
		}
		__syncthreads();


		const unsigned long warpId_p = threadIdx.x/(32*point_dim);
		const unsigned long warpIdToDim_p =threadIdx.x/32;
		const unsigned long dataPointsLeft = no_data_p-i_p;
		const unsigned long limit = min(dataPointsLeft,dataSharedMemorySize_p);
		for(unsigned long indexDataSM_p = 0 ; indexDataSM_p < limit ; indexDataSM_p++)
		{

			//const unsigned long indexPointDataSM_p = indexDataSM_p;
			//const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p;
			const unsigned long indexDimsNoDimension_f = offEntry_p * point_dim;
			const unsigned long indexPointDataSM_p = ( indexDataSM_p + warpId_p ) % limit;
			const unsigned long indexPointDataSMNotOffset_p = indexDataSM_p;
			const long offset_p = indexPointDataSM_p - indexPointDataSMNotOffset_p;
			//const unsigned long indexDimsNoDimension_f = ( offEntry_p + offset_p ) * point_dim;
			const unsigned long indexOutput_p = offEntry_p * no_data_p + i_p + indexDataSM_p + offset_p;

			if( howLongOnM_p < ammountOfSamplesThatUseSameCentroid and offEntry_p < no_dims)
			{

				bool d = true;

				for(unsigned long dimensionIndex_f = 0 ; dimensionIndex_f < point_dim ; dimensionIndex_f++){
					/*
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + dimensionIndex_f;
					const unsigned long indexDims_f = indexDimsNoDimension_f + dimensionIndex_f;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[dimensionIndex_f];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);
					*/

					const unsigned long offDimensionIndex = (dimensionIndex_f+warpIdToDim_p)%point_dim;
					const unsigned long indexDataSM_f = indexPointDataSM_p*point_dim + offDimensionIndex;
					const unsigned long indexDims_f = indexDimsNoDimension_f + offDimensionIndex;
					//assert(indexDims_f < no_dims*point_dim);
					const bool dim = dims[indexDims_f];
					const float cen = centroidSharedMemory[offDimensionIndex];
					//assert(indexDataSM_f < dataSharedMemorySize_f);
					const float dat = dataSharedMemory[indexDataSM_f];
					d &= (not (dim)) || (abs(cen - dat) < width);

				}

				//assert(indexDataSM_p < dataSharedMemorySize_p);

				//assert(indexOutput_p < ammountOfSamplesThatUseSameCentroid*numberOfCentroids*no_data_p);
				output[indexOutput_p] = d;

				Csum += d;
			}

		}
		__syncthreads();

	}
	if(offEntry_p < no_dims and howLongOnM_p < ammountOfSamplesThatUseSameCentroid){
		Csum_out[offEntry_p] = Csum;

	}

}

__global__ void gpuWhereThingsGo(unsigned int* d_outData,
								const unsigned int* d_data,
								const unsigned int size){
	const size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	/*
	if(idx == 0){
		printf("size %u \n",size);
	}*/
	if(idx < size){
		d_outData[idx] = size+2;
		const unsigned int offset = d_data[idx];
		unsigned int nextOffset = offset+1;
		if(idx != size-1){
			nextOffset = d_data[idx+1];
		}

		if(offset == nextOffset){
			d_outData[idx] = idx-offset;
		}
	}

}





__global__ void gpuDimensionChanger(float* d_outData,
								    const unsigned int* d_wereThingsGoArray,
								    const float* d_data,
								    const unsigned int numElements,
								    const unsigned int dimensions,
								    const unsigned int dimensionRemaning){
	const size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	/*
	if(idx == 0){
		printf("numElements %u \n",numElements);
		printf("dimensions %u \n",dimensions);
		printf("dimensionRemaning %u \n",dimensionRemaning);
	}*/
	if(idx < numElements*dimensions){

		const size_t pointIdex = idx/dimensions;
		const size_t dimIndex = idx%dimensions;
		const size_t newPointIdex = pointIdex*dimensionRemaning;
		const size_t go = d_wereThingsGoArray[dimIndex];
		if(go<dimensions){
			const size_t newIndex = newPointIdex+go;
			const float theData = d_data[idx];
			d_outData[newIndex] = theData;
		}
	}

}

__global__ void gpuNotBoolArray(bool* inputAndOutPut,
							 std::size_t lenght){
	const size_t idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx < lenght){
		inputAndOutPut[idx] = not inputAndOutPut[idx];
	}
}


/*
 * This fuction returns if the points are in the hypercube made by the centroid by using a subset of the dimensions
 * It does not use anything fancy.
 */
__global__ void pointsContainedDeviceNaiveFewPoints(float* data,
													unsigned int* centroids,
													bool* output,
													unsigned int* Csum_out,
													const float width,
													const unsigned int point_dim,
													const unsigned int no_data,
													const unsigned int m,
													const unsigned int numberOfCentroids){

	const unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
	const unsigned int currentCentroid = idx/m;

	//check that we are not going out of bound on the centroids array
	if(idx < 1){
		const unsigned int centroidIndex = centroids[currentCentroid];
		const float* centroidArray = &data[centroidIndex*point_dim];
		unsigned int Csum = 0;

		for(unsigned int dataIndex = 0; dataIndex < no_data; ++dataIndex){

			const float* pointArray = &data[dataIndex*point_dim];
			bool d = true;
			for(unsigned int dimensionIndex = 0; dimensionIndex < point_dim; ++dimensionIndex){
				const float centroid_f = centroidArray[dimensionIndex];
				const float data_f = pointArray[dimensionIndex];
				const float diference = abs(centroid_f - data_f);
				d &= (diference < width);
			}
			output[(size_t)idx*(size_t)no_data+(size_t)dataIndex] = d;
			Csum += d;
			//printf(" no_data: %u \n dataIndex: %u \n",no_data,dataIndex);
		}

		Csum_out[idx] = Csum;
	}
	/*
	// one kernel for each hypercube
	unsigned int entry = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int currentCentroid = entry/m;
	if(entry < no_dims){
		//assert(currentCentroid < no_of_ps);
		// for each data point
		unsigned int Csum = 0;
		for(unsigned int j = 0; j < no_data; j++){
			// for all dimmensions in each hypercube / point
			bool d = true;
			for(unsigned int i = 0; i < point_dim; i++){
				//(not (dims[entry*point_dim+i])) ||
				unsigned int centroid_index = centroids[currentCentroid];
				//if(!(centroid_index < no_data)){
				//    printf("num_data: %u, centroid_index: %u, currentCentroid: %u \n", no_data, centroid_index, currentCentroid);
				//}
				assert(centroid_index < no_data);
				assert(entry*point_dim+i < no_dims*point_dim);
				assert(centroid_index*point_dim+i < no_data*point_dim);
				assert(j*point_dim+i < no_data*point_dim);
				const unsigned long entryDims = entry*point_dim+i;
				const float centro = data[centroid_index*point_dim+i];
				const float punto = data[j*point_dim+i];
				const float abss = abs(centro - punto);
				d &= (not (dims[entryDims])) || (abss < width);
			}
			assert(entry < no_dims);
			assert((size_t)entry*(size_t)no_data+(size_t)j < (size_t)no_dims*(size_t)no_data+(size_t)j);
			output[(size_t)entry*(size_t)no_data+(size_t)j] = d;
			Csum += d;
		}
		Csum_out[entry] = Csum;
	}
	*/
}



void pointsContainedKernelNaive(unsigned int dimGrid,
							    unsigned int dimBlock,
							    hipStream_t stream,
							    float* data,
							    unsigned int* centroids,
							    bool* dims, bool* output,
							    unsigned int* Csum_out,
							    float width,
							    unsigned int point_dim,
							    unsigned int no_data,
							    unsigned int number_of_samples,
							    unsigned int m){

	pointsContainedDeviceNaive<<<dimGrid, dimBlock, 0, stream>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m);


};

void pointsContainedKernelSharedMemory(unsigned int dimGrid,
									   unsigned int dimBlock,
									   hipStream_t stream,
									   float* data,
									   unsigned int* centroids,
									   bool* dims,
									   bool* output,
									   unsigned int* Csum_out,
									   float width,
									   unsigned int point_dim,
									   unsigned int no_data,
									   unsigned int number_of_samples,
									   unsigned int m,
									   unsigned int numberOfCentroids){

	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;


	pointsContainedDeviceSharedMemory<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float), stream>>>(data,
																								 centroids,
																								 dims,
																								 output,
																								 Csum_out,
																								 width,
																								 point_dim,
																								 no_data,
																								 number_of_samples,
																								 m,
																								 numberOfCentroids,
																								 centroidSharedMemorySize_f,
																								 dataSharedMemorySize_f,
																								 blocksWithSameCentroid);

};
/*
 * dimBlock needs to be multiple of 32.
 * dimBlock needs to be >= then point_dim
 */
void pointsContainedKernelSharedMemoryFewBank(unsigned int dimGrid,
											  unsigned int dimBlock,
											  hipStream_t stream,
											  float* data, unsigned int* centroids,
											  bool* dims,
											  bool* output,
											  unsigned int* Csum_out,
											  float width,
											  unsigned int point_dim,
											  unsigned int no_data,
											  unsigned int number_of_samples,
											  unsigned int m,
											  unsigned int numberOfCentroids){

	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;

	//std::cout << "no_data_p " << no_data << std::endl << "point_dim " << point_dim << std::endl;
	/*//calculate how much we are going to use
	unsigned long long sharedMemorySize = ( centroidSharedMemorySize +dataSharedMemorySize )*sizeof(float);
	*/
	//std::cout << "sharedMemorySize " << dataSharedMemorySize << std::endl;

	//std::cout << "dimGridv2*dimBlock*20: " << dimGridv2*dimBlock*20 << std::endl;
	pointsContainedDeviceSharedMemoryFewBank<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float), stream>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m, numberOfCentroids,centroidSharedMemorySize_f,dataSharedMemorySize_f,blocksWithSameCentroid);
	//std::cout << "done with kernel call" << std::endl;
};

void pointsContainedKernelSharedMemoryFewerBank(unsigned int dimGrid,
												unsigned int dimBlock,
												hipStream_t stream,
												float* data,
												unsigned int* centroids,
												bool* dims,
												bool* output,
												unsigned int* Csum_out,
												float width,
												unsigned int point_dim,
												unsigned int no_data,
												unsigned int number_of_samples,
												unsigned int m,
												unsigned int numberOfCentroids){

	unsigned long maxSharedmemory = 48000; //48kb can probably go up to more but...
	//we block takes care of only on centroid. a centroid is made of point_dim floats
	unsigned long centroidSharedMemorySize_f = point_dim;
	//how many blocks needed to cover m.
	unsigned long blocksWithSameCentroid = ceil((float)m/dimBlock);
	//we need blocksWithSameCentroid per centroid to cover all the sampels
	unsigned long dimGridv2 = blocksWithSameCentroid*numberOfCentroids;

	//need to know how much shared memory we are going to use.
	unsigned long dataSharedMemorySize_f = maxSharedmemory/sizeof(float)-centroidSharedMemorySize_f;
	dataSharedMemorySize_f = (dataSharedMemorySize_f/point_dim);
	dataSharedMemorySize_f = dataSharedMemorySize_f*point_dim;

	unsigned long sharedMemorySize_f = dataSharedMemorySize_f+centroidSharedMemorySize_f;

	//std::cout << "no_data_p " << no_data << std::endl << "point_dim " << point_dim << std::endl;
	/*//calculate how much we are going to use
	unsigned long long sharedMemorySize = ( centroidSharedMemorySize +dataSharedMemorySize )*sizeof(float);
	*/
	//std::cout << "sharedMemorySize " << dataSharedMemorySize << std::endl;

	//std::cout << "dimGridv2*dimBlock*20: " << dimGridv2*dimBlock*20 << std::endl;
	pointsContainedDeviceSharedMemoryFewerBank<<<dimGridv2, dimBlock, sharedMemorySize_f*sizeof(float)>>>(data, centroids, dims,
												 output, Csum_out,
												 width, point_dim, no_data, number_of_samples, m, numberOfCentroids,centroidSharedMemorySize_f,dataSharedMemorySize_f,blocksWithSameCentroid);
	//std::cout << "done with kernel call" << std::endl;
};


void pointsContainedKernelFewPoints(unsigned int dimGrid,
									unsigned int dimBlock,
									hipStream_t stream,
									float* data,
									unsigned int* centroids,
									bool* dims,
									bool* output,
									unsigned int* Csum_out,
									float width,
									unsigned int point_dim,
									unsigned int no_data,
									unsigned int m,
									unsigned int numberOfCentroids){

	/*
	//test stuff
	int size_of_dims = sizeof(bool)*point_dim;
	bool* h_dims = (bool*)malloc(size_of_dims);
	hipMemcpy(h_dims, dims, size_of_dims, hipMemcpyDeviceToHost);
	std::cout << "dims not negated" << std::endl;
	for(int i = 0 ; i < point_dim ; ++i){
		std::cout << h_dims[i] << " ";
	}
	std::cout << std::endl;
	*/

	//i want a prefix sum of what dimensions are used.
	int size_of_out_blelloch = sizeof(unsigned int)*(point_dim+1);


	unsigned int* d_out_blelloch;
	checkCudaErrors(hipMalloc(&d_out_blelloch, size_of_out_blelloch));
	/*
	//we try to not that array
	notBoolArray(dimBlock,stream,dims,point_dim);
	*/
	sum_scan_blelloch(stream, d_out_blelloch,dims,point_dim+1, true);

	unsigned int* h_out_blelloch;
	hipHostMalloc(&h_out_blelloch,sizeof(unsigned int));
	hipMemcpy(h_out_blelloch, d_out_blelloch+point_dim, sizeof(unsigned int), hipMemcpyDeviceToHost);
	/*
	//this was to test that it worked
	unsigned int* h_out_blelloch_all;
	hipHostMalloc(&h_out_blelloch_all,sizeof(unsigned int)*(point_dim+1));
	hipMemcpy(h_out_blelloch_all, d_out_blelloch, sizeof(unsigned int)*(point_dim+1), hipMemcpyDeviceToHost);

	std::cout << "out_blelloch" << std::endl;
	for(int i = 0 ; i < point_dim+1 ; ++i){
		std::cout << h_out_blelloch_all[i] << " ";
	}
	std::cout << std::endl;
	*/

	//std::cout << "h_out_blelloch[0] " << h_out_blelloch[0] << std::endl;
	//std::cout << "point_dim " << point_dim << std::endl;

	const unsigned int dimensionsLeft = point_dim-(h_out_blelloch[0]);
	/*
	std::cout << "h_out_blelloch[0] " << h_out_blelloch[0] << std::endl;
	std::cout << "point_dim " << point_dim << std::endl;
	std::cout << "dimensionsLeft " << dimensionsLeft << std::endl;
	std::cout << "m " << m << std::endl;
	*/


	unsigned int* d_out_whereThingsGo;
	checkCudaErrors(hipMalloc(&d_out_whereThingsGo, size_of_out_blelloch));

	const unsigned int dimBlockWhereThingsGo = dimBlock;
	unsigned int dimGridWhereThingsGo = point_dim/dimBlock;
	if(point_dim%dimBlock != 0){
		dimGridWhereThingsGo++;
	}

	gpuWhereThingsGo<<<dimGridWhereThingsGo,dimBlockWhereThingsGo,0,stream>>>(d_out_whereThingsGo,d_out_blelloch,point_dim);

	/*
	//this is for testing that the output is the right one
	unsigned int* h_out_whereThingsGo = (unsigned int*)malloc(size_of_out_blelloch);
	hipMemcpy(h_out_whereThingsGo, d_out_whereThingsGo, size_of_out_blelloch, hipMemcpyDeviceToHost);
	std::cout << "h_out_whereThingsGo" << std::endl;
	for(int i = 0 ; i < point_dim ; ++i){
		std::cout << h_out_whereThingsGo[i] << " ";
	}
	std::cout << std::endl;
	*/

	unsigned int size_of_reducedData = sizeof(float)*dimensionsLeft*no_data;
	//std::cout << "reducedDimension*no_data " << reducedDimension*no_data << std::endl;

	float* d_reducedData;
	checkCudaErrors(hipMalloc(&d_reducedData, size_of_reducedData));

	const unsigned int dimBlockgpuDimensionChanger = dimBlock;
	unsigned int dimGridgpuDimensionChanger = (no_data*point_dim)/dimBlock;
	if((no_data*point_dim)%dimBlock != 0){
		dimGridgpuDimensionChanger++;
	}
	//std::cout << "point_dim " << point_dim << std::endl;
	gpuDimensionChanger<<<dimGridgpuDimensionChanger,dimBlockgpuDimensionChanger,0,stream>>>(d_reducedData,d_out_whereThingsGo,data,no_data,point_dim,dimensionsLeft);


	/*
	//this is for testing that the output is the right one
	float* h_data = (float*)malloc(no_data*point_dim*sizeof(float));
	hipMemcpy(h_data, data, no_data*point_dim*sizeof(float), hipMemcpyDeviceToHost);
	std::cout << "dimensionsLeft " << dimensionsLeft << std::endl;
	std::cout << "data" << std::endl;
	for(int i = 0 ; i < no_data*point_dim ; ++i){
		std::cout << h_data[i] << " ";
	}
	std::cout << std::endl;

	float* h_out_gpuDimensionChanger = (float*)malloc(size_of_reducedData);
	hipMemcpy(h_out_gpuDimensionChanger, d_reducedData, size_of_reducedData, hipMemcpyDeviceToHost);
	std::cout << "h_out_gpuDimensionChanger" << std::endl;
	for(int i = 0 ; i < dimensionsLeft*no_data; ++i){
		std::cout << h_out_gpuDimensionChanger[i] << " ";
	}
	std::cout << std::endl;
	*/

	whatDataIsInCentroidFewPoints(stream,
								  dimBlockgpuDimensionChanger,
								  output,
								  Csum_out,
								  d_reducedData,
								  centroids,
								  width,
								  dimensionsLeft,
								  no_data);

	//pointsContainedDeviceNaiveFewPoints<<<dimGrid,dimBlock,0,stream>>>(d_reducedData,centroids,output,Csum_out,width,dimensionsLeft,no_data,m,numberOfCentroids);



};


void notBoolArray(unsigned int dimBlock,
				  hipStream_t stream,
				  bool* imputAndOutput,
				  std::size_t lenght){

	unsigned int dimGrid = lenght/dimBlock;
	if(lenght%dimBlock != 0){
		dimGrid++;
	}

	gpuNotBoolArray<<<dimGrid,dimBlock,0,stream>>>(imputAndOutput,lenght);
}





std::pair<std::vector<std::vector<bool>*>*,std::vector<unsigned int>*> pointsContained(std::vector<std::vector<bool>*>* dims,
																					   std::vector<std::vector<float>*>* data,
																					   std::vector<unsigned int>* centroids,
																					   int m, float width){

	// Calculaating sizes
	int point_dim = data->at(0)->size();
	int no_of_points = data->size();
	int no_of_dims = dims->size();
	int no_of_centroids = centroids->size();

	int floats_in_data = point_dim * no_of_points;
	int bools_in_dims = no_of_dims * point_dim;
	int bools_in_output = no_of_points * no_of_dims;
	int ints_in_output_count = no_of_dims;

	int size_of_data = floats_in_data*sizeof(float);
	int size_of_dims = bools_in_dims*sizeof(bool);
	int size_of_centroids = no_of_centroids*sizeof(unsigned int);
	int size_of_output = bools_in_output*sizeof(bool);
	int size_of_output_count = ints_in_output_count*sizeof(unsigned int);

	// allocating on the host
	float* data_h = (float*) malloc(size_of_data);
	bool* dims_h = (bool*) malloc(size_of_dims);
	unsigned int* centroids_h = (unsigned int*) malloc(size_of_centroids);
	bool* output_h = (bool*) malloc(size_of_output);
	unsigned int* output_count_h = (unsigned int*) malloc(size_of_output_count);

	// filling data array
	for(int i= 0; i < no_of_points; i++){
		for(int j = 0; j < point_dim; j++){
			data_h[i*point_dim+j] = data->at(i)->at(j);
		}
	}

	// filling dims array
	for(int i= 0; i < no_of_dims; i++){
		for(int j = 0; j < point_dim; j++){
			dims_h[i*point_dim+j] = dims->at(i)->at(j);
		}
	}

	// filling centroid array
	for(int i= 0; i < no_of_centroids; i++){
		centroids_h[i] = centroids->at(i);
	}

	// allocating on device
	float* data_d;
	bool* dims_d;
	unsigned int* centroids_d;
	bool* output_d;
	unsigned int* output_count_d;

	hipMalloc((void **) &data_d, size_of_data);
	hipMalloc((void **) &dims_d, size_of_dims);
	hipMalloc((void **) &centroids_d, size_of_centroids);
	hipMalloc((void **) &output_d, size_of_output);
	hipMalloc((void **) &output_count_d, size_of_output_count);

	//Copy from host to device

	hipMemcpy(data_d, data_h, size_of_data, hipMemcpyHostToDevice);
	hipMemcpy(dims_d, dims_h, size_of_dims, hipMemcpyHostToDevice);
	hipMemcpy(centroids_d, centroids_h, size_of_centroids, hipMemcpyHostToDevice);


	// Call kernel
	pointsContainedDeviceNaive<<<ceil((no_of_dims)/256.0), 256>>>(data_d, centroids_d, dims_d, output_d, output_count_d,
															 width, point_dim, no_of_points, no_of_dims, m);


	// copy from device
	hipMemcpy(output_h, output_d, size_of_output, hipMemcpyDeviceToHost);
	hipMemcpy(output_count_h, output_count_d, size_of_output_count, hipMemcpyDeviceToHost);

	// construnct output
	auto output =  new std::vector<std::vector<bool>*>;
	auto output_count =  new std::vector<unsigned int>;


	for(int i = 0; i < no_of_dims; i++){
		auto a =  new std::vector<bool>;
		for(int j = 0; j < no_of_points; j++){
			a->push_back(output_h[i*no_of_points+j]);
		}
		output->push_back(a);
	}

	for(int i = 0; i < no_of_dims; i++){
		output_count->push_back(output_count_h[i]);
	}


	hipFree(data_d);
	hipFree(dims_d);
	hipFree(centroids_d);
	hipFree(output_d);
	hipFree(output_count_d);
	free(data_h);
	free(dims_h);
	free(centroids_h);
	free(output_h);
	free(output_count_h);


	return std::make_pair(output,output_count);
};





